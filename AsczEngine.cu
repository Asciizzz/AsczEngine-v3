#include "hip/hip_runtime.h"
#include <FpsHandler.cuh>
#include <CsLogHandler.cuh>

#include <VertexShader.cuh>
#include <FragmentShader.cuh>
#include <SFMLTexture.cuh>

#include <Playground.cuh>

int main() {
    // Initialize Default stuff
    FpsHandler &FPS = FpsHandler::instance();
    CsLogHandler LOG = CsLogHandler();

    Graphic3D &GRAPHIC = Graphic3D::instance();
    GRAPHIC.setResolution(1600, 1000);

    Camera3D &CAMERA = GRAPHIC.camera;
    std::ifstream("cfg/cameraPos.txt") >> CAMERA.pos.x >> CAMERA.pos.y >> CAMERA.pos.z;
    std::ifstream("cfg/cameraSpd.txt") >> CAMERA.slowFactor >> CAMERA.fastFactor;

    SFMLTexture SFTex = SFMLTexture(1600, 1000);
    sf::RenderWindow window(sf::VideoMode(1600, 1000), "AsczEngine");
    window.setMouseCursorVisible(false);
    sf::Mouse::setPosition(sf::Vector2i(
        GRAPHIC.res_half.x, GRAPHIC.res_half.y
    ), window);

    std::string objPath = "";
    float objScale = 1;
    // File: <path> <scale>
    std::ifstream file("cfg/model.txt");
    file >> objPath >> objScale;

    // Create a .obj mesh (Work in progress)
    Mesh3D obj = Playground::readObjFile(0, objPath, true);
    obj.scale(Vec3f(), Vec3f(objScale));
    // obj.rotate(0, Vec3f(), Vec3f(0, 0, 0));

    // Graphing calculator for y = f(x, z)
    Vecs3f world;
    Vecs3f normal;
    Vecs2f texture;
    Vecs4f color;
    Vecs3x3uli faces;

    // Append points to the grid
    Vec2f rangeX(-100, 100);
    Vec2f rangeZ(-100, 100);
    Vec2f step(1, 1);

    int sizeX = (rangeX.y - rangeX.x) / step.x + 1;
    int sizeZ = (rangeZ.y - rangeZ.x) / step.y + 1;

    float maxY = -INFINITY;
    float minY = INFINITY;
    for (float x = rangeX.x; x <= rangeX.y; x += step.x) {
        for (float z = rangeZ.x; z <= rangeZ.y; z += step.y) {
            // World pos of the point
            // float y = sin(x / 10) * cos(z / 10) * 10;
            float y = rand() % 30 - 10;

            maxY = std::max(maxY, y);
            minY = std::min(minY, y);

            world.push_back(Vec3f(x, y, z));

            // x y in range (-0.01, 0.01)
            float nx = (rand() % 200 - 100) / 10000.0;
            float nz = (rand() % 200 - 100) / 10000.0;
            Vec3f n = Vec3f(nx, 1, nz);
            n.norm();
            normal.push_back(n);

            // x and z ratio (0 - 1)
            float ratioX = (x - rangeX.x) / (rangeX.y - rangeX.x);
            float ratioZ = (z - rangeZ.x) / (rangeZ.y - rangeZ.x);

            // Texture
            texture.push_back(Vec2f(ratioX, ratioZ));

            // Cool color
            color.push_back(Vec4f(40 * ratioX + 130, 255, 40 * ratioX + 130, 255));
        }
    }

    for (ULLInt i = 0; i < world.size(); i++) {
        // Set green color based on y value
        float ratioY = (world[i].y - minY) / (maxY - minY);
        color[i].y = 150 + 100 * ratioY;

        // Random ratio range from 0.8 to 1.2
        color[i].y *= 0.8 + 0.4 * (rand() % 100) / 100;
        color[i].y = std::min(255.0f, std::max(0.0f, color[i].y));
    }

    // Append faces to the grid
    for (ULLInt x = 0; x < sizeX - 1; x++) {
        for (ULLInt z = 0; z < sizeZ - 1; z++) {
            ULLInt i = x * sizeZ + z;
            faces.push_back(Vec3x3uli(i, i + 1, i + sizeZ));
            faces.push_back(Vec3x3uli(i + 1, i + sizeZ + 1, i + sizeZ));
        }
    }

    Mesh3D graph(1, world, normal, texture, color, faces);

    GRAPHIC.mesh += obj;
    // GRAPHIC.mesh += graph;
    GRAPHIC.allocateProjection();
    GRAPHIC.allocateEdges();

    // Free memory
    obj.free();

    // To avoid floating point errors
    // We will use a float that doesnt have a lot of precision
    float fovDeg = 90;

    // Cool rainbow effect for title
    double rainbowR = 255;
    double rainbowG = 0;
    double rainbowB = 0;
    short cycle = 0;

    while (window.isOpen()) {
        // Frame start
        FPS.startFrame();

        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed ||
                sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {
                window.close();
            }

            if (event.type == sf::Event::KeyPressed) {
                // Press f1 to toggle focus
                if (event.key.code == sf::Keyboard::F1) {
                    CAMERA.focus = !CAMERA.focus;
                    window.setMouseCursorVisible(!CAMERA.focus);
                    sf::Mouse::setPosition(sf::Vector2i(
                        GRAPHIC.res_half.x, GRAPHIC.res_half.y
                    ), window);
                }

                // Press L to read light.txt file and set its prop
                if (event.key.code == sf::Keyboard::L) {
                    std::ifstream dir("cfg/lightDir.txt");
                    dir >> GRAPHIC.light.dir.x >> GRAPHIC.light.dir.y >> GRAPHIC.light.dir.z;

                    std::ifstream color("cfg/lightColor.txt");
                    color >> GRAPHIC.light.color.x >> GRAPHIC.light.color.y >> GRAPHIC.light.color.z;
                }
            }

            // Scroll to zoom in/out
            if (event.type == sf::Event::MouseWheelScrolled) {
                if (event.mouseWheelScroll.delta > 0) fovDeg -= 10;
                else                                  fovDeg += 10;

                if (fovDeg < 10) fovDeg = 10;
                if (fovDeg > 170) fovDeg = 170;

                float fovRad = fovDeg * M_PI / 180;
                CAMERA.fov = fovRad;
            }
        }

        bool m_left = sf::Mouse::isButtonPressed(sf::Mouse::Left);
        bool m_right = sf::Mouse::isButtonPressed(sf::Mouse::Right);
        bool k_ctrl = sf::Keyboard::isKeyPressed(sf::Keyboard::LControl);
        bool k_shift = sf::Keyboard::isKeyPressed(sf::Keyboard::LShift);
        bool k_r = sf::Keyboard::isKeyPressed(sf::Keyboard::R);

        if (CAMERA.focus) {
            // Mouse movement handling
            sf::Vector2i mousepos = sf::Mouse::getPosition(window);
            sf::Mouse::setPosition(sf::Vector2i(
                GRAPHIC.res_half.x, GRAPHIC.res_half.y
            ), window);

            // Move from center
            int dMx = mousepos.x - GRAPHIC.res_half.x;
            int dMy = mousepos.y - GRAPHIC.res_half.y;

            // Camera look around
            CAMERA.rot.x -= dMy * CAMERA.mSens * FPS.dTimeSec;
            CAMERA.rot.y -= dMx * CAMERA.mSens * FPS.dTimeSec;
            CAMERA.restrictRot();
            CAMERA.updateMVP();

            // Mouse Click = move forward
            float vel = 0;
            // Move forward/backward
            if (m_left && !m_right)      vel = 20;
            else if (m_right && !m_left) vel = -20;
            else                         vel = 0;
            // Move slower/faster
            if (k_ctrl && !k_shift)      vel *= CAMERA.slowFactor;
            else if (k_shift && !k_ctrl) vel *= CAMERA.fastFactor;
            // Update camera World pos
            CAMERA.pos += CAMERA.forward * vel * FPS.dTimeSec;
        }

        // Press R to rotate the object
        if (k_r) {
            float rot = M_PI / 3 * FPS.dTimeSec;
            if (k_ctrl) rot *= -1;
            if (k_shift) rot *= 3;
            GRAPHIC.mesh.rotate(0, Vec3f(), Vec3f(0, rot, 0));
        }

        // ========== Render Pipeline ==========

        VertexShader::cameraProjection();
        VertexShader::createDepthMap();
        VertexShader::rasterization();

        FragmentShader::phongShading();

        // From buffer to texture
        // (clever way to incorporate CUDA into SFML)
        SFTex.updateTexture(
            GRAPHIC.buffer.color,
            GRAPHIC.buffer.width,
            GRAPHIC.buffer.height,
            GRAPHIC.pixelSize
        );

        // ========== Log handling ==========

        // Rainbow color
        double step = 120 * FPS.dTimeSec;
        if (cycle == 0) {
            rainbowG += step; rainbowR -= step;
            if (rainbowG >= 255) cycle = 1;
        } else if (cycle == 1) {
            rainbowB += step; rainbowG -= step;
            if (rainbowB >= 255) cycle = 2;
        } else if (cycle == 2) {
            rainbowR += step; rainbowB -= step;
            if (rainbowR >= 255) cycle = 0;
        }
        sf::Color rainbow = sf::Color(rainbowR, rainbowG, rainbowB);
        LOG.addLog("Welcome to AsczEngine 3.0", rainbow, 1);

        double gRatio = double(FPS.fps - 10) / 50;
        gRatio = std::max(0.0, std::min(gRatio, 1.0));
        sf::Color fpsColor((1 - gRatio) * 255, gRatio * 255, 0);
        LOG.addLog("FPS: " + std::to_string(FPS.fps), fpsColor);

        // Camera data
        LOG.addLog(CAMERA.data(), sf::Color::White);

        // Displays
        window.clear(sf::Color(0, 0, 0));
        window.draw(SFTex.sprite);
        LOG.drawLog(window);
        window.display();

        // Frame end
        FPS.endFrame();
    }

    // Clean up
    GRAPHIC.free();
    SFTex.free();

    return 0;
}
#include "hip/hip_runtime.h"
#include <FpsHandler.cuh>
#include <CsLogHandler.cuh>
#include <Render3D.cuh>

#include <SFMLTexture.cuh>

int main() {
    // Initialize Default stuff
    FpsHandler &FPS = FpsHandler::instance();
    CsLogHandler LOG = CsLogHandler();

    Render3D &RENDER = Render3D::instance();
    RENDER.setResolution(1600, 900);

    SFMLTexture SFTex = SFMLTexture(1600, 900);

    sf::RenderWindow window(sf::VideoMode(1600, 900), "AsczEngine");
    window.setMouseCursorVisible(false);

    // Graphing calculator for y = f(x, z)
    Vecs3f world;
    Vecs3f normal;
    Vecs2f texture;
    Vecs4f color;

    Vecs3uli faces;

    // Append points to the grid
    Vec2f rangeX(-100, 100);
    Vec2f rangeZ(-100, 100);
    Vec2f step(1, 1);

    int sizeX = (rangeX.y - rangeX.x) / step.x + 1;
    int sizeZ = (rangeZ.y - rangeZ.x) / step.y + 1;

    float maxY = -INFINITY;
    float minY = INFINITY;
    for (float x = rangeX.x; x <= rangeX.y; x += step.x) {
        for (float z = rangeZ.x; z <= rangeZ.y; z += step.y) {
            // World pos of the point
            float y = sin(x / 10) * cos(z / 10) * 10;
            // float y = rand() % 20 - 10;

            maxY = std::max(maxY, y);
            minY = std::min(minY, y);

            world.push_back(Vec3f(x, y, z));
            normal.push_back(Vec3f(0, 1, 0));

            // x and z ratio (0 - 1)
            float ratioX = (x - rangeX.x) / (rangeX.y - rangeX.x);
            float ratioZ = (z - rangeZ.x) / (rangeZ.y - rangeZ.x);

            // Texture
            texture.push_back(Vec2f(ratioX, ratioZ));

            // Cool color
            color.push_back(Vec4f(255 * ratioX, 255, 255 * ratioZ, 255));
        }
    }

    for (ULLInt i = 0; i < world.size(); i++) {
        // Set opacity based on the height
        float ratioY = (world[i].y - minY) / (maxY - minY);
        color[i].w = 100 + 150 * ratioY;
    }

    // Append faces to the grid
    for (ULLInt x = 0; x < sizeX - 1; x++) {
        for (ULLInt z = 0; z < sizeZ - 1; z++) {
            ULLInt i = x * sizeZ + z;
            faces.push_back(Vec3uli(i, i + 1, i + sizeZ));
            faces.push_back(Vec3uli(i + 1, i + sizeZ + 1, i + sizeZ));
        }
    }

    Mesh graph(0, world, normal, texture, color, faces);

    Mesh cube(1,
        Vecs3f({
            Vec3f(-1, -1, -1), Vec3f(1, -1, -1),
            Vec3f(1, 1, -1), Vec3f(-1, 1, -1),
            Vec3f(-1, -1, 1), Vec3f(1, -1, 1),
            Vec3f(1, 1, 1), Vec3f(-1, 1, 1)
        }),
        Vecs3f({
            Vec3f(-1, -1, -1), Vec3f(1, -1, -1),
            Vec3f(1, 1, -1), Vec3f(-1, 1, -1),
            Vec3f(-1, -1, 1), Vec3f(1, -1, 1),
            Vec3f(1, 1, 1), Vec3f(-1, 1, 1)
        }),
        Vecs2f({
            Vec2f(0, 0), Vec2f(1, 0),
            Vec2f(1, 1), Vec2f(0, 1),
            Vec2f(0, 0), Vec2f(1, 0),
            Vec2f(1, 1), Vec2f(0, 1)
        }),
        Vecs4f({ // Red Green Blue Yellow Cyan Magenta Orange Purple 
            Vec4f(255, 0, 0, 255), Vec4f(0, 255, 0, 255),
            Vec4f(0, 0, 255, 255), Vec4f(255, 255, 0, 255),
            Vec4f(0, 255, 255, 255), Vec4f(255, 0, 255, 255),
            Vec4f(255, 125, 0, 255), Vec4f(125, 0, 255, 255)
        }),
        Vecs3uli({
            Vec3uli(0, 1, 2), Vec3uli(0, 2, 3),
            Vec3uli(4, 5, 6), Vec3uli(4, 6, 7),
            Vec3uli(0, 4, 7), Vec3uli(0, 7, 3),
            Vec3uli(1, 5, 6), Vec3uli(1, 6, 2),
            Vec3uli(0, 1, 5), Vec3uli(0, 5, 4),
            Vec3uli(3, 2, 6), Vec3uli(3, 6, 7)
        })
    );

    RENDER.mesh += Mesh3D(graph);
    RENDER.allocateProjection();

    while (window.isOpen()) {
        // Frame start
        FPS.startFrame();
        window.clear(sf::Color::Black);

        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed ||
                sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {
                window.close();
            }

            if (event.type == sf::Event::KeyPressed) {
                // Press f1 to toggle focus
                if (event.key.code == sf::Keyboard::F1) {
                    RENDER.camera.focus = !RENDER.camera.focus;
                    window.setMouseCursorVisible(!RENDER.camera.focus);
                    sf::Mouse::setPosition(sf::Vector2i(
                        RENDER.res_half.x, RENDER.res_half.y
                    ), window);
                }
            }
        }

        if (RENDER.camera.focus) {
            // Mouse movement handling
            sf::Vector2i mousepos = sf::Mouse::getPosition(window);
            sf::Mouse::setPosition(sf::Vector2i(
                RENDER.res_half.x, RENDER.res_half.y
            ), window);

            // Move from center
            int dMx = mousepos.x - RENDER.res_half.x;
            int dMy = mousepos.y - RENDER.res_half.y;

            // Camera look around
            RENDER.camera.rot.x -= dMy * RENDER.camera.mSens * FPS.dTimeSec;
            RENDER.camera.rot.y -= dMx * RENDER.camera.mSens * FPS.dTimeSec;
            RENDER.camera.restrictRot();
            RENDER.camera.updateMVP();

            // Mouse Click = move forward
            float vel = 0;
            bool m_left = sf::Mouse::isButtonPressed(sf::Mouse::Left);
            bool m_right = sf::Mouse::isButtonPressed(sf::Mouse::Right);
            bool k_ctrl = sf::Keyboard::isKeyPressed(sf::Keyboard::LControl);
            bool k_shift = sf::Keyboard::isKeyPressed(sf::Keyboard::LShift);
            // Move forward/backward
            if (m_left && !m_right)      vel = 20;
            else if (m_right && !m_left) vel = -20;
            else                         vel = 0;
            // Move slower/faster
            if (k_ctrl && !k_shift)      vel *= 0.2;
            else if (k_shift && !k_ctrl) vel *= 4;
            // Update camera World pos
            RENDER.camera.pos += RENDER.camera.forward * vel * FPS.dTimeSec;
        }

        // Rotate the mesh
        // float rotY = M_PI_2 / 6 * FPS.dTimeSec;
        // RENDER.mesh.rotate(0, Vec3f(0, 0, 0), Vec3f(0, rotY, 0));

        // Render Pipeline
        RENDER.vertexProjection();

        // Not working for some reason
        RENDER.rasterizeFaces();
        SFTex.updateTexture(
            RENDER.buffer.color,
            RENDER.buffer.width,
            RENDER.buffer.height,
            RENDER.pixelSize
        );
        window.draw(SFTex.sprite);

        // Log handling

        // FPS <= 10: Fully Red
        // FPS >= 60: Fully Green
        double gRatio = double(FPS.fps - 10) / 50;
        gRatio = std::max(0.0, std::min(gRatio, 1.0));
        sf::Color fpsColor((1 - gRatio) * 255, gRatio * 255, 0);
        LOG.addLog("FPS: " + std::to_string(FPS.fps), fpsColor);
        LOG.drawLog(window);

        window.display();

        // Frame end
        FPS.endFrame();
    }

    return 0;
}
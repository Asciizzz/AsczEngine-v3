#include "hip/hip_runtime.h"
#include <FpsHandler.cuh>
#include <CsLogHandler.cuh>

#include <VertexShader.cuh>
#include <FragmentShader.cuh>
#include <SFMLTexture.cuh>

#include <Playground.cuh>

int main() {
    // Initialize Default stuff
    FpsHandler &FPS = FpsHandler::instance();
    CsLogHandler LOG = CsLogHandler();

    int width, height, pixelSize, tileWidth, tileHeight;
    // Note: higher pixelSize = lower resolution
    std::ifstream("cfg/resolution.txt")
        >> width >> height >> pixelSize >> tileWidth >> tileHeight;

    Graphic3D &GRAPHIC = Graphic3D::instance();
    GRAPHIC.setResolution(width, height, pixelSize);
    GRAPHIC.setTileSize(tileWidth, tileHeight);

    Camera3D &CAMERA = GRAPHIC.camera;
    std::ifstream("cfg/cameraPos.txt") >> CAMERA.pos.x >> CAMERA.pos.y >> CAMERA.pos.z;
    std::ifstream("cfg/cameraSpd.txt") >> CAMERA.slowFactor >> CAMERA.fastFactor;

    SFMLTexture SFTex = SFMLTexture(width, height);
    sf::RenderWindow window(sf::VideoMode(width, height), "AsczEngine");
    window.setMouseCursorVisible(false);
    sf::Mouse::setPosition(sf::Vector2i(
        GRAPHIC.res_half.x, GRAPHIC.res_half.y
    ), window);

    // ===================== INITIALIZATION =====================

    std::string objPath = "";
    float objScale = 1;
    // File: <path> <scale>
    std::ifstream file("cfg/model.txt");
    file >> objPath >> objScale;

    // Create a .obj mesh (Work in progress)
    Mesh3D obj = Playground::readObjFile(0, objPath, true);
    obj.scale(Vec3f(), Vec3f(objScale));
    // obj.rotate(0, Vec3f(), Vec3f(0, 0, 0));

    Vecs3f cubeWorld = {
        Vec3f(-1, -1, -1), Vec3f(1, -1, -1),
        Vec3f(1, 1, -1), Vec3f(-1, 1, -1),
        Vec3f(-1, -1, 1), Vec3f(1, -1, 1),
        Vec3f(1, 1, 1), Vec3f(-1, 1, 1)
    };
    Vecs3f cubeNormal = {
        Vec3f(-1, -1, -1), Vec3f(1, -1, -1),
        Vec3f(1, 1, -1), Vec3f(-1, 1, -1),
        Vec3f(-1, -1, 1), Vec3f(1, -1, 1),
        Vec3f(1, 1, 1), Vec3f(-1, 1, 1)
    };
    Vecs2f cubeTexture = {
        Vec2f(0, 0), Vec2f(1, 0),
        Vec2f(1, 1), Vec2f(0, 1),
        Vec2f(0, 0), Vec2f(1, 0),
        Vec2f(1, 1), Vec2f(0, 1)
    };
    Vecs4f cubeColor = {
        Vec4f(255, 0, 0, 255), Vec4f(0, 255, 0, 255),
        Vec4f(255, 255, 0, 255), Vec4f(0, 0, 255, 255), 
        Vec4f(0, 255, 255, 255), Vec4f(255, 0, 255, 255),
        Vec4f(255, 125, 0, 255), Vec4f(125, 0, 255, 255)
    };
    Vecs3ulli cubeFaces = {
        Vec3ulli(0, 1, 2), Vec3ulli(0, 2, 3),
        Vec3ulli(4, 5, 6), Vec3ulli(4, 6, 7),
        Vec3ulli(0, 4, 7), Vec3ulli(0, 7, 3),
        Vec3ulli(1, 5, 6), Vec3ulli(1, 6, 2),
        Vec3ulli(0, 1, 5), Vec3ulli(0, 5, 4),
        Vec3ulli(3, 2, 6), Vec3ulli(3, 6, 7)
    };
    Mesh3D cube(1, cubeWorld, cubeNormal, cubeTexture, cubeColor, cubeFaces);
    cube.scale(Vec3f(), Vec3f(4));

    // Create a white wall behind the cube
    float wallSize = 10;
    Vecs3f wallWorld = {
        Vec3f(-wallSize, -wallSize, wallSize), Vec3f(wallSize, -wallSize, wallSize),
        Vec3f(wallSize, wallSize, wallSize), Vec3f(-wallSize, wallSize, wallSize)
    };
    Vecs3f wallNormal = { // Facing towards the cube
        Vec3f(0, 0, -1), Vec3f(0, 0, -1),
        Vec3f(0, 0, -1), Vec3f(0, 0, -1)
    };
    Vecs2f wallTexture = {
        Vec2f(0, 0), Vec2f(1, 0),
        Vec2f(1, 1), Vec2f(0, 1)
    };
    Vecs4f wallColor = {
        Vec4f(255, 125, 125, 255), Vec4f(125, 255, 125, 255),
        Vec4f(125, 125, 255, 255), Vec4f(255, 255, 125, 255)
    };
    Vecs3ulli wallFaces = {
        Vec3ulli(0, 1, 2), Vec3ulli(0, 2, 3)
    };
    Mesh3D wall(2, wallWorld, wallNormal, wallTexture, wallColor, wallFaces);

    // Graphing calculator for y = f(x, z)
    Vecs3f world;
    Vecs3f normal;
    Vecs2f texture;
    Vecs4f color;
    Vecs3ulli faces;

    // Append points to the grid
    Vec2f rangeX(-500, 500);
    Vec2f rangeZ(-500, 500);
    Vec2f step(1, 1);

    int sizeX = (rangeX.y - rangeX.x) / step.x + 1;
    int sizeZ = (rangeZ.y - rangeZ.x) / step.y + 1;

    float maxY = -INFINITY;
    float minY = INFINITY;
    int numX = 0;
    int numZ = 0;
    for (float x = rangeX.x; x <= rangeX.y; x += step.x) {
        numX++;

        for (float z = rangeZ.x; z <= rangeZ.y; z += step.y) {
            numZ++;

            // World pos of the point
            float y = sin(x / 50) * cos(z / 50) * 50;
            // float y = rand() % 30 - 10;

            maxY = std::max(maxY, y);
            minY = std::min(minY, y);

            world.push_back(Vec3f(x, y, z));

            // x and z ratio (0 - 1)
            float ratioX = (x - rangeX.x) / (rangeX.y - rangeX.x);
            float ratioZ = (z - rangeZ.x) / (rangeZ.y - rangeZ.x);
            // Texture
            texture.push_back(Vec2f(ratioX, ratioZ));
        }
    }
    numZ /= numX;

    for (ULLInt i = 0; i < world.size(); i++) {
        // Set color based on ratio
        float r = (world[i].x - rangeX.x) / (rangeX.y - rangeX.x);
        float g = (world[i].y - minY) / (maxY - minY);
        float b = (world[i].z - rangeZ.x) / (rangeZ.y - rangeZ.x);
        color.push_back(Vec4f(255 - r * 255, g * 255, b * 255, 255));

        // Set normal based on the triangle of surrounding points
        int x = i / numZ;
        int z = i % numZ;

        int edge = 10;
        if (x < edge || x >= numX - edge || z < edge || z >= numZ - edge) {
            normal.push_back(Vec3f(0, 1, 0));
            continue;
        }

        if (x % 100 == 0 || z % 100 == 0) {
            normal.push_back(Vec3f(0, 1, 0));
            continue;
        }

        int idxLeft = x * numZ + z - 1;
        int idxRight = x * numZ + z + 1;
        int idxUp = (x - 1) * numZ + z;
        int idxDown = (x + 1) * numZ + z;

        std::vector<int> idxDir = {
            idxLeft, idxRight, idxUp, idxDown
        };

        // Triangle group: mid left up, mid up right, mid right down, mid down left
        std::vector<Vec3f> triNormals;

        for (int j = 0; j < 4; j++) {
            int idx = idxDir[j];
            Vec3f mid = world[i];
            Vec3f left = world[idxLeft];
            Vec3f right = world[idxRight];
            Vec3f up = world[idxUp];
            Vec3f down = world[idxDown];

            if (j == 0) triNormals.push_back((mid - left) & (up - left));
            if (j == 1) triNormals.push_back((mid - up) & (right - up));
            if (j == 2) triNormals.push_back((mid - right) & (down - right));
            if (j == 3) triNormals.push_back((mid - down) & (left - down));
        }

        Vec3f avgNormal = Vec3f();
        for (Vec3f triNormal : triNormals) {
            avgNormal += triNormal;
        }
        avgNormal.norm();
        normal.push_back(avgNormal);
    }

    // Append faces to the grid
    for (ULLInt x = 0; x < sizeX - 1; x++) {
        for (ULLInt z = 0; z < sizeZ - 1; z++) {
            ULLInt i = x * sizeZ + z;
            faces.push_back(Vec3ulli(i, i + 1, i + sizeZ));
            faces.push_back(Vec3ulli(i + 1, i + sizeZ + 1, i + sizeZ));
        }
    }

    Mesh3D graph(3, world, normal, texture, color, faces);

    GRAPHIC += obj;
    // GRAPHIC.mesh += cube;
    // GRAPHIC.mesh += wall;
    // GRAPHIC += graph;

    GRAPHIC.mallocGFaces();
    GRAPHIC.mallocFaceStreams();

    // To avoid floating point errors
    // We will use a float that doesnt have a lot of precision
    float fovDeg = 90;

    // Cool rainbow effect for title
    double rainbowR = 255;
    double rainbowG = 0;
    double rainbowB = 0;
    short cycle = 0;

    while (window.isOpen()) {
        // Frame start
        FPS.startFrame();

        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed ||
                sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {
                window.close();
            }

            if (event.type == sf::Event::KeyPressed) {
                // Press f1 to toggle focus
                if (event.key.code == sf::Keyboard::F1) {
                    CAMERA.focus = !CAMERA.focus;
                    window.setMouseCursorVisible(!CAMERA.focus);
                    sf::Mouse::setPosition(sf::Vector2i(
                        GRAPHIC.res_half.x, GRAPHIC.res_half.y
                    ), window);
                }

                // Press L to read light.txt file and set its prop
                if (event.key.code == sf::Keyboard::L) {
                    std::ifstream dir("cfg/lightDir.txt");
                    dir >> GRAPHIC.light.dir.x >> GRAPHIC.light.dir.y >> GRAPHIC.light.dir.z;

                    std::ifstream color("cfg/lightColor.txt");
                    color >> GRAPHIC.light.color.x >> GRAPHIC.light.color.y >> GRAPHIC.light.color.z;
                }

                // Press f2 to set the resolution and pixel size again
                // Do not use this as I just introduced tile-based rasterization
                // if (event.key.code == sf::Keyboard::F2) {
                //     std::ifstream("cfg/resolution.txt") >> width >> height >> pixelSize;
                //     GRAPHIC.setResolution(width, height, pixelSize);
                //     SFTex.free();
                //     SFTex.resize(width, height);
                    
                //     window.setSize(sf::Vector2u(width, height));
                // }
            }

            // Scroll to zoom in/out
            if (event.type == sf::Event::MouseWheelScrolled) {
                if (event.mouseWheelScroll.delta > 0) fovDeg -= 10;
                else                                  fovDeg += 10;

                if (fovDeg < 10) fovDeg = 10;
                if (fovDeg > 170) fovDeg = 170;

                float fovRad = fovDeg * M_PI / 180;
                CAMERA.fov = fovRad;
            }
        }

        bool m_left = sf::Mouse::isButtonPressed(sf::Mouse::Left);
        bool m_right = sf::Mouse::isButtonPressed(sf::Mouse::Right);
        bool k_ctrl = sf::Keyboard::isKeyPressed(sf::Keyboard::LControl);
        bool k_shift = sf::Keyboard::isKeyPressed(sf::Keyboard::LShift);
        bool k_r = sf::Keyboard::isKeyPressed(sf::Keyboard::R);
        bool k_q = sf::Keyboard::isKeyPressed(sf::Keyboard::Q);
        bool k_e = sf::Keyboard::isKeyPressed(sf::Keyboard::E);

        if (CAMERA.focus) {
            // Mouse movement handling
            sf::Vector2i mousepos = sf::Mouse::getPosition(window);
            sf::Mouse::setPosition(sf::Vector2i(
                GRAPHIC.res_half.x, GRAPHIC.res_half.y
            ), window);

            // Move from center
            int dMx = mousepos.x - GRAPHIC.res_half.x;
            int dMy = mousepos.y - GRAPHIC.res_half.y;

            // Camera look around
            CAMERA.rot.x -= dMy * CAMERA.mSens * FPS.dTimeSec;
            CAMERA.rot.y -= dMx * CAMERA.mSens * FPS.dTimeSec;
            CAMERA.restrictRot();
            CAMERA.updateMVP();

            // Mouse Click = move forward
            float vel = 0;
            // Move forward/backward
            if (m_left && !m_right)      vel = 20;
            else if (m_right && !m_left) vel = -20;
            else                         vel = 0;
            // Move slower/faster
            if (k_ctrl && !k_shift)      vel *= CAMERA.slowFactor;
            else if (k_shift && !k_ctrl) vel *= CAMERA.fastFactor;
            // Update camera World pos
            CAMERA.pos += CAMERA.forward * vel * FPS.dTimeSec;
        }

        // Press R to rotate the object
        if (k_r) {
            float rot = M_PI / 3 * FPS.dTimeSec;
            if (k_ctrl) rot *= -1;
            if (k_shift) rot *= 3;
            GRAPHIC.mesh.rotate(0, Vec3f(), Vec3f(0, rot, 0));
        }
        // Press Q to rotate light source in x axis
        if (k_q) {
            float rot = M_PI / 3 * FPS.dTimeSec;
            if (k_ctrl) rot *= -1;
            if (k_shift) rot *= 3;

            GRAPHIC.light.dir.rotate(Vec3f(0), Vec3f(rot, 0, 0));
        }
        // Press E to rotate light source in z axis
        if (k_e) {
            float rot = M_PI / 3 * FPS.dTimeSec;
            if (k_ctrl) rot *= -1;
            if (k_shift) rot *= 3;

            GRAPHIC.light.dir.rotate(Vec3f(0), Vec3f(0, 0, rot));
        }

        // ========== Render Pipeline ==========

        VertexShader::cameraProjection();
        VertexShader::filterVisibleFaces();
        VertexShader::createDepthMap();
        VertexShader::rasterization();

        FragmentShader::phongShading();

        // Custom Fragment Shader
        FragmentShader::customFragmentShader();

        // From buffer to texture
        // (clever way to incorporate CUDA into SFML)
        SFTex.updateTexture(
            GRAPHIC.buffer.color,
            GRAPHIC.buffer.width,
            GRAPHIC.buffer.height,
            GRAPHIC.pixelSize
        );

        // ========== Log handling ==========

        // Rainbow title
        double step = 120 * FPS.dTimeSec;
        if (cycle == 0) {
            rainbowG += step; rainbowR -= step;
            if (rainbowG >= 255) cycle = 1;
        } else if (cycle == 1) {
            rainbowB += step; rainbowG -= step;
            if (rainbowB >= 255) cycle = 2;
        } else if (cycle == 2) {
            rainbowR += step; rainbowB -= step;
            if (rainbowR >= 255) cycle = 0;
        }
        sf::Color rainbow = sf::Color(rainbowR, rainbowG, rainbowB);

        // Dynamic FPS color
        double gRatio = double(FPS.fps - 10) / 50;
        gRatio = std::max(0.0, std::min(gRatio, 1.0));
        sf::Color fpsColor((1 - gRatio) * 255, gRatio * 255, 0);

        // Log all the data
        LOG.addLog("Welcome to AsczEngine 3.0", rainbow, 1);
        LOG.addLog("FPS: " + std::to_string(FPS.fps), fpsColor);
        LOG.addLog(
            "Screen:\n| Res: " + std::to_string(width) +
            " x " + std::to_string(height) +
            " | Pixel Size: " + std::to_string(pixelSize) + "\n" +
            "| Tile Size: " + std::to_string(tileWidth) + " x " + std::to_string(tileHeight) + "\n" +
            "| Visible Face: " + std::to_string(GRAPHIC.numVisibFs) + " / " + std::to_string(GRAPHIC.mesh.numFs),
            sf::Color(255, 160, 160)
        );
        LOG.addLog(CAMERA.data(), sf::Color(160, 160, 255));
        LOG.addLog(GRAPHIC.light.data(), sf::Color(160, 255, 160));

        // Displays
        window.clear(sf::Color(0, 0, 0));
        window.draw(SFTex.sprite);
        LOG.drawLog(window);
        window.display();

        // Frame end
        FPS.endFrame();
    }

    // Clean up
    GRAPHIC.free();
    SFTex.free();

    return 0;
}
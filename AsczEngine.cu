#include "hip/hip_runtime.h"
#include <FpsHandler.cuh>
#include <CsLogHandler.cuh>
#include <Utility.cuh>

#include <VertexShader.cuh>
#include <FragmentShader.cuh>
#include <SFMLTexture.cuh>

// Playgrounds
#include <SolarSystem.cuh>
#include <DotObj.cuh>

// Main
int main() {
    // Initialize Default stuff
    FpsHandler &FPS = FpsHandler::instance();
    CsLogHandler LOG = CsLogHandler();

    int width, height, pixelSize, tileSizeX, tileSizeY;
    // Note: higher pixelSize = lower resolution
    std::ifstream("assets/cfg/resolution.txt")
        >> width >> height >> pixelSize >> tileSizeX >> tileSizeY;

    Graphic3D &GRAPHIC = Graphic3D::instance();
    GRAPHIC.setResolution(width, height, pixelSize);
    GRAPHIC.createRuntimeStreams();

    Camera3D &CAMERA = GRAPHIC.camera;
    std::ifstream("assets/cfg/cameraPos.txt")
        >> CAMERA.pos.x >> CAMERA.pos.y >> CAMERA.pos.z;
    std::ifstream("assets/cfg/cameraSpd.txt")
        >> CAMERA.velSpec >> CAMERA.slowFactor >> CAMERA.fastFactor;
    std::ifstream("assets/cfg/cameraView.txt")
        >> CAMERA.near >> CAMERA.far;

    SFMLTexture SFTex = SFMLTexture(width, height);
    sf::RenderWindow window(sf::VideoMode(width, height), "AsczEngine");
    window.setMouseCursorVisible(false);
    sf::Mouse::setPosition(sf::Vector2i(
        GRAPHIC.res_half.x, GRAPHIC.res_half.y
    ), window);

    // ===================== INITIALIZATION =====================
    // Each model in models.txt will contain:
    // src scl rotX rotY rotZ transX transY transZ
    DotObj dotObjs("assets/cfg/models.txt");
    GRAPHIC.mesh.push(dotObjs.objs);

    SolarSystem solarSystem;
    solarSystem.setStars(4, 400, 6000, 8000, 6);
    GRAPHIC.mesh.push(solarSystem.stars);

    GRAPHIC.mallocRuntimeFaces();

    std::string texturePath = "";
    std::ifstream("assets/cfg/texture.txt") >> texturePath;
    GRAPHIC.createTexture(texturePath);

    int shdwWidth, shdwHeight, shdwTileSizeX, shdwTileSizeY;
    std::ifstream("assets/cfg/shadow.txt") >> shdwWidth >> shdwHeight >> shdwTileSizeX >> shdwTileSizeY;
    GRAPHIC.createShadowMap(shdwWidth, shdwHeight, shdwTileSizeX, shdwTileSizeY);

    // To avoid floating point errors
    // We will use a float that doesnt have a lot of precision
    float fovDeg = 90;

    // Cool rainbow effect for title
    Vec3f rainbow;
    short cycle = 0;

    // Turn on/off features
    bool textureMode = true;
    bool shadowMode = false;
    bool shadeMode = true;
    bool customMode = false;

    bool moveMode = true;

    // Other miscellaneus stuff
    bool k_t_hold = false;

    // =====================================================
    // ===================== MAIN LOOP =====================
    // =====================================================

    while (window.isOpen()) {
        // Frame start
        FPS.startFrame();

        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed ||
                sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {
                window.close();
            }

            if (event.type == sf::Event::KeyPressed) {
                // Press f1 to toggle focus
                if (event.key.code == sf::Keyboard::F1) {
                    CAMERA.focus = !CAMERA.focus;
                    window.setMouseCursorVisible(!CAMERA.focus);
                    sf::Mouse::setPosition(sf::Vector2i(
                        GRAPHIC.res_half.x, GRAPHIC.res_half.y
                    ), window);
                }
                
                // Press f2 to read texture.txt file and set its prop
                if (event.key.code == sf::Keyboard::F2) {
                    std::string texturePath = "";
                    std::ifstream("assets/cfg/texture.txt") >> texturePath;
                    GRAPHIC.createTexture(texturePath);
                }

                // Press 1 to toggle texture mode
                if (event.key.code == sf::Keyboard::Num1)
                    textureMode = !textureMode;
                // Press 2 to toggle shadow mode
                if (event.key.code == sf::Keyboard::Num2)
                    shadowMode = !shadowMode;
                // Press 3 to toggle shade mode
                if (event.key.code == sf::Keyboard::Num3)
                    shadeMode = !shadeMode;
                // Press 4 to toggle custom mode
                if (event.key.code == sf::Keyboard::Num4)
                    customMode = !customMode;

                // Press Z to toggle move mode
                if (event.key.code == sf::Keyboard::Z)
                    moveMode = !moveMode;

                // Press L to read light.txt file and set its prop
                if (event.key.code == sf::Keyboard::L) {
                    std::ifstream dir("assets/cfg/lightDir.txt");
                    dir >> GRAPHIC.light.dir.x >> GRAPHIC.light.dir.y >> GRAPHIC.light.dir.z;

                    std::ifstream color("assets/cfg/lightColor.txt");
                    color >> GRAPHIC.light.color.x >> GRAPHIC.light.color.y >> GRAPHIC.light.color.z;
                }

                // Press C to place a cube
                if (event.key.code == sf::Keyboard::C) {
                    Mesh cube = Utils::readObjFile(
                        "assets/Models/Shapes/Cube2.obj", 1, 1, true
                    );
                    cube.scaleIni(Vec3f(), Vec3f(0.5));
                    Vec3f place = CAMERA.pos + CAMERA.forward * 1;
                    float gridX = .5 + int(place.x);
                    float gridY = .5 + int(place.y);
                    float gridZ = .5 + int(place.z);

                    Vec3f grid = Vec3f(gridX, gridY, gridZ);

                    cube.translateIni(grid);

                    GRAPHIC.mesh.push(cube);
                    GRAPHIC.mallocRuntimeFaces();
                }
            }

            // Scroll to zoom in/out
            if (event.type == sf::Event::MouseWheelScrolled) {
                if (event.mouseWheelScroll.delta > 0) fovDeg -= 10;
                else                                  fovDeg += 10;

                if (fovDeg < 10) fovDeg = 10;
                if (fovDeg > 170) fovDeg = 170;

                float fovRad = fovDeg * M_PI / 180;
                CAMERA.fov = fovRad;
            }
        }

        bool m_left = sf::Mouse::isButtonPressed(sf::Mouse::Left);
        bool m_right = sf::Mouse::isButtonPressed(sf::Mouse::Right);
        bool k_ctrl = sf::Keyboard::isKeyPressed(sf::Keyboard::LControl);
        bool k_shift = sf::Keyboard::isKeyPressed(sf::Keyboard::LShift);

        bool k_w = sf::Keyboard::isKeyPressed(sf::Keyboard::W);
        bool k_a = sf::Keyboard::isKeyPressed(sf::Keyboard::A);
        bool k_s = sf::Keyboard::isKeyPressed(sf::Keyboard::S);
        bool k_d = sf::Keyboard::isKeyPressed(sf::Keyboard::D);
        bool k_space = sf::Keyboard::isKeyPressed(sf::Keyboard::Space);

        bool k_q = sf::Keyboard::isKeyPressed(sf::Keyboard::Q);
        bool k_e = sf::Keyboard::isKeyPressed(sf::Keyboard::E);
        bool k_t = sf::Keyboard::isKeyPressed(sf::Keyboard::T);

        // Mouse movement => Look around
        if (CAMERA.focus) {
            sf::Vector2i mousepos = sf::Mouse::getPosition(window);
            sf::Mouse::setPosition(sf::Vector2i(
                GRAPHIC.res_half.x, GRAPHIC.res_half.y
            ), window);

            // Move from center
            int dMx = mousepos.x - GRAPHIC.res_half.x;
            int dMy = mousepos.y - GRAPHIC.res_half.y;


            // Camera look around
            CAMERA.rot.x -= dMy * CAMERA.mSens * FPS.dTimeSec;
            CAMERA.rot.y += dMx * CAMERA.mSens * FPS.dTimeSec;
        }

        // Update camera
        CAMERA.update();

        // Csgo perspective mode
        if (CAMERA.focus && !moveMode) {
            float vel = CAMERA.velSpec;
            // Hold ctrl to go slow, hold shift to go fast
            if (k_ctrl && !k_shift)      vel *= CAMERA.slowFactor;
            else if (k_shift && !k_ctrl) vel *= CAMERA.fastFactor;
            // Press W/S to move forward/backward
            if (k_w && !k_s) CAMERA.pos += CAMERA.forward * vel * FPS.dTimeSec;
            if (k_s && !k_w) CAMERA.pos -= CAMERA.forward * vel * FPS.dTimeSec;
            // Press A/D to move left/right
            if (k_a && !k_d) CAMERA.pos += CAMERA.right * vel * FPS.dTimeSec;
            if (k_d && !k_a) CAMERA.pos -= CAMERA.right * vel * FPS.dTimeSec;
        }

        if (CAMERA.focus && moveMode) {
            // Gravity
            CAMERA.vel.y -= 1.15 * FPS.dTimeSec;

            // On ground
            if (CAMERA.pos.y < 1.5) {
                CAMERA.vel.y = 0;
                CAMERA.pos.y = 1.5;
            }

            // Jump
            if (k_space && abs(CAMERA.vel.y) < 0.01) CAMERA.vel.y = .3;

            float vel_xz = sqrt(
                CAMERA.vel.x * CAMERA.vel.x + CAMERA.vel.z * CAMERA.vel.z
            );

            // Move
            bool moving = false;
            if (k_w && !k_s) {
                moving = true;
                CAMERA.vel.x += CAMERA.forward.x * FPS.dTimeSec;
                CAMERA.vel.z += CAMERA.forward.z * FPS.dTimeSec;
            }
            if (k_s && !k_w) {
                moving = true;
                CAMERA.vel.x -= CAMERA.forward.x * FPS.dTimeSec;
                CAMERA.vel.z -= CAMERA.forward.z * FPS.dTimeSec;
            }
            if (k_a && !k_d) {
                moving = true;
                CAMERA.vel.x += CAMERA.right.x * FPS.dTimeSec;
                CAMERA.vel.z += CAMERA.right.z * FPS.dTimeSec;
            }
            if (k_d && !k_a) {
                moving = true;
                CAMERA.vel.x -= CAMERA.right.x * FPS.dTimeSec;
                CAMERA.vel.z -= CAMERA.right.z * FPS.dTimeSec;
            }
            if (vel_xz > 0 && !moving) {
                CAMERA.vel.x /= 1.5;
                CAMERA.vel.z /= 1.5;
            }

            // Limit and restrict horizontal speed
            if (vel_xz > 1) {
                CAMERA.vel.x /= vel_xz;
                CAMERA.vel.z /= vel_xz;
            }

            CAMERA.pos += CAMERA.vel * .1;
        }

        // Press T to read an transform.txt file and apply it
        // Note: hold ctrl to switch keyT from hold to tap
        if (k_t && (!k_t_hold || !k_ctrl)) {
            k_t_hold = true;

            Utils::applyTransformation(dotObjs.objs);
        }
        if (!k_t) k_t_hold = false;

        // ========== Playgrounds ==============

        // Set light position to camera position
        GRAPHIC.light.dir = CAMERA.pos;

        // Rotate stars
        std::vector<Mesh> &stars = solarSystem.stars;
        stars[0].rotateRuntime(Vec3f(), M_PI_2 * FPS.dTimeSec / 190, 1);
        stars[1].rotateRuntime(Vec3f(), M_PI_2 * FPS.dTimeSec / 210, 1);
        stars[2].rotateRuntime(Vec3f(), M_PI_2 * FPS.dTimeSec / 230, 1);
        stars[3].rotateRuntime(Vec3f(), M_PI_2 * FPS.dTimeSec / 340, 1);

        stars[0].rotateRuntime(Vec3f(), M_PI_2 * FPS.dTimeSec / 990, 0);
        stars[1].rotateRuntime(Vec3f(), M_PI_2 * FPS.dTimeSec / 810, 0);
        stars[2].rotateRuntime(Vec3f(), M_PI_2 * FPS.dTimeSec / 1030, 0);
        stars[3].rotateRuntime(Vec3f(), M_PI_2 * FPS.dTimeSec / 740, 0);

        // ========== Render Pipeline ==========

        // Vertex Shader
        VertexShader::cameraProjection();
        VertexShader::frustumCulling();
        VertexShader::createDepthMap();
        VertexShader::rasterization();

        // Fragment Shader (bunch of beta features)
        if (textureMode) FragmentShader::applyTexture();
        if (shadowMode) {
            FragmentShader::resetShadowMap();
            FragmentShader::createShadowMap();
            FragmentShader::applyShadowMap();
        }
        if (shadeMode) FragmentShader::phongShading();
        if (customMode) FragmentShader::customShader();

        // From buffer to SFMLtexture
        SFTex.updateTexture(
            GRAPHIC.buffer.color.x,
            GRAPHIC.buffer.color.y,
            GRAPHIC.buffer.color.z,
            GRAPHIC.buffer.color.w,
            GRAPHIC.buffer.width,
            GRAPHIC.buffer.height,
            GRAPHIC.pixelSize
        );

        // ========== Log handling ==========

        // Rainbow title
        double step = 120 * FPS.dTimeSec;
        if (cycle == 0) {
            rainbow.y += step; rainbow.x -= step;
            if (rainbow.y >= 255) cycle = 1;
        } else if (cycle == 1) {
            rainbow.z += step; rainbow.y -= step;
            if (rainbow.z >= 255) cycle = 2;
        } else if (cycle == 2) {
            rainbow.x += step; rainbow.z -= step;
            if (rainbow.x >= 255) cycle = 0;
        }
        sf::Color rainbowColor = sf::Color(rainbow.x, rainbow.y, rainbow.z);

        // Dynamic FPS color
        double gRatio = double(FPS.fps - 10) / 50;
        gRatio = std::max(0.0, std::min(gRatio, 1.0));
        sf::Color fpsColor((1 - gRatio) * 255, gRatio * 255, 0);

        // Log all the data
        LOG.addLog("Welcome to AsczEngine 3.0", rainbowColor, 1);
        LOG.addLog("FPS: " + std::to_string(FPS.fps), fpsColor);
        LOG.addLog(
            "Screen:\n| Res: " + std::to_string(width) +
            " x " + std::to_string(height) + "\n" +
            "| Pixel Size: " + std::to_string(pixelSize) + "\n" +
            "| RTFace1: " + std::to_string(GRAPHIC.rtCount1) + " / "
            + std::to_string(GRAPHIC.mesh.faces.size / 3) + "\n" +
            "| RTFace2: " + std::to_string(GRAPHIC.rtCount2) + " / "
            + std::to_string(GRAPHIC.mesh.faces.size / 3) + "\n" +
            "| RTFace3: " + std::to_string(GRAPHIC.rtCount3) + " / "
            + std::to_string(GRAPHIC.mesh.faces.size / 3) + "\n" +
            "| RTFace4: " + std::to_string(GRAPHIC.rtCount4) + " / "
            + std::to_string(GRAPHIC.mesh.faces.size / 3),
            sf::Color(255, 160, 160)
        );
        LOG.addLog(CAMERA.data(), sf::Color(160, 255, 160));
        LOG.addLog(GRAPHIC.light.data(), sf::Color(160, 160, 255));
        LOG.addLog("Shader (BETA)", sf::Color(255, 255, 255), 1);
        LOG.addLog(
            "| Texture: " + std::to_string(textureMode),
            sf::Color(textureMode ? 255 : 100, 50, 50)
        );
        LOG.addLog(
            "| Shadow: " + std::to_string(shadowMode),
            sf::Color(50, shadowMode ? 255 : 100, 50)
        );
        LOG.addLog(
            "| Shade: " + std::to_string(shadeMode),
            sf::Color(50, 50, shadeMode ? 255 : 100)
        );
        LOG.addLog(
            "| Custom: " + std::to_string(customMode),
            sf::Color(customMode ? 255 : 100, 50, customMode ? 255 : 100)
        );

        LOG.addLog(
            "vx: " + std::to_string(CAMERA.vel.x) +
            " vy: " + std::to_string(CAMERA.vel.y) +
            " vz: " + std::to_string(CAMERA.vel.z),
            sf::Color(255, 255, 255)
        );

        // Displays
        window.clear(sf::Color(0, 0, 0));
        window.draw(SFTex.sprite);
        LOG.drawLog(window);
        window.display();

        // Frame end
        FPS.endFrame();
    }

    // Clean up
    GRAPHIC.free();
    SFTex.free();

    return 0;
}
#include "hip/hip_runtime.h"
#include <Buffer3D.cuh>

Buffer3D::Buffer3D() {}

void Buffer3D::resize(int width, int height, int pixelSize) {
    this->width = width / pixelSize;
    this->height = height / pixelSize;
    size = width * height;
    blockNum = (size + blockSize - 1) / blockSize;

    free(); // Free the previous buffer

    hipMalloc(&active, size * sizeof(bool));
    hipMalloc(&depth, size * sizeof(float));
    hipMalloc(&color, size * sizeof(Vec4f));

    hipMalloc(&world, size * sizeof(Vec3f));
    hipMalloc(&normal, size * sizeof(Vec3f));
    hipMalloc(&texture, size * sizeof(Vec2f));
    hipMalloc(&wMeshId, size * sizeof(UInt));
    hipMalloc(&nMeshId, size * sizeof(UInt));
    hipMalloc(&tMeshId, size * sizeof(UInt));

    hipMalloc(&faceID, size * sizeof(ULLInt));
    hipMalloc(&bary, size * sizeof(Vec3f));
}

void Buffer3D::free() {
    if (active) hipFree(active);
    if (depth) hipFree(depth);
    if (color) hipFree(color);
    if (world) hipFree(world);
    if (normal) hipFree(normal);
    if (texture) hipFree(texture);
    if (wMeshId) hipFree(wMeshId);
    if (nMeshId) hipFree(nMeshId);
    if (tMeshId) hipFree(tMeshId);
    if (faceID) hipFree(faceID);
    if (bary) hipFree(bary);
}

void Buffer3D::clearBuffer() {
    clearBufferKernel<<<blockNum, blockSize>>>(
        active, depth, color,
        world, normal, texture,
        wMeshId, nMeshId, tMeshId,
        faceID, bary, size
    );
    hipDeviceSynchronize();
}

// Kernel for clearing the buffer
__global__ void clearBufferKernel(
    bool *active, float *depth, Vec4f *color,
    Vec3f *world, Vec3f *normal, Vec2f *texture,
    UInt *wMeshId, UInt *nMeshId, UInt *tMeshId,
    ULLInt *faceID, Vec3f *bary, int size
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;

    active[i] = false; // Inactive
    depth[i] = 1; // Furthest depth
    color[i] = Vec4f(); // Black

    world[i] = Vec3f(); // Limbo
    normal[i] = Vec3f(); // Limbo
    texture[i] = Vec2f(); // Limbo
    wMeshId[i] = NULL; // No mesh
    nMeshId[i] = NULL; // No mesh
    tMeshId[i] = NULL; // No mesh

    faceID[i] = NULL; // No face
    bary[i] = Vec3f(); // Limbo
}

// Night sky
void Buffer3D::nightSky() {
    nightSkyKernel<<<blockNum, blockSize>>>(color, width, height);
    hipDeviceSynchronize();
}

__global__ void nightSkyKernel(Vec4f *color, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= width * height) return;

    int x = i % width;
    int y = i / width;

    float ratioX = float(x) / float(width);
    float ratioY = float(y) / float(height);

    color[i] = Vec4f(0, 0, 0, 255);
    color[i].x = 4 * (1 - ratioY);
    color[i].y = 10 * (1 - ratioX);
    color[i].z = 20 * (1 - ratioY);
}
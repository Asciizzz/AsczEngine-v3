#include <Buffer3D.cuh>

Buffer3D::Buffer3D(int width, int height) {
    hipMalloc(&depth, width * height * sizeof(float));
    hipMalloc(&normal, width * height * 3 * sizeof(float));
    hipMalloc(&color, width * height * 3 * sizeof(float));
    hipMalloc(&world, width * height * 3 * sizeof(float));
}
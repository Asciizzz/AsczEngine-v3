#include <Buffer3D.cuh>

Buffer3D::Buffer3D() {}

void Buffer3D::resize(int width, int height, int pixelSize) {
    buffWidth = width / pixelSize;
    buffHeight = height / pixelSize;
    buffSize = buffWidth * buffHeight;

    free();

    hipMalloc(&depth, buffSize * sizeof(float));
    hipMalloc(&color, buffSize * sizeof(Vec3f));
    hipMalloc(&normal, buffSize * sizeof(Vec3f));
    hipMalloc(&world, buffSize * sizeof(Vec3f));
    hipMalloc(&tex, buffSize * sizeof(Vec2f));
}

void Buffer3D::free() {
    if (depth) hipFree(depth);
    if (color) hipFree(color);
    if (normal) hipFree(normal);
    if (world) hipFree(world);
    if (tex) hipFree(tex);
}
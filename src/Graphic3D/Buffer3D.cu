#include "hip/hip_runtime.h"
#include <Buffer3D.cuh>

Buffer3D::Buffer3D() {}
Buffer3D::~Buffer3D() { free(); }

void Buffer3D::resize(int width, int height, int pixelSize) {
    this->width = width / pixelSize;
    this->height = height / pixelSize;
    size = width * height;
    blockCount = (size + blockSize - 1) / blockSize;

    free(); // Free the previous buffer

    // For depth checking
    hipMalloc(&depth, size * sizeof(float));
    // For lighting
    hipMalloc(&color, size * sizeof(Vec4f));
    hipMalloc(&world, size * sizeof(Vec3f));
    hipMalloc(&normal, size * sizeof(Vec3f));
    // For texture mapping
    hipMalloc(&texture, size * sizeof(Vec2f));
    hipMalloc(&meshID, size * sizeof(UInt));
}

void Buffer3D::free() {
    if (depth) hipFree(depth);
    if (color) hipFree(color);
    if (world) hipFree(world);
    if (normal) hipFree(normal);
    if (texture) hipFree(texture);
    if (meshID) hipFree(meshID);
}

void Buffer3D::clearBuffer() {
    clearBufferKernel<<<blockCount, blockSize>>>(
        depth, color, world, normal, texture, meshID, size
    );
    hipDeviceSynchronize();
}

// Kernel for clearing the buffer
__global__ void clearBufferKernel(
    float *depth,
    Vec4f *color,
    Vec3f *world,
    Vec3f *normal,
    Vec2f *texture,
    UInt *meshID,
    int size
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;

    depth[i] = 1;
    color[i] = Vec4f(0, 0, 0, 0);
    world[i] = Vec3f(0, 0, 0);
    normal[i] = Vec3f(0, 0, 0);
    texture[i] = Vec2f(0, 0);
    meshID[i] = NULL;
}
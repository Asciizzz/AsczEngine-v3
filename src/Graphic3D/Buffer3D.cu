#include "hip/hip_runtime.h"
#include <Buffer3D.cuh>

Buffer3D::Buffer3D() {}
Buffer3D::~Buffer3D() { free(); }

void Buffer3D::resize(int width, int height, int pixelSize) {
    this->width = width / pixelSize;
    this->height = height / pixelSize;
    size = width * height;
    blockCount = (size + blockSize - 1) / blockSize;

    free(); // Free the previous buffer

    // For depth checking
    hipMalloc(&depth, size * sizeof(float));
    // For lighting
    hipMalloc(&color, size * sizeof(Vec4f));
    hipMalloc(&world, size * sizeof(Vec3f));
    hipMalloc(&normal, size * sizeof(Vec3f));
    // For texture mapping
    hipMalloc(&texture, size * sizeof(Vec2f));
    hipMalloc(&meshID, size * sizeof(UInt));
    // Other
    hipMalloc(&faceID, size * sizeof(ULLInt));
    hipMalloc(&bary, size * sizeof(Vec3f));
}

void Buffer3D::free() {
    if (depth) hipFree(depth);
    if (color) hipFree(color);
    if (world) hipFree(world);
    if (normal) hipFree(normal);
    if (texture) hipFree(texture);
    if (meshID) hipFree(meshID);
    if (faceID) hipFree(faceID);
    if (bary) hipFree(bary);
}

void Buffer3D::clearBuffer() {
    clearBufferKernel<<<blockCount, blockSize>>>(
        depth, color, world, normal, texture, meshID, faceID, bary, size
    );
    hipDeviceSynchronize();
}

// Kernel for clearing the buffer
__global__ void clearBufferKernel(
    float *depth,
    Vec4f *color,
    Vec3f *world,
    Vec3f *normal,
    Vec2f *texture,
    UInt *meshID,
    ULLInt *faceID,
    Vec3f *bary,
    int size
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;

    depth[i] = 1; // Furthest depth
    color[i] = Vec4f(); // Black
    world[i] = Vec3f(); // Limbo
    normal[i] = Vec3f(); // Limbo
    texture[i] = Vec2f(); // Limbo
    meshID[i] = NULL; // No mesh
    faceID[i] = NULL; // No face
    bary[i] = Vec3f(); // Limbo
}
#include <Buffer3D.cuh>

Buffer3D::Buffer3D() {}

void Buffer3D::resize(int width, int height, int pixelSize) {
    buffWidth = width / pixelSize;
    buffHeight = height / pixelSize;
    buffSize = buffWidth * buffHeight;
    free(); // Free the previous buffer

    // For depth checking
    hipMalloc(&depth, buffSize * sizeof(float));
    // For lighting
    hipMalloc(&color, buffSize * sizeof(Vec4f));
    hipMalloc(&world, buffSize * sizeof(Vec3f));
    hipMalloc(&normal, buffSize * sizeof(Vec3f));
    // For texture mapping
    hipMalloc(&texture, buffSize * sizeof(Vec2f));
    hipMalloc(&meshID, buffSize * sizeof(UInt));
}

void Buffer3D::free() {
    if (depth) hipFree(depth);
    if (color) hipFree(color);
    if (world) hipFree(world);
    if (normal) hipFree(normal);
    if (texture) hipFree(texture);
    if (meshID) hipFree(meshID);
}
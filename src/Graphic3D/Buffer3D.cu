#include "hip/hip_runtime.h"
#include <Buffer3D.cuh>

Buffer3D::Buffer3D() {}

void Buffer3D::resize(int width, int height, int pixelSize) {
    buffWidth = width / pixelSize;
    buffHeight = height / pixelSize;
    buffSize = buffWidth * buffHeight;
    blockCount = (buffSize + blockSize - 1) / blockSize;

    free(); // Free the previous buffer

    // For depth checking
    hipMalloc(&depth, buffSize * sizeof(float));
    // For lighting
    hipMalloc(&color, buffSize * sizeof(Vec4f));
    hipMalloc(&world, buffSize * sizeof(Vec3f));
    hipMalloc(&normal, buffSize * sizeof(Vec3f));
    // For texture mapping
    hipMalloc(&texture, buffSize * sizeof(Vec2f));
    hipMalloc(&meshID, buffSize * sizeof(UInt));
}

void Buffer3D::free() {
    if (depth) hipFree(depth);
    if (color) hipFree(color);
    if (world) hipFree(world);
    if (normal) hipFree(normal);
    if (texture) hipFree(texture);
    if (meshID) hipFree(meshID);
}

void Buffer3D::clearBuffer() {
    clearBufferKernel<<<blockCount, blockSize>>>(
        depth, color, world, normal, texture, meshID, buffSize
    );
    hipDeviceSynchronize();
}

// Kernel for clearing the buffer
__global__ void clearBufferKernel(
    float *depth,
    Vec4f *color,
    Vec3f *world,
    Vec3f *normal,
    Vec2f *texture,
    UInt *meshID,
    int buffSize
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= buffSize) return;

    depth[i] = INFINITY;
    color[i] = Vec4f(0, 0, 0, 0);
    world[i] = Vec3f(0, 0, 0);
    normal[i] = Vec3f(0, 0, 0);
    texture[i] = Vec2f(0, 0);
    meshID[i] = NULL;
}
#include "hip/hip_runtime.h"
#include <FragmentShader.cuh>

// ======================== Static functions ========================

void FragmentShader::applyTexture() { // Beta
    Graphic3D &grphic = Graphic3D::instance();
    Buffer3D &buffer = grphic.buffer;

    applyTextureKernel<<<buffer.blockNum, buffer.blockSize>>>(
        buffer.active,
        buffer.texture.x, buffer.texture.y,
        buffer.color.x, buffer.color.y, buffer.color.z, buffer.color.w,
        buffer.width, buffer.height,

        grphic.d_texture, grphic.textureWidth, grphic.textureHeight
    );
    hipDeviceSynchronize();
}

void FragmentShader::phongShading() {
    Graphic3D &grphic = Graphic3D::instance();
    Buffer3D &buffer = grphic.buffer;

    phongShadingKernel<<<buffer.blockNum, buffer.blockSize>>>(
        buffer.active,
        buffer.world.x, buffer.world.y, buffer.world.z,
        buffer.texture.x, buffer.texture.y,
        buffer.normal.x, buffer.normal.y, buffer.normal.z,
        buffer.color.x, buffer.color.y, buffer.color.z, buffer.color.w,
        buffer.width, buffer.height,

        grphic.light
    );
    hipDeviceSynchronize();
}

void FragmentShader::resetShadowMap() {
    Graphic3D &grphic = Graphic3D::instance();

    int blockNum = (grphic.shdwWidth * grphic.shdwHeight + 255) / 256;
    resetShadowMapKernel<<<blockNum, 256>>>(
        grphic.shadowDepth, grphic.shdwWidth, grphic.shdwHeight
    );
    hipDeviceSynchronize();
}

void FragmentShader::createShadowMap() {
    Graphic3D &grphic = Graphic3D::instance();
    Mesh3D &mesh = grphic.mesh;

    dim3 blockSize(8, 32);

    size_t blockNumTile = (grphic.shdwTileNum + blockSize.x - 1) / blockSize.x;
    size_t blockNumFace = (mesh.faces.size / 3 + blockSize.y - 1) / blockSize.y;
    dim3 blockNum(blockNumTile, blockNumFace);

    createShadowMapKernel<<<blockNum, blockSize>>>(
        mesh.world.x, mesh.world.y, mesh.world.z,
        mesh.faces.v, mesh.faces.size / 3,
        grphic.shadowDepth, grphic.shdwWidth, grphic.shdwHeight,
        grphic.shdwTileNumX, grphic.shdwTileNumY, grphic.shdwTileSizeX, grphic.shdwTileSizeY
    );
    hipDeviceSynchronize();
}

void FragmentShader::applyShadowMap() {
    Graphic3D &grphic = Graphic3D::instance();
    Buffer3D &buffer = grphic.buffer;

    applyShadowMapKernel<<<buffer.blockNum, buffer.blockSize>>>(
        buffer.active,
        buffer.world.x, buffer.world.y, buffer.world.z,
        buffer.normal.x, buffer.normal.y, buffer.normal.z,
        buffer.color.x, buffer.color.y, buffer.color.z, buffer.color.w,
        buffer.width, buffer.height,

        grphic.shadowDepth, grphic.shdwWidth, grphic.shdwHeight
    );
    hipDeviceSynchronize();
}

void FragmentShader::customShader() {
    Graphic3D &grphic = Graphic3D::instance();
    Buffer3D &buffer = grphic.buffer;

    customShaderKernel<<<buffer.blockNum, buffer.blockSize>>>(
        buffer.active, buffer.faceID, buffer.depth,
        buffer.bary.x, buffer.bary.y, buffer.bary.z,
        buffer.world.x, buffer.world.y, buffer.world.z,
        buffer.texture.x, buffer.texture.y,
        buffer.normal.x, buffer.normal.y, buffer.normal.z,
        buffer.color.x, buffer.color.y, buffer.color.z, buffer.color.w,
        buffer.width, buffer.height
    );
    hipDeviceSynchronize();
}

// ======================== Kernels ========================

__global__ void applyTextureKernel( // Beta
    bool *buffActive, float *buffTu, float *buffTv,
    float *buffCr, float *buffCg, float *buffCb, float *buffCa,
    int buffWidth, int buffHeight,
    Vec3f *texture, int textureWidth, int textureHeight
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= buffWidth * buffHeight || !buffActive[i]) return;

    int x = buffTu[i] * textureWidth;
    int y = buffTv[i] * textureHeight;
    int tIdx = x + y * textureWidth;

    if (tIdx >= textureWidth * textureHeight ||
        tIdx < 0) return;

    buffCr[i] = texture[tIdx].x;
    buffCg[i] = texture[tIdx].y;
    buffCb[i] = texture[tIdx].z;
}

__global__ void phongShadingKernel(
    bool *buffActive,
    float *buffWx, float *buffWy, float *buffWz,
    float *buffTu, float *buffTv,
    float *buffNx, float *buffNy, float *buffNz,
    float *buffCr, float *buffCg, float *buffCb, float *buffCa,
    int buffWidth, int buffHeight,

    LightSrc light
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= buffWidth * buffHeight || !buffActive[i]) return;

    buffCr[i] *= light.color.x;
    buffCg[i] *= light.color.y;
    buffCb[i] *= light.color.z;

    // Find the light direction
    // Vec3f lightDir = light.dir * -1;
    Vec3f lightDir = light.dir - Vec3f(buffWx[i], buffWy[i], buffWz[i]);
    Vec3f n = Vec3f(buffNx[i], buffNy[i], buffNz[i]);

    // Calculate the cosine of the angle between the normal and the light direction
    float dot = n * lightDir;
    
    float cosA = dot / (n.mag() * lightDir.mag());
    if (cosA < 0) cosA = -cosA;

    float diff = light.ambient * (1 - cosA) + light.specular * cosA;

    // Apply the light
    buffCr[i] *= diff;
    buffCg[i] *= diff;
    buffCb[i] *= diff;

    // Limit the color
    buffCr[i] = fminf(fmaxf(buffCr[i], 0), 255);
    buffCg[i] = fminf(fmaxf(buffCg[i], 0), 255);
    buffCb[i] = fminf(fmaxf(buffCb[i], 0), 255);
}

__global__ void resetShadowMapKernel(
    float *shadowDepth, int shdwWidth, int shdwHeight
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= shdwWidth * shdwHeight) return;

    shadowDepth[i] = 100000;
}

__global__ void createShadowMapKernel(
    const float *worldX, const float *worldY, const float *worldZ,
    const ULLInt *faceWs, ULLInt numFs,
    float *shadowDepth, int shdwWidth, int shdwHeight,
    int shdwTileNumX, int shdwTileNumY, int shdwTileSizeX, int shdwTileSizeY
) {
    ULLInt tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    ULLInt fIdx = blockIdx.y * blockDim.y + threadIdx.y;
    if (tIdx >= shdwTileNumX * shdwTileNumY || fIdx >= numFs) return;

    ULLInt idx0 = fIdx * 3;
    ULLInt idx1 = fIdx * 3 + 1;
    ULLInt idx2 = fIdx * 3 + 2;

    ULLInt fw0 = faceWs[idx0];
    ULLInt fw1 = faceWs[idx1];
    ULLInt fw2 = faceWs[idx2];

    float sx0 = (worldX[fw0] / 20 + 1) * shdwWidth / 2;
    float sx1 = (worldX[fw1] / 20 + 1) * shdwWidth / 2;
    float sx2 = (worldX[fw2] / 20 + 1) * shdwWidth / 2;

    float sy0 = (worldY[fw0] / 20 + 1) * shdwHeight / 2;
    float sy1 = (worldY[fw1] / 20 + 1) * shdwHeight / 2;
    float sy2 = (worldY[fw2] / 20 + 1) * shdwHeight / 2;

    float sz0 = worldZ[fw0];
    float sz1 = worldZ[fw1];
    float sz2 = worldZ[fw2];

    int tX = tIdx % shdwTileNumX;
    int tY = tIdx / shdwTileNumX;

    int shdwMinX = tX * shdwTileSizeX;
    int shdwMaxX = shdwMinX + shdwTileSizeX;
    int shdwMinY = tY * shdwTileSizeY;
    int shdwMaxY = shdwMinY + shdwTileSizeY;

    // Bound the shadow map
    int minX = min(min(sx0, sx1), sx2);
    int maxX = max(max(sx0, sx1), sx2);
    int minY = min(fmin(sy0, sy1), sy2);
    int maxY = max(max(sy0, sy1), sy2);

    // If bounding box is outside the tile, return
    if (minX > shdwMaxX || maxX < shdwMinX || minY > shdwMaxY || maxY < shdwMinY) return;

    // Clip based on the tile
    minX = max(minX, shdwMinX);
    maxX = min(maxX, shdwMaxX);
    minY = max(minY, shdwMinY);
    maxY = min(maxY, shdwMaxY);

    for (int x = minX; x <= maxX; x++)
    for (int y = minY; y <= maxY; y++) {
        int sIdx = x + y * shdwWidth;

        Vec3f bary = Vec3f::bary(
            Vec2f(x, y), Vec2f(sx0, sy0), Vec2f(sx1, sy1), Vec2f(sx2, sy2)
        );
        // Out of bound => Ignore
        if (bary.x < 0 || bary.y < 0 || bary.z < 0) continue;

        float zDepth = bary.x * sz0 + bary.y * sz1 + bary.z * sz2;

        if (atomicMinFloat(&shadowDepth[sIdx], zDepth)) {
            shadowDepth[sIdx] = zDepth;
        }
    }
}

__global__ void applyShadowMapKernel(
    bool *buffActive,
    float *buffWx, float *buffWy, float *buffWz,
    float *buffNx, float *buffNy, float *buffNz,
    float *buffCr, float *buffCg, float *buffCb, float *buffCa,
    int buffWidth, int buffHeight,

    float *shadowDepth, int shdwWidth, int shdwHeight
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= buffWidth * buffHeight || !buffActive[i]) return;

    float sfx = (buffWx[i] / 20 + 1) * shdwWidth / 2;
    float sfy = (buffWy[i] / 20 + 1) * shdwHeight / 2;
    float sfz = buffWz[i];

    // Create slight offset based on the normal direction with the light
    sfx += buffNx[i] * 0.8;
    sfy += buffNy[i] * 0.8;
    sfz += buffNz[i] * 0.8;

    int sx = int(sfx);
    int sy = int(sfy);

    if (sx < 0 || sx >= shdwWidth ||
        sy < 0 || sy >= shdwHeight) return;

    // Get the index of the shadow map
    int sIdx = sx + sy * shdwWidth;

    // If the fragment is closer than the shadow map, ignore
    if (sfz <= shadowDepth[sIdx] + 0.0001) return;

    // Apply the shadow
    buffCr[i] *= 0.15;
    buffCg[i] *= 0.15;
    buffCb[i] *= 0.15;
}

__global__ void customShaderKernel(
    bool *buffActive, ULLInt *buffFaceId, float *buffDepth,
    float *buffBrx, float *buffBry, float *buffBrz, // Bary
    float *buffWx, float *buffWy, float *buffWz, // World
    float *buffTu, float *buffTv, // Texture
    float *buffNx, float *buffNy, float *buffNz, // Normal
    float *buffCr, float *buffCg, float *buffCb, float *buffCa, // Color
    int buffWidth, int buffHeight
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= buffWidth * buffHeight || !buffActive[i]) return;

    ULLInt fIdx = buffFaceId[i];

    // If fId even, red, else blue
    float red = fIdx % 2 == 0 ? 255 : 100;
    float blue = fIdx % 2 == 0 ? 100 : 255;

    buffCr[i] = red;
    buffCg[i] = 100;
    buffCb[i] = blue;
    buffCa[i] = 255;
}
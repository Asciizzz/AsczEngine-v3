#include "hip/hip_runtime.h"
#include <Mesh3D.cuh>
#include <Graphic3D.cuh> // For the Graphic.mesh object

// ======================= Mesh object =======================

Mesh::Mesh() {}
Mesh::Mesh(
    VectF wx, VectF wy, VectF wz,
    VectF tu, VectF tv,
    VectF nx, VectF ny, VectF nz,
    VectF cr, VectF cg, VectF cb, VectF ca,
    VectULLI fw, VectLLI ft, VectLLI fn, VectLLI fm,
    VectF kdr, VectF kdg, VectF kdb
) : wx(wx), wy(wy), wz(wz),
    tu(tu), tv(tv),
    nx(nx), ny(ny), nz(nz), 
    cr(cr), cg(cg), cb(cb), ca(ca),

    fw(fw), ft(ft), fn(fn), fm(fm),

    kdr(kdr), kdg(kdg), kdb(kdb)
{}

void Mesh::push(Mesh &mesh) {
    // Vertex data
    for (ULLInt i = 0; i < mesh.wx.size(); i++) {
        wx.push_back(mesh.wx[i]);
        wy.push_back(mesh.wy[i]);
        wz.push_back(mesh.wz[i]);
    }
    for (ULLInt i = 0; i < mesh.tu.size(); i++) {
        tu.push_back(mesh.tu[i]);
        tv.push_back(mesh.tv[i]);
    }
    for (ULLInt i = 0; i < mesh.nx.size(); i++) {
        nx.push_back(mesh.nx[i]);
        ny.push_back(mesh.ny[i]);
        nz.push_back(mesh.nz[i]);
    }

    for (ULLInt i = 0; i < mesh.cr.size(); i++) {
        cr.push_back(mesh.cr[i]);
        cg.push_back(mesh.cg[i]);
        cb.push_back(mesh.cb[i]);
        ca.push_back(mesh.ca[i]);
    }

    // Increment face indices
    for (ULLInt i = 0; i < mesh.fw.size(); i++) {
        fw.push_back(mesh.fw[i] + wx.size());

        if (mesh.ft[i] < 0) ft.push_back(-1);
        else ft.push_back(mesh.ft[i] + tu.size());

        fn.push_back(mesh.fn[i] + nx.size());
        fm.push_back(mesh.fm[i]);
    }

    // Material data
    for (ULLInt i = 0; i < mesh.kdr.size(); i++) {
        kdr.push_back(mesh.kdr[i]);
        kdg.push_back(mesh.kdg[i]);
        kdb.push_back(mesh.kdb[i]);
    }
}

Vec3f Mesh::w3f(ULLInt i) { return Vec3f(wx[i], wy[i], wz[i]); }
Vec2f Mesh::t2f(ULLInt i) { return Vec2f(tu[i], tv[i]); }
Vec3f Mesh::n3f(ULLInt i) { return Vec3f(nx[i], ny[i], nz[i]); }
Vec4f Mesh::c4f(ULLInt i) { return Vec4f(cr[i], cg[i], cb[i], ca[i]); }

void Mesh::translateIni(Vec3f t) {
    #pragma omp parallel for
    for (ULLInt i = 0; i < wx.size(); i++) {
        wx[i] += t.x; wy[i] += t.y; wz[i] += t.z;
    }
}

void Mesh::rotateIni(Vec3f origin, float r, short axis) {
    #pragma omp parallel for
    for (ULLInt i = 0; i < wx.size(); i++) {
        Vec3f p = Vec3f(wx[i], wy[i], wz[i]);

        switch (axis) {
            case 0: p.rotateX(origin, r); break;
            case 1: p.rotateY(origin, r); break;
            case 2: p.rotateZ(origin, r); break;
        }

        wx[i] = p.x; wy[i] = p.y; wz[i] = p.z;
    }

    #pragma omp parallel for
    for (ULLInt i = 0; i < nx.size(); i++) {
        Vec3f n = Vec3f(nx[i], ny[i], nz[i]);
        
        switch (axis) {
            case 0: n.rotateX(Vec3f(), r); break;
            case 1: n.rotateY(Vec3f(), r); break;
            case 2: n.rotateZ(Vec3f(), r); break;
        }

        n /= n.mag(); // Normalize

        nx[i] = n.x; ny[i] = n.y; nz[i] = n.z;
    }
}

void Mesh::scaleIni(Vec3f origin, Vec3f scl, bool sclNormal) {
    #pragma omp parallel for
    for (ULLInt i = 0; i < wx.size(); i++) {
        Vec3f p = Vec3f(wx[i], wy[i], wz[i]);
        p.scale(origin, scl);

        wx[i] = p.x; wy[i] = p.y; wz[i] = p.z;
    }

    if (!sclNormal) return;
    #pragma omp parallel for
    for (ULLInt i = 0; i < nx.size(); i++) {
        Vec3f n = Vec3f(nx[i], ny[i], nz[i]);
        n.scale(Vec3f(), scl);

        n /= n.mag(); // Normalize

        nx[i] = n.x; ny[i] = n.y; nz[i] = n.z;
    }
}

void Mesh::translateRuntime(Vec3f t) {
    Vec3f_ptr &w = Graphic3D::instance().mesh.v.w;

    ULLInt start = w_range.x, end = w_range.y;

    ULLInt numWs = end - start;
    ULLInt gridSize = (numWs + 255) / 256;

    translateMeshKernel<<<gridSize, 256>>>(
        w.x + start, w.y + start, w.z + start,
        t.x, t.y, t.z, numWs
    );
    hipDeviceSynchronize();
}
void Mesh::rotateRuntime(Vec3f origin, float r, short axis) {
    Vec3f_ptr &w = Graphic3D::instance().mesh.v.w;
    Vec3f_ptr &n = Graphic3D::instance().mesh.v.n;

    ULLInt startW = w_range.x, endW = w_range.y;
    ULLInt startN = n_range.x, endN = n_range.y;

    ULLInt numWs = endW - startW;
    ULLInt numNs = endN - startN;

    ULLInt num = max(numWs, numNs);
    ULLInt gridSize = (num + 255) / 256;    

    rotateMeshKernel<<<gridSize, 256>>>(
        w.x + startW, w.y + startW, w.z + startW, numWs,
        n.x + startN, n.y + startN, n.z + startN, numNs,
        origin.x, origin.y, origin.z, r, axis
    );
    hipDeviceSynchronize();
}
void Mesh::scaleRuntime(Vec3f origin, Vec3f scl) {
    Vec3f_ptr &w = Graphic3D::instance().mesh.v.w;
    Vec3f_ptr &n = Graphic3D::instance().mesh.v.n;

    ULLInt startW = w_range.x, endW = w_range.y;
    ULLInt startN = n_range.x, endN = n_range.y;

    ULLInt numWs = endW - startW;
    ULLInt numNs = endN - startN;

    ULLInt num = max(numWs, numNs);
    ULLInt gridSize = (num + 255) / 256;

    scaleMeshKernel<<<gridSize, 256>>>(
        w.x + startW, w.y + startW, w.z + startW, numWs,
        n.x + startN, n.y + startN, n.z + startN, numNs,
        origin.x, origin.y, origin.z, scl.x, scl.y, scl.z
    );
    hipDeviceSynchronize();
}

// ======================= Vertex_ptr =======================


void Vertex_ptr::malloc(ULLInt size) {
    s.malloc(size);
    w.malloc(size);
    t.malloc(size);
    n.malloc(size);
    c.malloc(size);
}
void Vertex_ptr::free() {
    s.free();
    w.free();
    t.free();
    n.free();
    c.free();
}
void Vertex_ptr::operator+=(Vertex_ptr &vertex) {
    s += vertex.s;
    w += vertex.w;
    t += vertex.t;
    n += vertex.n;
    c += vertex.c;
}

// ======================= Face_ptr =======================


void Face_ptr::malloc(ULLInt size) {
    hipMalloc(&v, size * sizeof(ULLInt));
    hipMalloc(&t, size * sizeof(LLInt));
    hipMalloc(&n, size * sizeof(LLInt));
    hipMalloc(&m, size * sizeof(LLInt));
    this->size = size;
}
void Face_ptr::free() {
    if (v) hipFree(v);
    if (t) hipFree(t);
    if (n) hipFree(n);
    if (m) hipFree(m);
}
void Face_ptr::operator+=(Face_ptr &face) {
    ULLInt size = this->size + face.size;

    ULLInt *newV;
    LLInt *newT;
    LLInt *newN;
    LLInt *newM;
    hipMalloc(&newV, size * sizeof(ULLInt));
    hipMalloc(&newT, size * sizeof(LLInt));
    hipMalloc(&newN, size * sizeof(LLInt));
    hipMalloc(&newM, size * sizeof(LLInt));

    hipMemcpy(newV, v, this->size * sizeof(ULLInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newT, t, this->size * sizeof(LLInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newN, n, this->size * sizeof(LLInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newM, m, this->size * sizeof(LLInt), hipMemcpyDeviceToDevice);

    hipMemcpy(newV + this->size, face.v, face.size * sizeof(ULLInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newT + this->size, face.t, face.size * sizeof(LLInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newN + this->size, face.n, face.size * sizeof(LLInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newM + this->size, face.m, face.size * sizeof(LLInt), hipMemcpyDeviceToDevice);

    free();

    v = newV;
    t = newT;
    n = newN;
    m = newM;
    this->size = size;
}

// ======================= Material_ptr =======================


void Material_ptr::malloc(ULLInt size) {
    ka.malloc(size);
    kd.malloc(size);
    ks.malloc(size);
    hipMalloc(&map_Kd, size * sizeof(ULLInt));
    hipMalloc(&ns, size * sizeof(float));
}
void Material_ptr::free() {
    ka.free();
    kd.free();
    ks.free();
    if (map_Kd) hipFree(map_Kd);
    if (ns) hipFree(ns);
}
void Material_ptr::operator+=(Material_ptr &material) {
    ka += material.ka;
    kd += material.kd;
    ks += material.ks;

    ULLInt size = ka.size;

    ULLInt *newMap_Kd;
    float *newNs;
    hipMalloc(&newMap_Kd, size * sizeof(ULLInt));
    hipMalloc(&newNs, size * sizeof(float));

    hipMemcpy(newMap_Kd, map_Kd, size * sizeof(ULLInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newNs, ns, size * sizeof(float), hipMemcpyDeviceToDevice);

    hipMemcpy(newMap_Kd + size, material.map_Kd, material.ka.size * sizeof(ULLInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newNs + size, material.ns, material.ka.size * sizeof(float), hipMemcpyDeviceToDevice);

    free();

    map_Kd = newMap_Kd;
    ns = newNs;
}


// ======================= Mesh3D =======================

// Free
void Mesh3D::free() {
    v.free();
    f.free();

    ka.free();
    kd.free();
    ks.free();
    hipFree(map_Kd);
    hipFree(ns);
    hipFree(txtr);
    hipFree(tstart);
    hipFree(tsize);
}

// Push
void Mesh3D::push(Mesh &mesh) {
    ULLInt offsetV = v.w.size;
    ULLInt offsetT = v.t.size;
    ULLInt offsetN = v.n.size;
    ULLInt offsetKd = kd.size;

    // Set the range of stuff
    mesh.w_range = {offsetV, offsetV + mesh.wx.size()};
    mesh.t_range = {offsetT, offsetT + mesh.tu.size()};
    mesh.n_range = {offsetN, offsetN + mesh.nx.size()};
    mesh.c_range = {offsetV, offsetV + mesh.cr.size()};

    // Vertex data
    Vec3f_ptr newW;
    Vec2f_ptr newT;
    Vec3f_ptr newN;
    Vec4f_ptr newC;
    ULLInt wSize = mesh.wx.size();
    ULLInt tSize = mesh.tu.size();
    ULLInt nSize = mesh.nx.size();
    ULLInt cSize = mesh.cr.size();
    newW.malloc(wSize);
    newT.malloc(tSize);
    newN.malloc(nSize);
    newC.malloc(cSize);

    // Material data
    Vec3f_ptr newKd;
    ULLInt kdSize = mesh.kdr.size();
    newKd.malloc(kdSize);

    // Stream for async memory copy
    hipStream_t stream;
    hipStreamCreate(&stream);

    hipMemcpyAsync(newW.x, mesh.wx.data(), wSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newW.y, mesh.wy.data(), wSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newW.z, mesh.wz.data(), wSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newT.x, mesh.tu.data(), tSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newT.y, mesh.tv.data(), tSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newN.x, mesh.nx.data(), nSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newN.y, mesh.ny.data(), nSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newN.z, mesh.nz.data(), nSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newC.x, mesh.cr.data(), cSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newC.y, mesh.cg.data(), cSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newC.z, mesh.cb.data(), cSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newC.w, mesh.ca.data(), cSize * sizeof(float), hipMemcpyHostToDevice, stream);

    hipMemcpyAsync(newKd.x, mesh.kdr.data(), kdSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newKd.y, mesh.kdg.data(), kdSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newKd.z, mesh.kdb.data(), kdSize * sizeof(float), hipMemcpyHostToDevice, stream);

    v.w += newW;
    v.t += newT;
    v.n += newN;
    v.c += newC;
    v.s.free(); // Free the s buffer
    v.s.malloc(v.w.size);

    kd += newKd;

    // Face data

    Face_ptr newF;
    ULLInt fSize = mesh.fw.size();
    newF.malloc(fSize);

    hipMemcpyAsync(newF.v, mesh.fw.data(), fSize * sizeof(ULLInt), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newF.t, mesh.ft.data(), fSize * sizeof(LLInt), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newF.n, mesh.fn.data(), fSize * sizeof(LLInt), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newF.m, mesh.fm.data(), fSize * sizeof(LLInt), hipMemcpyHostToDevice, stream);

    // Increment face indices
    ULLInt gridSize = (fSize + 255) / 256;
    incFaceIdxKernel1<<<gridSize, 256>>>(newF.v, offsetV, fSize);
    incFaceIdxKernel2<<<gridSize, 256>>>(newF.t, offsetT, fSize);
    incFaceIdxKernel2<<<gridSize, 256>>>(newF.n, offsetN, fSize);
    incFaceIdxKernel2<<<gridSize, 256>>>(newF.m, offsetKd, fSize);
    f += newF;
}
void Mesh3D::push(std::vector<Mesh> &meshes) {
    for (Mesh &mesh : meshes) push(mesh);
}

// Kernel for incrementing face indices
__global__ void incFaceIdxKernel1(ULLInt *f, ULLInt offset, ULLInt numFs) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numFs) f[idx] += offset;
}
__global__ void incFaceIdxKernel2(LLInt *f, ULLInt offset, ULLInt numFs) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numFs && f[idx] >= 0) f[idx] += offset;
}

// Kernel for transforming vertices

__global__ void translateMeshKernel(
    float *wx, float *wy, float *wz,
    float tx, float ty, float tz,
    ULLInt numWs
) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWs) {
        wx[idx] += tx;
        wy[idx] += ty;
        wz[idx] += tz;
    }
}

__global__ void rotateMeshKernel(
    float *wx, float *wy, float *wz, ULLInt numWs,
    float *nx, float *ny, float *nz, ULLInt numNs,
    float ox, float oy, float oz,
    float r, short axis
) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numWs) {
        Vec3f p = Vec3f(wx[idx], wy[idx], wz[idx]);

        switch (axis) {
            case 0: p.rotateX(Vec3f(ox, oy, oz), r); break;
            case 1: p.rotateY(Vec3f(ox, oy, oz), r); break;
            case 2: p.rotateZ(Vec3f(ox, oy, oz), r); break;
        }

        wx[idx] = p.x;
        wy[idx] = p.y;
        wz[idx] = p.z;
    }

    if (idx < numNs) {
        Vec3f n = Vec3f(nx[idx], ny[idx], nz[idx]);

        switch (axis) {
            case 0: n.rotateX(Vec3f(), r); break;
            case 1: n.rotateY(Vec3f(), r); break;
            case 2: n.rotateZ(Vec3f(), r); break;
        }

        n /= n.mag(); // Normalize

        nx[idx] = n.x;
        ny[idx] = n.y;
        nz[idx] = n.z;
    }
}

__global__ void scaleMeshKernel(
    float *wx, float *wy, float *wz, ULLInt numWs,
    float *nx, float *ny, float *nz, ULLInt numNs,
    float ox, float oy, float oz,
    float sx, float sy, float sz
) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numWs) {
        wx[idx] = (wx[idx] - ox) * sx + ox;
        wy[idx] = (wy[idx] - oy) * sy + oy;
        wz[idx] = (wz[idx] - oz) * sz + oz;
    }

    if (idx < numNs) {
        nx[idx] *= sx;
        ny[idx] *= sy;
        nz[idx] *= sz;
        // Normalize
        float mag = sqrt(
            nx[idx] * nx[idx] +
            ny[idx] * ny[idx] +
            nz[idx] * nz[idx]
        );
        nx[idx] /= mag;
        ny[idx] /= mag;
        nz[idx] /= mag;
    }
}
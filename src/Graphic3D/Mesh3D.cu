#include "hip/hip_runtime.h"
#include <Mesh3D.cuh>

// Mesh struct

Mesh::Mesh(UInt id, Vecs3f &world, Vecs3f &normal, Vecs2f &texture, Vecs4f &color, Vecs3uli &faces) :
    world(world), normal(normal), texture(texture), color(color), mID(world.size(), id), faces(faces)
{}

Mesh::Mesh(Mesh &mesh) :
    world(mesh.world), normal(mesh.normal),
    texture(mesh.texture), color(mesh.color),
    mID(mesh.mID), faces(mesh.faces)
{}

Mesh Mesh::operator+=(Mesh &mesh) {
    ULLInt oldSize = world.size();
    world.insert(world.end(), mesh.world.begin(), mesh.world.end());
    normal.insert(normal.end(), mesh.normal.begin(), mesh.normal.end());
    texture.insert(texture.end(), mesh.texture.begin(), mesh.texture.end());
    color.insert(color.end(), mesh.color.begin(), mesh.color.end());
    mID.insert(mID.end(), mesh.mID.begin(), mesh.mID.end());
    faces.insert(faces.end(), mesh.faces.begin(), mesh.faces.end());

    // Shift the faces indices
    for (ULLInt i = oldSize; i < faces.size(); i++) {
        faces[i] += oldSize;
    }
    return *this;
}

// Constructor

Mesh3D::Mesh3D(ULLInt numVs, ULLInt numFs) :
    numVs(numVs), numFs(numFs)
{
    mallocVertices();
    mallocFaces();
}

Mesh3D::Mesh3D(UInt id, Vecs3f &world, Vecs3f &normal, Vecs2f &texture, Vecs4f &color, Vecs3uli &faces) :
    numVs(world.size()), numFs(faces.size())
{
    mallocVertices();
    mallocFaces();
    uploadData(id, world, normal, texture, color, faces);
}

Mesh3D::Mesh3D(Mesh &mesh) :
    numVs(mesh.world.size()), numFs(mesh.faces.size())
{
    mallocVertices();
    mallocFaces();
    uploadData(mesh.mID[0], mesh.world, mesh.normal, mesh.texture, mesh.color, mesh.faces);
}

Mesh3D::~Mesh3D() {
    freeVertices();
    freeFaces();
}

// Memory management

void Mesh3D::mallocVertices() {
    blockNumVs = (numVs + blockSize - 1) / blockSize;
    hipMalloc(&world, numVs * sizeof(Vec3f));
    hipMalloc(&normal, numVs * sizeof(Vec3f));
    hipMalloc(&texture, numVs * sizeof(Vec2f));
    hipMalloc(&color, numVs * sizeof(Vec4f));
    hipMalloc(&mID, numVs * sizeof(UInt));
}

void Mesh3D::resizeVertices(ULLInt numVs) {
    freeVertices();
    this->numVs = numVs;
    mallocVertices();
}

void Mesh3D::freeVertices() {
    hipFree(world);
    hipFree(normal);
    hipFree(texture);
    hipFree(color);
    hipFree(mID);
}

void Mesh3D::mallocFaces() {
    blockNumFs = (numFs + blockSize - 1) / blockSize;
    hipMalloc(&faces, numFs * sizeof(Vec3uli));
}

void Mesh3D::resizeFaces(ULLInt numFs) {
    freeFaces();
    this->numFs = numFs;
    mallocFaces();
}

void Mesh3D::freeFaces() {
    hipFree(faces);
}

// Upload host data to device

void Mesh3D::uploadData(UInt id, Vecs3f &world, Vecs3f &normal, Vecs2f &texture, Vecs4f &color, Vecs3uli &faces) {
    // Vertices
    setMeshIDKernel<<<blockNumVs, blockSize>>>(this->mID, numVs, id);
    hipMemcpy(this->world, world.data(), world.size() * sizeof(Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(this->normal, normal.data(), normal.size() * sizeof(Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(this->texture, texture.data(), texture.size() * sizeof(Vec2f), hipMemcpyHostToDevice);
    hipMemcpy(this->color, color.data(), color.size() * sizeof(Vec4f), hipMemcpyHostToDevice);
    // Faces indices
    hipMemcpy(this->faces, faces.data(), faces.size() * sizeof(Vec3uli), hipMemcpyHostToDevice);
}

// Mesh operators

void Mesh3D::operator+=(Mesh3D &mesh) {
    // Resize vertices
    ULLInt newNumVs = numVs + mesh.numVs;
    Vec3f *newWorld;
    Vec3f *newNormal;
    Vec2f *newTexture;
    Vec4f *newColor;
    UInt *newMID;
    hipMalloc(&newWorld, newNumVs * sizeof(Vec3f));
    hipMalloc(&newNormal, newNumVs * sizeof(Vec3f));
    hipMalloc(&newTexture, newNumVs * sizeof(Vec2f));
    hipMalloc(&newColor, newNumVs * sizeof(Vec4f));
    hipMalloc(&newMID, newNumVs * sizeof(UInt));
    // Copy old data
    hipMemcpy(newWorld, world, numVs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newNormal, normal, numVs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newTexture, texture, numVs * sizeof(Vec2f), hipMemcpyDeviceToDevice);
    hipMemcpy(newColor, color, numVs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    hipMemcpy(newMID, mID, numVs * sizeof(UInt), hipMemcpyDeviceToDevice);
    // Copy new data
    hipMemcpy(newWorld + numVs, mesh.world, mesh.numVs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newNormal + numVs, mesh.normal, mesh.numVs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newTexture + numVs, mesh.texture, mesh.numVs * sizeof(Vec2f), hipMemcpyDeviceToDevice);
    hipMemcpy(newColor + numVs, mesh.color, mesh.numVs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    hipMemcpy(newMID + numVs, mesh.mID, mesh.numVs * sizeof(UInt), hipMemcpyDeviceToDevice);
    // Free old data
    freeVertices();
    // Update vertices
    world = newWorld;
    normal = newNormal;
    texture = newTexture;
    color = newColor;
    mID = newMID;

    // Resize faces (with offset for the added vertices)
    ULLInt newNumFs = numFs + mesh.numFs;
    Vec3uli *newFaces;
    hipMalloc(&newFaces, newNumFs * sizeof(Vec3uli));
    hipMemcpy(newFaces, faces, numFs * sizeof(Vec3uli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaces + numFs, mesh.faces, mesh.numFs * sizeof(Vec3uli), hipMemcpyDeviceToDevice);
    incrementFaceIdxKernel<<<(mesh.numFs + 255) / 256, 256>>>(newFaces, mesh.numFs, numVs);
    hipFree(faces);
    faces = newFaces;

    // Update number of vertices and faces
    numVs = newNumVs;
    numFs = newNumFs;
    blockNumVs = (numVs + blockSize - 1) / blockSize;
    blockNumFs = (numFs + blockSize - 1) / blockSize;
}

// Transformations

void Mesh3D::translate(UInt meshID, Vec3f t) {
    translateVertexKernel<<<blockNumVs, blockSize>>>(world, mID, numVs, meshID, t);
}
void Mesh3D::rotate(UInt meshID, Vec3f origin, Vec3f rot) {
    rotateVertexKernel<<<blockNumVs, blockSize>>>(world, mID, numVs, meshID, origin, rot);
}
void Mesh3D::scale(UInt meshID, Vec3f origin, Vec3f scl) {
    scaleVertexKernel<<<blockNumVs, blockSize>>>(world, mID, numVs, meshID, origin, scl);
}

// DEBUG

void Mesh3D::printVertices(bool p_world, bool p_normal, bool p_tex, bool p_color, bool p_mID) {
    Vec3f *hworld = new Vec3f[numVs];
    Vec3f *hNormal = new Vec3f[numVs];
    Vec2f *hTexture = new Vec2f[numVs];
    Vec4f *hColor = new Vec4f[numVs];
    UInt *hMID = new UInt[numVs];

    hipMemcpy(hworld, world, numVs * sizeof(Vec3f), hipMemcpyDeviceToHost);
    hipMemcpy(hNormal, normal, numVs * sizeof(Vec3f), hipMemcpyDeviceToHost);
    hipMemcpy(hTexture, texture, numVs * sizeof(Vec2f), hipMemcpyDeviceToHost);
    hipMemcpy(hColor, color, numVs * sizeof(Vec4f), hipMemcpyDeviceToHost);
    hipMemcpy(hMID, mID, numVs * sizeof(UInt), hipMemcpyDeviceToHost);

    for (ULLInt i = 0; i < numVs; i++) {
        printf("Vertex %llu\n", i);
        if (p_world) printf("| world: (%f, %f, %f)\n", hworld[i].x, hworld[i].y, hworld[i].z);
        if (p_normal) printf("| Normal: (%f, %f, %f)\n", hNormal[i].x, hNormal[i].y, hNormal[i].z);
        if (p_tex) printf("| Tex: (%f, %f)\n", hTexture[i].x, hTexture[i].y);
        if (p_color) printf("| Color: (%f, %f, %f, %f)\n", hColor[i].x, hColor[i].y, hColor[i].z, hColor[i].w);
        if (p_mID) printf("| mID: %u\n", hMID[i]);
    }

    delete[] hworld, hNormal, hTexture, hColor, hMID;
}

void Mesh3D::printFaces() {
    Vec3uli *hFaces = new Vec3uli[numFs];
    hipMemcpy(hFaces, faces, numFs * sizeof(Vec3uli), hipMemcpyDeviceToHost);

    for (ULLInt i = 0; i < numFs; i++) {
        printf("Face %llu: (%lu, %lu, %lu)\n", i, hFaces[i].x, hFaces[i].y, hFaces[i].z);
    }

    delete[] hFaces;
}

// Kernel for preparing vertices
__global__ void incrementFaceIdxKernel(Vec3uli *faces, ULLInt numFs, ULLInt offset) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numFs) faces[idx] += offset;
}

__global__ void setMeshIDKernel(UInt *mID, ULLInt numVs, UInt id) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numVs) return;

    mID[idx] = id;
}

// Kernel for transforming vertices
__global__ void translateVertexKernel(Vec3f *world, UInt *mID, ULLInt numVs, UInt meshID, Vec3f t) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numVs || mID[idx] != meshID) return;

    world[idx].translate(t);
}
__global__ void rotateVertexKernel(Vec3f *world, UInt *mID, ULLInt numVs, UInt meshID, Vec3f origin, Vec3f rot) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numVs || mID[idx] != meshID) return;

    world[idx].rotate(origin, rot);
}
__global__ void scaleVertexKernel(Vec3f *world, UInt *mID, ULLInt numVs, UInt meshID, Vec3f origin, Vec3f scl) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numVs || mID[idx] != meshID) return;

    world[idx].scale(origin, scl);
}
#include "hip/hip_runtime.h"
#include <Mesh3D.cuh>
#include <Graphic3D.cuh> // For the Graphic.mesh object

// ======================= Mesh object =======================

Mesh::Mesh(
    std::vector<float> wx, std::vector<float> wy, std::vector<float> wz,
    std::vector<float> tu, std::vector<float> tv,
    std::vector<float> nx, std::vector<float> ny, std::vector<float> nz,
    std::vector<float> cr, std::vector<float> cg, std::vector<float> cb, std::vector<float> ca,
    std::vector<ULLInt> fw, std::vector<ULLInt> ft, std::vector<ULLInt> fn
) : wx(wx), wy(wy), wz(wz), tu(tu), tv(tv), nx(nx), ny(ny), nz(nz), 
    cr(cr), cg(cg), cb(cb), ca(ca), fw(fw), ft(ft), fn(fn) {}

Mesh::Mesh() {}

void Mesh::push(Mesh &mesh) {
    for (ULLInt i = 0; i < mesh.wx.size(); i++) {
        wx.push_back(mesh.wx[i]);
        wy.push_back(mesh.wy[i]);
        wz.push_back(mesh.wz[i]);
    }
    for (ULLInt i = 0; i < mesh.tu.size(); i++) {
        tu.push_back(mesh.tu[i]);
        tv.push_back(mesh.tv[i]);
    }
    for (ULLInt i = 0; i < mesh.nx.size(); i++) {
        nx.push_back(mesh.nx[i]);
        ny.push_back(mesh.ny[i]);
        nz.push_back(mesh.nz[i]);
    }

    for (ULLInt i = 0; i < mesh.cr.size(); i++) {
        cr.push_back(mesh.cr[i]);
        cg.push_back(mesh.cg[i]);
        cb.push_back(mesh.cb[i]);
        ca.push_back(mesh.ca[i]);
    }

    // Increment face indices
    for (ULLInt i = 0; i < mesh.fw.size(); i++) {
        fw.push_back(mesh.fw[i] + wx.size());
        ft.push_back(mesh.ft[i] + tu.size());
        fn.push_back(mesh.fn[i] + nx.size());
    }
}

Vec3f Mesh::w3f(ULLInt i) { return Vec3f(wx[i], wy[i], wz[i]); }
Vec2f Mesh::t2f(ULLInt i) { return Vec2f(tu[i], tv[i]); }
Vec3f Mesh::n3f(ULLInt i) { return Vec3f(nx[i], ny[i], nz[i]); }
Vec4f Mesh::c4f(ULLInt i) { return Vec4f(cr[i], cg[i], cb[i], ca[i]); }

void Mesh::translateIni(Vec3f t) {
    #pragma omp parallel for
    for (ULLInt i = 0; i < wx.size(); i++) {
        wx[i] += t.x; wy[i] += t.y; wz[i] += t.z;
    }
}

void Mesh::rotateIni(Vec3f origin, float r, short axis) {
    #pragma omp parallel for
    for (ULLInt i = 0; i < wx.size(); i++) {
        Vec3f p = Vec3f(wx[i], wy[i], wz[i]);

        switch (axis) {
            case 0: p.rotateX(origin, r); break;
            case 1: p.rotateY(origin, r); break;
            case 2: p.rotateZ(origin, r); break;
        }

        wx[i] = p.x; wy[i] = p.y; wz[i] = p.z;
    }

    #pragma omp parallel for
    for (ULLInt i = 0; i < nx.size(); i++) {
        Vec3f n = Vec3f(nx[i], ny[i], nz[i]);
        
        switch (axis) {
            case 0: n.rotateX(Vec3f(), r); break;
            case 1: n.rotateY(Vec3f(), r); break;
            case 2: n.rotateZ(Vec3f(), r); break;
        }

        n /= n.mag(); // Normalize

        nx[i] = n.x; ny[i] = n.y; nz[i] = n.z;
    }
}

void Mesh::scaleIni(Vec3f origin, Vec3f scl, bool sclNormal) {
    #pragma omp parallel for
    for (ULLInt i = 0; i < wx.size(); i++) {
        Vec3f p = Vec3f(wx[i], wy[i], wz[i]);
        p.scale(origin, scl);

        wx[i] = p.x; wy[i] = p.y; wz[i] = p.z;
    }

    if (!sclNormal) return;
    #pragma omp parallel for
    for (ULLInt i = 0; i < nx.size(); i++) {
        Vec3f n = Vec3f(nx[i], ny[i], nz[i]);
        n.scale(Vec3f(), scl);

        n /= n.mag(); // Normalize

        nx[i] = n.x; ny[i] = n.y; nz[i] = n.z;
    }
}

void Mesh::translateRuntime(Vec3f t) {
    Vec3f_ptr &w = Graphic3D::instance().mesh.w;

    ULLInt start = w_range.x, end = w_range.y;

    ULLInt numWs = end - start;
    ULLInt gridSize = (numWs + 255) / 256;

    translateMeshKernel<<<gridSize, 256>>>(
        w.x + start, w.y + start, w.z + start,
        t.x, t.y, t.z, numWs
    );
    hipDeviceSynchronize();
}
void Mesh::rotateRuntime(Vec3f origin, float r, short axis) {
    Vec3f_ptr &w = Graphic3D::instance().mesh.w;
    Vec3f_ptr &n = Graphic3D::instance().mesh.n;

    ULLInt startW = w_range.x, endW = w_range.y;
    ULLInt startN = n_range.x, endN = n_range.y;

    ULLInt numWs = endW - startW;
    ULLInt numNs = endN - startN;

    ULLInt num = max(numWs, numNs);
    ULLInt gridSize = (num + 255) / 256;    

    rotateMeshKernel<<<gridSize, 256>>>(
        w.x + startW, w.y + startW, w.z + startW, numWs,
        n.x + startN, n.y + startN, n.z + startN, numNs,
        origin.x, origin.y, origin.z, r, axis
    );
    hipDeviceSynchronize();
}
void Mesh::scaleRuntime(Vec3f origin, Vec3f scl) {
    Vec3f_ptr &w = Graphic3D::instance().mesh.w;
    Vec3f_ptr &n = Graphic3D::instance().mesh.n;

    ULLInt startW = w_range.x, endW = w_range.y;
    ULLInt startN = n_range.x, endN = n_range.y;

    ULLInt numWs = endW - startW;
    ULLInt numNs = endN - startN;

    ULLInt num = max(numWs, numNs);
    ULLInt gridSize = (num + 255) / 256;

    scaleMeshKernel<<<gridSize, 256>>>(
        w.x + startW, w.y + startW, w.z + startW, numWs,
        n.x + startN, n.y + startN, n.z + startN, numNs,
        origin.x, origin.y, origin.z, scl.x, scl.y, scl.z
    );
    hipDeviceSynchronize();
}

// ======================= Mesh3D =======================

// Free
void Mesh3D::free() {
    s.free();
    w.free();
    t.free();
    n.free();
    c.free();
    f.free();
}

// Push
void Mesh3D::push(Mesh &mesh) {
    ULLInt offsetV = w.size;
    ULLInt offsetT = t.size;
    ULLInt offsetN = n.size;

    // Set the range of stuff
    mesh.w_range = {offsetV, offsetV + mesh.wx.size()};
    mesh.t_range = {offsetT, offsetT + mesh.tu.size()};
    mesh.n_range = {offsetN, offsetN + mesh.nx.size()};
    mesh.c_range = {offsetV, offsetV + mesh.cr.size()};

    Vec3f_ptr newW;
    Vec2f_ptr newT;
    Vec3f_ptr newN;
    Vec4f_ptr newC;
    Vec4ulli_ptr newFvtnm;
    ULLInt wSize = mesh.wx.size();
    ULLInt tSize = mesh.tu.size();
    ULLInt nSize = mesh.nx.size();
    ULLInt cSize = mesh.cr.size();
    ULLInt fvtnmSize = mesh.fw.size();
    newW.malloc(wSize);
    newT.malloc(tSize);
    newN.malloc(nSize);
    newC.malloc(cSize);
    newFvtnm.malloc(fvtnmSize);

    // Stream for async memory copy
    hipStream_t stream;
    hipStreamCreate(&stream);

    hipMemcpyAsync(newW.x, mesh.wx.data(), wSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newW.y, mesh.wy.data(), wSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newW.z, mesh.wz.data(), wSize * sizeof(float), hipMemcpyHostToDevice, stream);

    hipMemcpyAsync(newT.x, mesh.tu.data(), tSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newT.y, mesh.tv.data(), tSize * sizeof(float), hipMemcpyHostToDevice, stream);

    hipMemcpyAsync(newN.x, mesh.nx.data(), nSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newN.y, mesh.ny.data(), nSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newN.z, mesh.nz.data(), nSize * sizeof(float), hipMemcpyHostToDevice, stream);

    hipMemcpyAsync(newC.x, mesh.cr.data(), cSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newC.y, mesh.cg.data(), cSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newC.z, mesh.cb.data(), cSize * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newC.w, mesh.ca.data(), cSize * sizeof(float), hipMemcpyHostToDevice, stream);

    hipMemcpyAsync(newFvtnm.v, mesh.fw.data(), fvtnmSize * sizeof(ULLInt), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newFvtnm.t, mesh.ft.data(), fvtnmSize * sizeof(ULLInt), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(newFvtnm.n, mesh.fn.data(), fvtnmSize * sizeof(ULLInt), hipMemcpyHostToDevice, stream);

    // Increment face indices
    ULLInt gridSize = (fvtnmSize + 255) / 256;
    incrementFaceIdxKernel<<<gridSize, 256>>>(newFvtnm.v, offsetV, fvtnmSize);
    incrementFaceIdxKernel<<<gridSize, 256>>>(newFvtnm.t, offsetT, fvtnmSize);
    incrementFaceIdxKernel<<<gridSize, 256>>>(newFvtnm.n, offsetN, fvtnmSize);

    w += newW;
    t += newT;
    n += newN;
    c += newC;
    f += newFvtnm;

    s.free();
    s.malloc(w.size);
}
void Mesh3D::push(std::vector<Mesh> &meshes) {
    for (Mesh &mesh : meshes) push(mesh);
}

// Kernel for incrementing face indices
__global__ void incrementFaceIdxKernel(ULLInt *f, ULLInt offset, ULLInt numFs) { // BETA
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numFs) f[idx] += offset;
}

// Kernel for transforming vertices

__global__ void translateMeshKernel(
    float *wx, float *wy, float *wz,
    float tx, float ty, float tz,
    ULLInt numWs
) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWs) {
        wx[idx] += tx;
        wy[idx] += ty;
        wz[idx] += tz;
    }
}

__global__ void rotateMeshKernel(
    float *wx, float *wy, float *wz, ULLInt numWs,
    float *nx, float *ny, float *nz, ULLInt numNs,
    float ox, float oy, float oz,
    float r, short axis
) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numWs) {
        Vec3f p = Vec3f(wx[idx], wy[idx], wz[idx]);

        switch (axis) {
            case 0: p.rotateX(Vec3f(ox, oy, oz), r); break;
            case 1: p.rotateY(Vec3f(ox, oy, oz), r); break;
            case 2: p.rotateZ(Vec3f(ox, oy, oz), r); break;
        }

        wx[idx] = p.x;
        wy[idx] = p.y;
        wz[idx] = p.z;
    }

    if (idx < numNs) {
        Vec3f n = Vec3f(nx[idx], ny[idx], nz[idx]);

        switch (axis) {
            case 0: n.rotateX(Vec3f(), r); break;
            case 1: n.rotateY(Vec3f(), r); break;
            case 2: n.rotateZ(Vec3f(), r); break;
        }

        n /= n.mag(); // Normalize

        nx[idx] = n.x;
        ny[idx] = n.y;
        nz[idx] = n.z;
    }
}

__global__ void scaleMeshKernel(
    float *wx, float *wy, float *wz, ULLInt numWs,
    float *nx, float *ny, float *nz, ULLInt numNs,
    float ox, float oy, float oz,
    float sx, float sy, float sz
) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numWs) {
        wx[idx] = (wx[idx] - ox) * sx + ox;
        wy[idx] = (wy[idx] - oy) * sy + oy;
        wz[idx] = (wz[idx] - oz) * sz + oz;
    }

    if (idx < numNs) {
        nx[idx] *= sx;
        ny[idx] *= sy;
        nz[idx] *= sz;
        // Normalize
        float mag = sqrt(
            nx[idx] * nx[idx] +
            ny[idx] * ny[idx] +
            nz[idx] * nz[idx]
        );
        nx[idx] /= mag;
        ny[idx] /= mag;
        nz[idx] /= mag;
    }
}
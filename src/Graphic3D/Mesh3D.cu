#include "hip/hip_runtime.h"
#include <Mesh3D.cuh>

// Constructor

Mesh3D::Mesh3D(ULLInt numWs, ULLInt numNs, ULLInt numTs, ULLInt numFs) :
    numWs(numWs), numNs(numNs), numTs(numTs), numFs(numFs)
{
    mallocVertices();
    mallocFaces();
}

Mesh3D::Mesh3D(
    UInt id, Vecs3f &world, Vecs3f &normal, Vecs2f &texture, Vecs4f &color,
    Vecs3ulli &faceWs, Vecs3ulli &faceNs, Vecs3ulli &faceTs
) : numWs(world.size()), numNs(normal.size()), numTs(texture.size()), numFs(faceWs.size())
{
    mallocVertices();
    mallocFaces();
    uploadData(id, world, normal, texture, color, faceWs, faceNs, faceTs);
}
Mesh3D::Mesh3D(
    UInt id, Vecs3f &world, Vecs3f &normal, Vecs2f &texture, Vecs4f &color, 
    Vecs3ulli &faceAll
) : numWs(world.size()), numNs(normal.size()), numTs(texture.size()), numFs(faceAll.size())
{
    mallocVertices();
    mallocFaces();
    uploadData(id, world, normal, texture, color, faceAll, faceAll, faceAll);
}

// Memory management

void Mesh3D::mallocVertices() {
    blockNumWs = (numWs + blockSize - 1) / blockSize;
    blockNumNs = (numNs + blockSize - 1) / blockSize;
    blockNumTs = (numTs + blockSize - 1) / blockSize;

    hipMalloc(&world, numWs * sizeof(Vec3f));
    hipMalloc(&normal, numNs * sizeof(Vec3f));
    hipMalloc(&texture, numTs * sizeof(Vec2f));
    hipMalloc(&screen, numWs * sizeof(Vec4f));

    hipMalloc(&color, numWs * sizeof(Vec4f));
}

void Mesh3D::resizeVertices(ULLInt numWs, ULLInt numNs, ULLInt numTs) {
    freeVertices();
    this->numWs = numWs;
    this->numNs = numNs;
    this->numTs = numTs;
    mallocVertices();
}

void Mesh3D::freeVertices() {
    if (world) hipFree(world);
    if (normal) hipFree(normal);
    if (texture) hipFree(texture);
    if (screen) hipFree(screen);
    if (color) hipFree(color);
}

void Mesh3D::mallocFaces() {
    blockNumFs = (numFs + blockSize - 1) / blockSize;

    hipMalloc(&faceWs, numFs * sizeof(Vec3ulli));
    hipMalloc(&faceNs, numFs * sizeof(Vec3ulli));
    hipMalloc(&faceTs, numFs * sizeof(Vec3ulli));
}

void Mesh3D::resizeFaces(ULLInt numFs) {
    freeFaces();
    this->numFs = numFs;
    mallocFaces();
}

void Mesh3D::freeFaces() {
    if (faceWs) hipFree(faceWs);
    if (faceNs) hipFree(faceNs);
    if (faceTs) hipFree(faceTs);
}

void Mesh3D::free() {
    freeVertices();
    freeFaces();
}

// Upload host data to device

void Mesh3D::uploadData(
    UInt id, Vecs3f &world, Vecs3f &normal, Vecs2f &texture, Vecs4f &color,
    Vecs3ulli &faceWs, Vecs3ulli &faceNs, Vecs3ulli &faceTs
) {
    // Set the vertices data
    hipMemcpy(this->world, world.data(), world.size() * sizeof(Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(this->normal, normal.data(), normal.size() * sizeof(Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(this->texture, texture.data(), texture.size() * sizeof(Vec2f), hipMemcpyHostToDevice);

    hipMemcpy(this->color, color.data(), color.size() * sizeof(Vec4f), hipMemcpyHostToDevice);

    // Faces indices
    hipMemcpy(this->faceWs, faceWs.data(), faceWs.size() * sizeof(Vec3ulli), hipMemcpyHostToDevice);
    hipMemcpy(this->faceNs, faceNs.data(), faceNs.size() * sizeof(Vec3ulli), hipMemcpyHostToDevice);
    hipMemcpy(this->faceTs, faceTs.data(), faceTs.size() * sizeof(Vec3ulli), hipMemcpyHostToDevice);
}

// Mesh operators

void Mesh3D::operator+=(Mesh3D &mesh) {
    // Resize vertices
    ULLInt newNumWs = numWs + mesh.numWs;
    ULLInt newNumNs = numNs + mesh.numNs;
    ULLInt newNumTs = numTs + mesh.numTs;
    Vec3f *newWorld;
    Vec3f *newNormal;
    Vec2f *newTexture;
    Vec4f *newScreen;
    UInt *newWObjId;
    UInt *newNObjId;
    UInt *newTObjId;
    Vec4f *newColor;
    hipMalloc(&newWorld, newNumWs * sizeof(Vec3f));
    hipMalloc(&newNormal, newNumNs * sizeof(Vec3f));
    hipMalloc(&newTexture, newNumTs * sizeof(Vec2f));
    hipMalloc(&newScreen, newNumWs * sizeof(Vec4f));
    hipMalloc(&newWObjId, newNumWs * sizeof(UInt));
    hipMalloc(&newNObjId, newNumNs * sizeof(UInt));
    hipMalloc(&newTObjId, newNumTs * sizeof(UInt));
    hipMalloc(&newColor, newNumWs * sizeof(Vec4f));
    // Copy old data
    hipMemcpy(newWorld, world, numWs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newNormal, normal, numNs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newTexture, texture, numTs * sizeof(Vec2f), hipMemcpyDeviceToDevice);
    hipMemcpy(newScreen, screen, numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    hipMemcpy(newColor, color, numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    // Copy new data
    hipMemcpy(newWorld + numWs, mesh.world, mesh.numWs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newNormal + numNs, mesh.normal, mesh.numNs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newTexture + numTs, mesh.texture, mesh.numTs * sizeof(Vec2f), hipMemcpyDeviceToDevice);
    hipMemcpy(newScreen + numWs, mesh.screen, mesh.numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    hipMemcpy(newColor + numWs, mesh.color, mesh.numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    // Free old data and update
    freeVertices();
    world = newWorld;
    normal = newNormal;
    texture = newTexture;
    screen = newScreen;
    color = newColor;

    // Resize faces (with offset for the added vertices)
    ULLInt newNumFs = numFs + mesh.numFs;
    ULLInt newBlockNumFs = (newNumFs + blockSize - 1) / blockSize;

    Vec3ulli *newFaceWs;
    Vec3ulli *newFaceNs;
    Vec3ulli *newFaceTs;

    hipMalloc(&newFaceWs, newNumFs * sizeof(Vec3ulli));
    hipMalloc(&newFaceNs, newNumFs * sizeof(Vec3ulli));
    hipMalloc(&newFaceTs, newNumFs * sizeof(Vec3ulli));

    hipMemcpy(newFaceWs, faceWs, numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceNs, faceNs, numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceTs, faceTs, numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);

    hipMemcpy(newFaceWs + numFs, mesh.faceWs, mesh.numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceNs + numFs, mesh.faceNs, mesh.numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceTs + numFs, mesh.faceTs, mesh.numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    incrementFaceIdxKernel<<<newBlockNumFs, blockSize>>>(newFaceWs, numWs, numFs, newNumFs);
    incrementFaceIdxKernel<<<newBlockNumFs, blockSize>>>(newFaceNs, numNs, numFs, newNumFs);
    incrementFaceIdxKernel<<<newBlockNumFs, blockSize>>>(newFaceTs, numTs, numFs, newNumFs);

    freeFaces();
    faceWs = newFaceWs;
    faceNs = newFaceNs;
    faceTs = newFaceTs;

    // Update number of vertices and faces
    numWs = newNumWs;
    numNs = newNumNs;
    numTs = newNumTs;
    numFs = newNumFs;
    blockNumWs = (numWs + blockSize - 1) / blockSize;
    blockNumNs = (numNs + blockSize - 1) / blockSize;
    blockNumTs = (numTs + blockSize - 1) / blockSize;
    blockNumFs = (numFs + blockSize - 1) / blockSize;
}

// Kernel for incrementing face indices
__global__ void incrementFaceIdxKernel(Vec3ulli *faces, ULLInt offset, ULLInt numFs, ULLInt newNumFs) { // BETA
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < newNumFs && idx >= numFs) faces[idx] += offset;
}
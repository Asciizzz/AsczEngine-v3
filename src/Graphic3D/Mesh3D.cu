#include "hip/hip_runtime.h"
#include <Mesh3D.cuh>

// Constructor

Mesh3D::Mesh3D(ULLInt numWs, ULLInt numNs, ULLInt numTs, ULLInt numFs) :
    numWs(numWs), numNs(numNs), numTs(numTs), numFs(numFs)
{
    mallocVertices();
    mallocFaces();
}

Mesh3D::Mesh3D(
    UInt id, Vecs3f &world, Vecs3f &normal, Vecs2f &texture, Vecs4f &color,
    Vecs3ulli &faceWs, Vecs3ulli &faceNs, Vecs3ulli &faceTs
) : numWs(world.size()), numNs(normal.size()), numTs(texture.size()), numFs(faceWs.size())
{
    mallocVertices();
    mallocFaces();
    uploadData(id, world, normal, texture, color, faceWs, faceNs, faceTs);
}
Mesh3D::Mesh3D(
    UInt id, Vecs3f &world, Vecs3f &normal, Vecs2f &texture, Vecs4f &color, 
    Vecs3ulli &faceAll
) : numWs(world.size()), numNs(normal.size()), numTs(texture.size()), numFs(faceAll.size())
{
    mallocVertices();
    mallocFaces();
    uploadData(id, world, normal, texture, color, faceAll, faceAll, faceAll);
}

// Memory management

void Mesh3D::mallocVertices() {
    blockNumWs = (numWs + blockSize - 1) / blockSize;
    blockNumNs = (numNs + blockSize - 1) / blockSize;
    blockNumTs = (numTs + blockSize - 1) / blockSize;

    hipMalloc(&world, numWs * sizeof(Vec3f));
    hipMalloc(&normal, numNs * sizeof(Vec3f));
    hipMalloc(&texture, numTs * sizeof(Vec2f));
    hipMalloc(&screen, numWs * sizeof(Vec4f));

    hipMalloc(&color, numWs * sizeof(Vec4f));

    hipMalloc(&wObjId, numWs * sizeof(UInt));
    hipMalloc(&nObjId, numNs * sizeof(UInt));
    hipMalloc(&tObjId, numTs * sizeof(UInt));
}

void Mesh3D::resizeVertices(ULLInt numWs, ULLInt numNs, ULLInt numTs) {
    freeVertices();
    this->numWs = numWs;
    this->numNs = numNs;
    this->numTs = numTs;
    mallocVertices();
}

void Mesh3D::freeVertices() {
    hipFree(world);
    hipFree(normal);
    hipFree(texture);
    hipFree(screen);
    hipFree(color);
    hipFree(wObjId);
    hipFree(nObjId);
    hipFree(tObjId);
}

void Mesh3D::mallocFaces() {
    blockNumFs = (numFs + blockSize - 1) / blockSize;

    hipMalloc(&faceWs, numFs * sizeof(Vec3ulli));
    hipMalloc(&faceNs, numFs * sizeof(Vec3ulli));
    hipMalloc(&faceTs, numFs * sizeof(Vec3ulli));

    hipMalloc(&d_numVisibFs, sizeof(ULLInt));
    hipMemset(d_numVisibFs, 0, sizeof(ULLInt));
    hipMalloc(&visibFWs, numFs * sizeof(Vec3ulli));
    hipMalloc(&visibFNs, numFs * sizeof(Vec3ulli));
    hipMalloc(&visibFTs, numFs * sizeof(Vec3ulli));

}

void Mesh3D::resizeFaces(ULLInt numFs) {
    freeFaces();
    this->numFs = numFs;
    mallocFaces();
}

void Mesh3D::freeFaces() {
    hipFree(faceWs);
    hipFree(faceNs);
    hipFree(faceTs);

    hipFree(d_numVisibFs);
    hipFree(visibFWs);
    hipFree(visibFNs);
    hipFree(visibFTs);
}

void Mesh3D::free() {
    freeVertices();
    freeFaces();
}

// Upload host data to device

void Mesh3D::uploadData(
    UInt id, Vecs3f &world, Vecs3f &normal, Vecs2f &texture, Vecs4f &color,
    Vecs3ulli &faceWs, Vecs3ulli &faceNs, Vecs3ulli &faceTs
) {
    // Vertices

    // Set obj Id for each vertex attribute
    setObjIdKernel<<<blockNumWs, blockSize>>>(this->wObjId, numWs, id);
    setObjIdKernel<<<blockNumNs, blockSize>>>(this->nObjId, numNs, id);
    setObjIdKernel<<<blockNumTs, blockSize>>>(this->tObjId, numTs, id);

    // Set the actual data
    hipMemcpy(this->world, world.data(), world.size() * sizeof(Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(this->normal, normal.data(), normal.size() * sizeof(Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(this->texture, texture.data(), texture.size() * sizeof(Vec2f), hipMemcpyHostToDevice);

    hipMemcpy(this->color, color.data(), color.size() * sizeof(Vec4f), hipMemcpyHostToDevice);

    // Faces indices
    hipMemcpy(this->faceWs, faceWs.data(), faceWs.size() * sizeof(Vec3ulli), hipMemcpyHostToDevice);
    hipMemcpy(this->faceNs, faceNs.data(), faceNs.size() * sizeof(Vec3ulli), hipMemcpyHostToDevice);
    hipMemcpy(this->faceTs, faceTs.data(), faceTs.size() * sizeof(Vec3ulli), hipMemcpyHostToDevice);
}

// Mesh operators

void Mesh3D::operator+=(Mesh3D &mesh) {
    // Resize vertices
    ULLInt newNumWs = numWs + mesh.numWs;
    ULLInt newNumNs = numNs + mesh.numNs;
    ULLInt newNumTs = numTs + mesh.numTs;
    Vec3f *newWorld;
    Vec3f *newNormal;
    Vec2f *newTexture;
    Vec4f *newScreen;
    UInt *newWObjId;
    UInt *newNObjId;
    UInt *newTObjId;
    Vec4f *newColor;
    hipMalloc(&newWorld, newNumWs * sizeof(Vec3f));
    hipMalloc(&newNormal, newNumNs * sizeof(Vec3f));
    hipMalloc(&newTexture, newNumTs * sizeof(Vec2f));
    hipMalloc(&newScreen, newNumWs * sizeof(Vec4f));
    hipMalloc(&newWObjId, newNumWs * sizeof(UInt));
    hipMalloc(&newNObjId, newNumNs * sizeof(UInt));
    hipMalloc(&newTObjId, newNumTs * sizeof(UInt));
    hipMalloc(&newColor, newNumWs * sizeof(Vec4f));
    // Copy old data
    hipMemcpy(newWorld, world, numWs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newNormal, normal, numNs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newTexture, texture, numTs * sizeof(Vec2f), hipMemcpyDeviceToDevice);
    hipMemcpy(newScreen, screen, numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    hipMemcpy(newWObjId, wObjId, numWs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newNObjId, nObjId, numNs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newTObjId, tObjId, numTs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newColor, color, numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    // Copy new data
    hipMemcpy(newWorld + numWs, mesh.world, mesh.numWs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newNormal + numNs, mesh.normal, mesh.numNs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newTexture + numTs, mesh.texture, mesh.numTs * sizeof(Vec2f), hipMemcpyDeviceToDevice);
    hipMemcpy(newScreen + numWs, mesh.screen, mesh.numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    hipMemcpy(newWObjId + numWs, mesh.wObjId, mesh.numWs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newNObjId + numNs, mesh.nObjId, mesh.numNs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newTObjId + numTs, mesh.tObjId, mesh.numTs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newColor + numWs, mesh.color, mesh.numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    // Free old data and update
    freeVertices();
    world = newWorld;
    normal = newNormal;
    texture = newTexture;
    screen = newScreen;
    wObjId = newWObjId;
    nObjId = newNObjId;
    tObjId = newTObjId;
    color = newColor;

    // Resize faces (with offset for the added vertices)
    ULLInt newNumFs = numFs + mesh.numFs;
    ULLInt newBlockNumFs = (newNumFs + blockSize - 1) / blockSize;

    Vec3ulli *newFaceWs;
    Vec3ulli *newFaceNs;
    Vec3ulli *newFaceTs;

    ULLInt *newDNumVisibleFs;
    Vec3ulli *newVisibleFWs;
    Vec3ulli *newVisibleFNs;
    Vec3ulli *newVisibleFTs;

    hipMalloc(&newFaceWs, newNumFs * sizeof(Vec3ulli));
    hipMalloc(&newFaceNs, newNumFs * sizeof(Vec3ulli));
    hipMalloc(&newFaceTs, newNumFs * sizeof(Vec3ulli));

    hipMemcpy(newFaceWs, faceWs, numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceNs, faceNs, numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceTs, faceTs, numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);

    hipMemcpy(newFaceWs + numFs, mesh.faceWs, mesh.numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceNs + numFs, mesh.faceNs, mesh.numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceTs + numFs, mesh.faceTs, mesh.numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    incrementFaceIdxKernel<<<newBlockNumFs, blockSize>>>(newFaceWs, numWs, numFs, newNumFs);
    incrementFaceIdxKernel<<<newBlockNumFs, blockSize>>>(newFaceNs, numNs, numFs, newNumFs);
    incrementFaceIdxKernel<<<newBlockNumFs, blockSize>>>(newFaceTs, numTs, numFs, newNumFs);

    hipMalloc(&newDNumVisibleFs, sizeof(ULLInt));
    hipMalloc(&newVisibleFWs, newNumFs * sizeof(Vec3ulli));
    hipMalloc(&newVisibleFNs, newNumFs * sizeof(Vec3ulli));
    hipMalloc(&newVisibleFTs, newNumFs * sizeof(Vec3ulli));

    freeFaces();
    faceWs = newFaceWs;
    faceNs = newFaceNs;
    faceTs = newFaceTs;
    d_numVisibFs = newDNumVisibleFs;
    visibFWs = newVisibleFWs;
    visibFNs = newVisibleFNs;
    visibFTs = newVisibleFTs;

    // Update number of vertices and faces
    numWs = newNumWs;
    numNs = newNumNs;
    numTs = newNumTs;
    numFs = newNumFs;
    blockNumWs = (numWs + blockSize - 1) / blockSize;
    blockNumNs = (numNs + blockSize - 1) / blockSize;
    blockNumTs = (numTs + blockSize - 1) / blockSize;
    blockNumFs = (numFs + blockSize - 1) / blockSize;
}

// Transformations (with obj Id)

void Mesh3D::translate(UInt objID, Vec3f t) {
    translateWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, false, numWs, objID, t);
}
void Mesh3D::rotate(UInt objID, Vec3f origin, Vec3f rot) {
    rotateWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, false, numWs, objID, origin, rot);
    rotateNormalKernel<<<blockNumNs, blockSize>>>(normal, nObjId, false, numNs, objID, origin, rot);
}
void Mesh3D::scale(UInt objID, Vec3f origin, Vec3f scl) {
    scaleWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, false, numWs, objID, origin, scl);
    scaleNormalKernel<<<blockNumNs, blockSize>>>(normal, nObjId, false, numNs, objID, origin, scl);
}

// Transformations (all obj Ids)

void Mesh3D::translate(Vec3f t) {
    translateWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, true, numWs, 0, t);
}
void Mesh3D::rotate(Vec3f origin, Vec3f rot) {
    rotateWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, true, numWs, 0, origin, rot);
    rotateNormalKernel<<<blockNumNs, blockSize>>>(normal, nObjId, true, numNs, 0, origin, rot);
}
void Mesh3D::scale(Vec3f origin, Vec3f scl) {
    scaleWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, true, numWs, 0, origin, scl);
    scaleNormalKernel<<<blockNumNs, blockSize>>>(normal, nObjId, true, numNs, 0, origin, scl);
}

// Kernel for incrementing face indices
__global__ void incrementFaceIdxKernel(Vec3ulli *faces, ULLInt offset, ULLInt numFs, ULLInt newNumFs) { // BETA
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < newNumFs && idx >= numFs) faces[idx] += offset;
}

// Kernel for preparing vertices
__global__ void setObjIdKernel(UInt *objId, ULLInt numWs, UInt id) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWs) objId[idx] = id;
}

// Kernel for transforming vertices
__global__ void translateWorldKernel(Vec3f *world, UInt *wObjId, bool allId, ULLInt numWs, UInt objID, Vec3f t) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numWs || (!allId && wObjId[idx] != objID)) return;

    world[idx].translate(t);
}
__global__ void rotateWorldKernel(Vec3f *world, UInt *wObjId, bool allId, ULLInt numWs, UInt objID, Vec3f origin, Vec3f rot) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numWs || (!allId && wObjId[idx] != objID)) return;

    world[idx].rotate(origin, rot);
}
__global__ void scaleWorldKernel(Vec3f *world, UInt *wObjId, bool allId, ULLInt numWs, UInt objID, Vec3f origin, Vec3f scl) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numWs || (!allId && wObjId[idx] != objID)) return;

    world[idx].scale(origin, scl);
}

// Rotate and scale normals
__global__ void rotateNormalKernel(Vec3f *normal, UInt *nObjId, bool allID, ULLInt numNs, UInt objID, Vec3f origin, Vec3f rot) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numNs || (!allID && nObjId[idx] != objID)) return;

    normal[idx].rotate(origin, rot);
}
__global__ void scaleNormalKernel(Vec3f *normal, UInt *nObjId, bool allID, ULLInt numNs, UInt objID, Vec3f origin, Vec3f scl) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numNs || (!allID && nObjId[idx] != objID)) return;

    normal[idx].scale(origin, scl);
}
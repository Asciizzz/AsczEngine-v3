#include "hip/hip_runtime.h"
#include <Mesh3D.cuh>

// Constructor

Mesh3D::Mesh3D(ULInt numWs, ULInt numNs, ULInt numTs, ULInt numFs) :
    numWs(numWs), numNs(numNs), numTs(numTs), numFs(numFs)
{
    mallocVertices();
    mallocFaces();
}

Mesh3D::Mesh3D(
    UInt id, Vecs3f &world, Vecs3f &normal, Vecs2f &texture, Vecs4f &color,
    Vecs3uli &faceWs, Vecs3uli &faceNs, Vecs3uli &faceTs
) : numWs(world.size()), numNs(normal.size()), numTs(texture.size()), numFs(faceWs.size())
{
    mallocVertices();
    mallocFaces();
    uploadData(id, world, normal, texture, color, faceWs, faceNs, faceTs);
}
Mesh3D::Mesh3D(
    UInt id, Vecs3f &world, Vecs3f &normal, Vecs2f &texture, Vecs4f &color, 
    Vecs3uli &faceAll
) : numWs(world.size()), numNs(normal.size()), numTs(texture.size()), numFs(faceAll.size())
{
    mallocVertices();
    mallocFaces();
    uploadData(id, world, normal, texture, color, faceAll, faceAll, faceAll);
}

// Memory management

void Mesh3D::mallocVertices() {
    blockNumWs = (numWs + blockSize - 1) / blockSize;
    blockNumNs = (numNs + blockSize - 1) / blockSize;
    blockNumTs = (numTs + blockSize - 1) / blockSize;

    hipMalloc(&world, numWs * sizeof(Vec3f));
    hipMalloc(&normal, numNs * sizeof(Vec3f));
    hipMalloc(&texture, numTs * sizeof(Vec2f));
    hipMalloc(&screen, numWs * sizeof(Vec4f));

    hipMalloc(&color, numWs * sizeof(Vec4f));

    hipMalloc(&wObjId, numWs * sizeof(UInt));
    hipMalloc(&nObjId, numNs * sizeof(UInt));
    hipMalloc(&tObjId, numTs * sizeof(UInt));
}

void Mesh3D::resizeVertices(ULInt numWs, ULInt numNs, ULInt numTs) {
    freeVertices();
    this->numWs = numWs;
    this->numNs = numNs;
    this->numTs = numTs;
    mallocVertices();
}

void Mesh3D::freeVertices() {
    hipFree(world);
    hipFree(normal);
    hipFree(texture);
    hipFree(screen);
    hipFree(color);
    hipFree(wObjId);
    hipFree(nObjId);
    hipFree(tObjId);
}

void Mesh3D::mallocFaces() {
    blockNumFs = (numFs + blockSize - 1) / blockSize;
    hipMalloc(&faceWs, numFs * sizeof(Vec3uli));
    hipMalloc(&faceNs, numFs * sizeof(Vec3uli));
    hipMalloc(&faceTs, numFs * sizeof(Vec3uli));
}

void Mesh3D::resizeFaces(ULInt numFs) {
    freeFaces();
    this->numFs = numFs;
    mallocFaces();
}

void Mesh3D::freeFaces() {
    hipFree(faceWs);
    hipFree(faceNs);
    hipFree(faceTs);
}

void Mesh3D::free() {
    freeVertices();
    freeFaces();
}

// Upload host data to device

void Mesh3D::uploadData(
    UInt id, Vecs3f &world, Vecs3f &normal, Vecs2f &texture, Vecs4f &color,
    Vecs3uli &faceWs, Vecs3uli &faceNs, Vecs3uli &faceTs
) {
    // Vertices

    // Set obj Id for each vertex attribute
    setObjIdKernel<<<blockNumWs, blockSize>>>(this->wObjId, numWs, id);
    setObjIdKernel<<<blockNumNs, blockSize>>>(this->nObjId, numNs, id);
    setObjIdKernel<<<blockNumTs, blockSize>>>(this->tObjId, numTs, id);

    // Set the actual data
    hipMemcpy(this->world, world.data(), world.size() * sizeof(Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(this->normal, normal.data(), normal.size() * sizeof(Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(this->texture, texture.data(), texture.size() * sizeof(Vec2f), hipMemcpyHostToDevice);
    
    hipMemcpy(this->color, color.data(), color.size() * sizeof(Vec4f), hipMemcpyHostToDevice);
    
    // Faces indices
    hipMemcpy(this->faceWs, faceWs.data(), faceWs.size() * sizeof(Vec3uli), hipMemcpyHostToDevice);
    hipMemcpy(this->faceNs, faceNs.data(), faceNs.size() * sizeof(Vec3uli), hipMemcpyHostToDevice);
    hipMemcpy(this->faceTs, faceTs.data(), faceTs.size() * sizeof(Vec3uli), hipMemcpyHostToDevice);
}

// Mesh operators

void Mesh3D::operator+=(Mesh3D &mesh) {
    // Resize vertices
    ULInt newNumWs = numWs + mesh.numWs;
    ULInt newNumNs = numNs + mesh.numNs;
    ULInt newNumTs = numTs + mesh.numTs;
    Vec3f *newWorld;
    Vec3f *newNormal;
    Vec2f *newTexture;
    Vec4f *newScreen;
    UInt *newWObjId;
    UInt *newNObjId;
    UInt *newTObjId;
    Vec4f *newColor;
    hipMalloc(&newWorld, newNumWs * sizeof(Vec3f));
    hipMalloc(&newNormal, newNumNs * sizeof(Vec3f));
    hipMalloc(&newTexture, newNumTs * sizeof(Vec2f));
    hipMalloc(&newScreen, newNumWs * sizeof(Vec4f));
    hipMalloc(&newWObjId, newNumWs * sizeof(UInt));
    hipMalloc(&newNObjId, newNumNs * sizeof(UInt));
    hipMalloc(&newTObjId, newNumTs * sizeof(UInt));
    hipMalloc(&newColor, newNumWs * sizeof(Vec4f));
    // Copy old data
    hipMemcpy(newWorld, world, numWs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newNormal, normal, numNs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newTexture, texture, numTs * sizeof(Vec2f), hipMemcpyDeviceToDevice);
    hipMemcpy(newScreen, screen, numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    hipMemcpy(newWObjId, wObjId, numWs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newNObjId, nObjId, numNs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newTObjId, tObjId, numTs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newColor, color, numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    // Copy new data
    hipMemcpy(newWorld + numWs, mesh.world, mesh.numWs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newNormal + numNs, mesh.normal, mesh.numNs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newTexture + numTs, mesh.texture, mesh.numTs * sizeof(Vec2f), hipMemcpyDeviceToDevice);
    hipMemcpy(newScreen + numWs, mesh.screen, mesh.numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    hipMemcpy(newWObjId + numWs, mesh.wObjId, mesh.numWs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newNObjId + numNs, mesh.nObjId, mesh.numNs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newTObjId + numTs, mesh.tObjId, mesh.numTs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newColor + numWs, mesh.color, mesh.numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    // Free old data and update
    freeVertices();
    world = newWorld;
    normal = newNormal;
    texture = newTexture;
    screen = newScreen;
    wObjId = newWObjId;
    nObjId = newNObjId;
    tObjId = newTObjId;
    color = newColor;

    // Resize faces (with offset for the added vertices)
    ULInt newNumFs = numFs + mesh.numFs;
    ULInt newBlockNumFs = (newNumFs + blockSize - 1) / blockSize;

    Vec3uli *newFaceWs;
    Vec3uli *newFaceNs;
    Vec3uli *newFaceTs;

    hipMalloc(&newFaceWs, newNumFs * sizeof(Vec3uli));
    hipMalloc(&newFaceNs, newNumFs * sizeof(Vec3uli));
    hipMalloc(&newFaceTs, newNumFs * sizeof(Vec3uli));

    hipMemcpy(newFaceWs, faceWs, numFs * sizeof(Vec3uli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceNs, faceNs, numFs * sizeof(Vec3uli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceTs, faceTs, numFs * sizeof(Vec3uli), hipMemcpyDeviceToDevice);

    hipMemcpy(newFaceWs + numFs, mesh.faceWs, mesh.numFs * sizeof(Vec3uli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceNs + numFs, mesh.faceNs, mesh.numFs * sizeof(Vec3uli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceTs + numFs, mesh.faceTs, mesh.numFs * sizeof(Vec3uli), hipMemcpyDeviceToDevice);
    incrementFaceIdxKernel<<<newBlockNumFs, blockSize>>>(newFaceWs, numWs, numFs, newNumFs);
    incrementFaceIdxKernel<<<newBlockNumFs, blockSize>>>(newFaceNs, numNs, numFs, newNumFs);
    incrementFaceIdxKernel<<<newBlockNumFs, blockSize>>>(newFaceTs, numTs, numFs, newNumFs);

    freeFaces();
    faceWs = newFaceWs;
    faceNs = newFaceNs;
    faceTs = newFaceTs;

    // Update number of vertices and faces
    numWs = newNumWs;
    numNs = newNumNs;
    numTs = newNumTs;
    numFs = newNumFs;
    blockNumWs = (numWs + blockSize - 1) / blockSize;
    blockNumNs = (numNs + blockSize - 1) / blockSize;
    blockNumTs = (numTs + blockSize - 1) / blockSize;
    blockNumFs = (numFs + blockSize - 1) / blockSize;
}

// Transformations (with obj Id)

void Mesh3D::translate(UInt objID, Vec3f t) {
    translateWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, false, numWs, objID, t);
}
void Mesh3D::rotate(UInt objID, Vec3f origin, Vec3f rot) {
    rotateWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, false, numWs, objID, origin, rot);
    rotateNormalKernel<<<blockNumNs, blockSize>>>(normal, nObjId, false, numNs, objID, origin, rot);
}
void Mesh3D::scale(UInt objID, Vec3f origin, Vec3f scl) {
    scaleWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, false, numWs, objID, origin, scl);
    scaleNormalKernel<<<blockNumNs, blockSize>>>(normal, nObjId, false, numNs, objID, origin, scl);
}

// Transformations (all obj Ids)

void Mesh3D::translate(Vec3f t) {
    translateWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, true, numWs, 0, t);
}
void Mesh3D::rotate(Vec3f origin, Vec3f rot) {
    rotateWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, true, numWs, 0, origin, rot);
    rotateNormalKernel<<<blockNumNs, blockSize>>>(normal, nObjId, true, numNs, 0, origin, rot);
}
void Mesh3D::scale(Vec3f origin, Vec3f scl) {
    scaleWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, true, numWs, 0, origin, scl);
    scaleNormalKernel<<<blockNumNs, blockSize>>>(normal, nObjId, true, numNs, 0, origin, scl);
}

// Kernel for incrementing face indices
__global__ void incrementFaceIdxKernel(Vec3uli *faces, ULInt offset, ULInt numFs, ULInt newNumFs) { // BETA
    ULInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < newNumFs && idx >= numFs) faces[idx] += offset;
}

// Kernel for preparing vertices
__global__ void setObjIdKernel(UInt *objId, ULInt numWs, UInt id) {
    ULInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWs) objId[idx] = id;
}

// Kernel for transforming vertices
__global__ void translateWorldKernel(Vec3f *world, UInt *wObjId, bool allId, ULInt numWs, UInt objID, Vec3f t) {
    ULInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numWs || (!allId && wObjId[idx] != objID)) return;

    world[idx].translate(t);
}
__global__ void rotateWorldKernel(Vec3f *world, UInt *wObjId, bool allId, ULInt numWs, UInt objID, Vec3f origin, Vec3f rot) {
    ULInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numWs || (!allId && wObjId[idx] != objID)) return;

    world[idx].rotate(origin, rot);
}
__global__ void scaleWorldKernel(Vec3f *world, UInt *wObjId, bool allId, ULInt numWs, UInt objID, Vec3f origin, Vec3f scl) {
    ULInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numWs || (!allId && wObjId[idx] != objID)) return;

    world[idx].scale(origin, scl);
}

// Rotate and scale normals
__global__ void rotateNormalKernel(Vec3f *normal, UInt *nObjId, bool allID, ULInt numNs, UInt objID, Vec3f origin, Vec3f rot) {
    ULInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numNs || (!allID && nObjId[idx] != objID)) return;

    normal[idx].rotate(origin, rot);
}
__global__ void scaleNormalKernel(Vec3f *normal, UInt *nObjId, bool allID, ULInt numNs, UInt objID, Vec3f origin, Vec3f scl) {
    ULInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numNs || (!allID && nObjId[idx] != objID)) return;

    normal[idx].scale(origin, scl);
}
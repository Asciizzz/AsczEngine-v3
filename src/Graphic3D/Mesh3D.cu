#include "hip/hip_runtime.h"
#include <Mesh3D.cuh>
#include <Render3D.cuh>

// VERTICES

Vertices::Vertices() :
    x(nullptr), y(nullptr), z(nullptr),
    nx(nullptr), ny(nullptr), nz(nullptr),
    u(nullptr), v(nullptr), meshId(nullptr) {}

void Vertices::allocate(uint32_t numVertices) {
    numVtxs = numVertices;
    hipMalloc(&x, numVertices * sizeof(float));
    hipMalloc(&y, numVertices * sizeof(float));
    hipMalloc(&z, numVertices * sizeof(float));
    hipMalloc(&nx, numVertices * sizeof(float));
    hipMalloc(&ny, numVertices * sizeof(float));
    hipMalloc(&nz, numVertices * sizeof(float));
    hipMalloc(&u, numVertices * sizeof(float));
    hipMalloc(&v, numVertices * sizeof(float));
    hipMalloc(&meshId, numVertices * sizeof(uint32_t));
}

void Vertices::resize(uint32_t numVertices) {
    float *newX, *newY, *newZ, *newNX, *newNY, *newNZ, *newU, *newV;
    uint32_t *newMeshId;
    hipMalloc(&newX, numVertices * sizeof(float));
    hipMalloc(&newY, numVertices * sizeof(float));
    hipMalloc(&newZ, numVertices * sizeof(float));
    hipMalloc(&newNX, numVertices * sizeof(float));
    hipMalloc(&newNY, numVertices * sizeof(float));
    hipMalloc(&newNZ, numVertices * sizeof(float));
    hipMalloc(&newU, numVertices * sizeof(float));
    hipMalloc(&newV, numVertices * sizeof(float));
    hipMalloc(&newMeshId, numVertices * sizeof(uint32_t));

    hipMemcpy(newX, x, numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(newY, y, numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(newZ, z, numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(newNX, nx, numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(newNY, ny, numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(newNZ, nz, numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(newU, u, numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(newV, v, numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(newMeshId, meshId, numVtxs * sizeof(uint32_t), hipMemcpyDeviceToDevice);

    free();
    numVtxs = numVertices;
    x = newX; y = newY; z = newZ;
    nx = newNX; ny = newNY; nz = newNZ;
    u = newU; v = newV; meshId = newMeshId;
}

void Vertices::free() {
    if (x) hipFree(x);
    if (y) hipFree(y);
    if (z) hipFree(z);
    if (nx) hipFree(nx);
    if (ny) hipFree(ny);
    if (nz) hipFree(nz);
    if (u) hipFree(u);
    if (v) hipFree(v);
    if (meshId) hipFree(meshId);
}

// INDICES

Indices::Indices() : vertexId(nullptr), meshId(nullptr) {}

void Indices::allocate(uint32_t numIndices) {
    numIdxs = numIndices;
    hipMalloc(&vertexId, numIndices * sizeof(uint32_t));
    hipMalloc(&meshId, numIndices * sizeof(uint32_t));
}

void Indices::resize(uint32_t numIndices) {
    uint32_t *newVertexId, *newMeshId;

    hipMalloc(&newVertexId, numIndices * sizeof(uint32_t));
    hipMalloc(&newMeshId, numIndices * sizeof(uint32_t));

    hipMemcpy(newVertexId, vertexId, numIdxs * sizeof(uint32_t), hipMemcpyDeviceToDevice);
    hipMemcpy(newMeshId, meshId, numIdxs * sizeof(uint32_t), hipMemcpyDeviceToDevice);
    
    free();
    numIdxs = numIndices;
    vertexId = newVertexId;
    meshId = newMeshId;
}

void Indices::free() {
    if (vertexId) hipFree(vertexId);
    if (meshId) hipFree(meshId);
}

// PROJECTIONS

Projections::Projections() : x(nullptr), y(nullptr), z(nullptr) {}

void Projections::allocate(uint32_t numVertices) {
    numVtxs = numVertices;
    hipMalloc(&x, numVertices * sizeof(float));
    hipMalloc(&y, numVertices * sizeof(float));
    hipMalloc(&z, numVertices * sizeof(float));
}

void Projections::resize(uint32_t numVertices) {
    float *newX, *newY, *newZ;
    hipMalloc(&newX, numVertices * sizeof(float));
    hipMalloc(&newY, numVertices * sizeof(float));
    hipMalloc(&newZ, numVertices * sizeof(float));

    hipMemcpy(newX, x, numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(newY, y, numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(newZ, z, numVtxs * sizeof(float), hipMemcpyDeviceToDevice);

    free();
    numVtxs = numVertices;
    x = newX; y = newY; z = newZ;
}

void Projections::free() {
    if (x) hipFree(x);
    if (y) hipFree(y);
    if (z) hipFree(z);
}

// MESH3D

Mesh3D::Mesh3D(uint32_t vertexCount, uint32_t indexCount, uint32_t meshId) :
    numVtxs(vertexCount),
    numIdxs(indexCount),
    meshId(meshId)
{
    allocate(vertexCount, indexCount);
}

Mesh3D::~Mesh3D() {
    free();
}

void Mesh3D::allocate(uint32_t vertexCount, uint32_t indexCount) {
    vtxs.allocate(vertexCount);
    idxs.allocate(indexCount);
    prjs.allocate(vertexCount);
}

void Mesh3D::free() {
    vtxs.free();
    idxs.free();
    prjs.free();
}

void Mesh3D::uploadVertices(const std::vector<float>& h_x,
                            const std::vector<float>& h_y,
                            const std::vector<float>& h_z,
                            const std::vector<float>& h_nx,
                            const std::vector<float>& h_ny,
                            const std::vector<float>& h_nz,
                            const std::vector<float>& h_u,
                            const std::vector<float>& h_v) 
{
    hipMemcpy(vtxs.x, h_x.data(), numVtxs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vtxs.y, h_y.data(), numVtxs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vtxs.z, h_z.data(), numVtxs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vtxs.nx, h_nx.data(), numVtxs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vtxs.ny, h_ny.data(), numVtxs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vtxs.nz, h_nz.data(), numVtxs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vtxs.u, h_u.data(), numVtxs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vtxs.v, h_v.data(), numVtxs * sizeof(float), hipMemcpyHostToDevice);
}

void Mesh3D::uploadIndices(const std::vector<uint32_t>& h_indices) {
    hipMemcpy(idxs.vertexId, h_indices.data(), numIdxs * sizeof(uint32_t), hipMemcpyHostToDevice);
    setMeshId<<<(numIdxs + 255) / 256, 256>>>(idxs.meshId, numIdxs, meshId);
    setMeshId<<<(numVtxs + 255) / 256, 256>>>(vtxs.meshId, numVtxs, meshId);
}

void Mesh3D::upload(const std::vector<float>& h_x,
                    const std::vector<float>& h_y,
                    const std::vector<float>& h_z,
                    const std::vector<float>& h_nx,
                    const std::vector<float>& h_ny,
                    const std::vector<float>& h_nz,
                    const std::vector<float>& h_u,
                    const std::vector<float>& h_v,
                    const std::vector<uint32_t>& h_indices) {
    uploadVertices(h_x, h_y, h_z, h_nx, h_ny, h_nz, h_u, h_v);
    uploadIndices(h_indices);
}

void Mesh3D::operator+=(const Mesh3D& mesh) {
    uint32_t newVertexCount = numVtxs + mesh.numVtxs;
    uint32_t newIndexCount = numIdxs + mesh.numIdxs;

    vtxs.resize(newVertexCount);
    idxs.resize(newIndexCount);
    prjs.resize(newVertexCount);

    hipMemcpy(vtxs.x + numVtxs, mesh.vtxs.x, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(vtxs.y + numVtxs, mesh.vtxs.y, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(vtxs.z + numVtxs, mesh.vtxs.z, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(vtxs.nx + numVtxs, mesh.vtxs.nx, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(vtxs.ny + numVtxs, mesh.vtxs.ny, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(vtxs.nz + numVtxs, mesh.vtxs.nz, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(vtxs.u + numVtxs, mesh.vtxs.u, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(vtxs.v + numVtxs, mesh.vtxs.v, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);

    hipMemcpy(idxs.vertexId + numIdxs, mesh.idxs.vertexId, mesh.numIdxs * sizeof(uint32_t), hipMemcpyDeviceToDevice);
    hipMemcpy(idxs.meshId + numIdxs, mesh.idxs.meshId, mesh.numIdxs * sizeof(uint32_t), hipMemcpyDeviceToDevice);

    incrementVertexId<<<(mesh.numIdxs + 255) / 256, 256>>>(
        idxs.vertexId + numIdxs, mesh.numIdxs, numVtxs
    );
    setMeshId<<<(mesh.numIdxs + 255) / 256, 256>>>(
        idxs.meshId + numIdxs, mesh.numIdxs, mesh.meshId
    );
    setMeshId<<<(numIdxs + 255) / 256, 256>>>(
        vtxs.meshId + numVtxs, mesh.numVtxs, mesh.meshId
    );
    hipDeviceSynchronize();

    numVtxs = newVertexCount;
    numIdxs = newIndexCount;
}

void Mesh3D::operator=(const Mesh3D& mesh) {
    numVtxs = mesh.numVtxs;
    numIdxs = mesh.numIdxs;

    free();
    allocate(mesh.numVtxs, mesh.numIdxs);

    hipMemcpy(vtxs.x, mesh.vtxs.x, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(vtxs.y, mesh.vtxs.y, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(vtxs.z, mesh.vtxs.z, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(vtxs.nx, mesh.vtxs.nx, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(vtxs.ny, mesh.vtxs.ny, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(vtxs.nz, mesh.vtxs.nz, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(vtxs.u, mesh.vtxs.u, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(vtxs.v, mesh.vtxs.v, mesh.numVtxs * sizeof(float), hipMemcpyDeviceToDevice);

    hipMemcpy(idxs.vertexId, mesh.idxs.vertexId, mesh.numIdxs * sizeof(uint32_t), hipMemcpyDeviceToDevice);
}

// Static transformations
void Mesh3D::translate(Mesh3D &MESH, uint32_t meshId, float dx, float dy, float dz) {
    translateVertices<<<(MESH.numVtxs + 255) / 256, 256>>>(
        MESH.vtxs.x, MESH.vtxs.y, MESH.vtxs.z,
        dx, dy, dz,
        MESH.vtxs.meshId,
        meshId,
        MESH.numVtxs
    );
    hipDeviceSynchronize();
}

void Mesh3D::rotate(Mesh3D &MESH, uint32_t meshId, float ox, float oy, float oz, float wx, float wy, float wz) {
    rotateVertices<<<(MESH.numVtxs + 255) / 256, 256>>>(
        MESH.vtxs.x, MESH.vtxs.y, MESH.vtxs.z,
        ox, oy, oz, wx, wy, wz,
        MESH.vtxs.meshId,
        meshId,
        MESH.numVtxs
    );
    hipDeviceSynchronize();
}

void Mesh3D::scale(Mesh3D &MESH, uint32_t meshId, float ox, float oy, float oz, float sx, float sy, float sz) {
    scaleVertices<<<(MESH.numVtxs + 255) / 256, 256>>>(
        MESH.vtxs.x, MESH.vtxs.y, MESH.vtxs.z,
        ox, oy, oz, sx, sy, sz,
        MESH.vtxs.meshId,
        meshId,
        MESH.numVtxs
    );
    hipDeviceSynchronize();
}

// Transformations
void Mesh3D::translate(float dx, float dy, float dz) {
    Render3D &RENDER = Render3D::instance();
    Mesh3D::translate(RENDER.MESH, meshId, dx, dy, dz);
}
void Mesh3D::rotate(float ox, float oy, float oz, float wx, float wy, float wz) {
    Render3D &RENDER = Render3D::instance();
    Mesh3D::rotate(RENDER.MESH, meshId, ox, oy, oz, wx, wy, wz);
}
void Mesh3D::scale(float ox, float oy, float oz, float sx, float sy, float sz) {
    Render3D &RENDER = Render3D::instance();
    Mesh3D::scale(RENDER.MESH, meshId, ox, oy, oz, sx, sy, sz);
}

void Mesh3D::printVtxs() {
    float *x = new float[numVtxs];
    float *y = new float[numVtxs];
    float *z = new float[numVtxs];
    uint32_t *meshId = new uint32_t[numVtxs];

    hipMemcpy(x, vtxs.x, numVtxs * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(y, vtxs.y, numVtxs * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(z, vtxs.z, numVtxs * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(meshId, vtxs.meshId, numVtxs * sizeof(uint32_t), hipMemcpyDeviceToHost);

    for (uint32_t i = 0; i < numVtxs; i++) {
        std::cout << "Vertex " << i << ": " << x[i] << ", " << y[i] << ", " << z[i] << " (" << meshId[i] << ")" << std::endl;
    }

    delete[] x, y, z, meshId;
}

void Mesh3D::printIdxs() {
    uint32_t *vertexIds = new uint32_t[numIdxs];
    uint32_t *meshIds = new uint32_t[numIdxs];

    hipMemcpy(vertexIds, idxs.vertexId, numIdxs * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(meshIds, idxs.meshId, numIdxs * sizeof(uint32_t), hipMemcpyDeviceToHost);

    for (uint32_t i = 0; i < numIdxs; i++) {
        std::cout << "Index " << i << ": " << vertexIds[i] << " (" << meshIds[i] << ")" << std::endl;
    }

    delete[] vertexIds, meshIds;
}

void Mesh3D::printPrjs() {
    float *x = new float[numVtxs];
    float *y = new float[numVtxs];
    float *z = new float[numVtxs];

    hipMemcpy(x, prjs.x, numVtxs * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(y, prjs.y, numVtxs * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(z, prjs.z, numVtxs * sizeof(float), hipMemcpyDeviceToHost);

    for (uint32_t i = 0; i < numVtxs; i++) {
        std::cout << "Projected " << i << ": (" << x[i] << ", " << y[i] << ", " << z[i] << ")" << std::endl;
    }

    delete[] x, y, z;
}

// KERNELS
__global__ void incrementVertexId(uint32_t* indices, uint32_t numIndices, uint32_t offset) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numIndices) indices[i] += offset;
}
__global__ void setMeshId(uint32_t* ids, uint32_t numIds, uint32_t meshId) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numIds) ids[i] = meshId;
}
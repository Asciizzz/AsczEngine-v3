#include "hip/hip_runtime.h"
#include <Mesh3D.cuh>

// Constructor

Mesh3D::Mesh3D(ULLInt numWs, ULLInt numNs, ULLInt numTs, ULLInt numFs) :
    numWs(numWs), numNs(numNs), numTs(numTs), numFs(numFs)
{
    mallocVertices();
    mallocFaces();
}

Mesh3D::Mesh3D(UInt id, Vecs3f &world, Vecs3f &normal, Vecs2f &texture, Vecs4f &color, Vecs3x3ulli &faces) :
    numWs(world.size()), numNs(normal.size()), numTs(texture.size()), numFs(faces.size())
{
    mallocVertices();
    mallocFaces();
    uploadData(id, world, normal, texture, color, faces);
}

// Memory management

void Mesh3D::mallocVertices() {
    blockNumWs = (numWs + blockSize - 1) / blockSize;
    blockNumNs = (numNs + blockSize - 1) / blockSize;
    blockNumTs = (numTs + blockSize - 1) / blockSize;

    hipMalloc(&world, numWs * sizeof(Vec3f));
    hipMalloc(&normal, numNs * sizeof(Vec3f));
    hipMalloc(&texture, numTs * sizeof(Vec2f));
    hipMalloc(&screen, numWs * sizeof(Vec4f));

    hipMalloc(&color, numWs * sizeof(Vec4f));

    hipMalloc(&wObjId, numWs * sizeof(UInt));
    hipMalloc(&nObjId, numNs * sizeof(UInt));
    hipMalloc(&tObjId, numTs * sizeof(UInt));
}

void Mesh3D::resizeVertices(ULLInt numWs, ULLInt numNs, ULLInt numTs) {
    freeVertices();
    this->numWs = numWs;
    this->numNs = numNs;
    this->numTs = numTs;
    mallocVertices();
}

void Mesh3D::freeVertices() {
    hipFree(world);
    hipFree(normal);
    hipFree(texture);
    hipFree(screen);
    hipFree(color);
    hipFree(wObjId);
    hipFree(nObjId);
    hipFree(tObjId);
}

void Mesh3D::mallocFaces() {
    blockNumFs = (numFs + blockSize - 1) / blockSize;
    hipMalloc(&faces, numFs * sizeof(Vec3x3ulli));
    hipMalloc(&fObjId, numFs * sizeof(UInt));

    // BETA
    hipMalloc(&faceWs, numFs * sizeof(Vec3ulli));
    hipMalloc(&faceNs, numFs * sizeof(Vec3ulli));
    hipMalloc(&faceTs, numFs * sizeof(Vec3ulli));

    // This will be thrown into a kernel
    hipMalloc(&fsVisible, numFs * sizeof(Vec3x3x1ulli));
    if (!numFsVisible) {
        hipMalloc(&numFsVisible, sizeof(ULLInt));
        hipMemset(numFsVisible, 0, sizeof(ULLInt));
    }
}

void Mesh3D::resizeFaces(ULLInt numFs) {
    freeFaces();
    this->numFs = numFs;
    mallocFaces();
}

void Mesh3D::freeFaces() {
    hipFree(faces);
    hipFree(fObjId);

    hipFree(faceWs);
    hipFree(faceNs);
    hipFree(faceTs);

    hipFree(fsVisible);
}

void Mesh3D::free() {
    freeVertices();
    freeFaces();
}

// Upload host data to device

void Mesh3D::uploadData(UInt id, Vecs3f &world, Vecs3f &normal, Vecs2f &texture, Vecs4f &color, Vecs3x3ulli &faces) {
    // Vertices

    // Set obj Id for each vertex attribute
    setObjIdKernel<<<blockNumWs, blockSize>>>(this->wObjId, numWs, id);
    setObjIdKernel<<<blockNumNs, blockSize>>>(this->nObjId, numNs, id);
    setObjIdKernel<<<blockNumTs, blockSize>>>(this->tObjId, numTs, id);

    // Set the actual data
    hipMemcpy(this->world, world.data(), world.size() * sizeof(Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(this->normal, normal.data(), normal.size() * sizeof(Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(this->texture, texture.data(), texture.size() * sizeof(Vec2f), hipMemcpyHostToDevice);
    
    hipMemcpy(this->color, color.data(), color.size() * sizeof(Vec4f), hipMemcpyHostToDevice);
    
    // Faces indices
    // Set obj Id for each face
    setObjIdKernel<<<blockNumFs, blockSize>>>(this->fObjId, numFs, id);
    hipMemcpy(this->faces, faces.data(), faces.size() * sizeof(Vec3x3ulli), hipMemcpyHostToDevice);
}

// Mesh operators

void Mesh3D::operator+=(Mesh3D &mesh) {
    // Resize vertices
    ULLInt newNumWs = numWs + mesh.numWs;
    ULLInt newNumNs = numNs + mesh.numNs;
    ULLInt newNumTs = numTs + mesh.numTs;
    Vec3f *newWorld;
    Vec3f *newNormal;
    Vec2f *newTexture;
    Vec4f *newScreen;
    UInt *newWObjId;
    UInt *newNObjId;
    UInt *newTObjId;
    Vec4f *newColor;
    hipMalloc(&newWorld, newNumWs * sizeof(Vec3f));
    hipMalloc(&newNormal, newNumNs * sizeof(Vec3f));
    hipMalloc(&newTexture, newNumTs * sizeof(Vec2f));
    hipMalloc(&newScreen, newNumWs * sizeof(Vec4f));
    hipMalloc(&newWObjId, newNumWs * sizeof(UInt));
    hipMalloc(&newNObjId, newNumNs * sizeof(UInt));
    hipMalloc(&newTObjId, newNumTs * sizeof(UInt));
    hipMalloc(&newColor, newNumWs * sizeof(Vec4f));
    // Copy old data
    hipMemcpy(newWorld, world, numWs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newNormal, normal, numNs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newTexture, texture, numTs * sizeof(Vec2f), hipMemcpyDeviceToDevice);
    hipMemcpy(newScreen, screen, numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    hipMemcpy(newWObjId, wObjId, numWs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newNObjId, nObjId, numNs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newTObjId, tObjId, numTs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newColor, color, numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    // Copy new data
    hipMemcpy(newWorld + numWs, mesh.world, mesh.numWs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newNormal + numNs, mesh.normal, mesh.numNs * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    hipMemcpy(newTexture + numTs, mesh.texture, mesh.numTs * sizeof(Vec2f), hipMemcpyDeviceToDevice);
    hipMemcpy(newScreen + numWs, mesh.screen, mesh.numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    hipMemcpy(newWObjId + numWs, mesh.wObjId, mesh.numWs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newNObjId + numNs, mesh.nObjId, mesh.numNs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newTObjId + numTs, mesh.tObjId, mesh.numTs * sizeof(UInt), hipMemcpyDeviceToDevice);
    hipMemcpy(newColor + numWs, mesh.color, mesh.numWs * sizeof(Vec4f), hipMemcpyDeviceToDevice);
    // Free old data
    freeVertices();
    // Update vertices
    world = newWorld;
    normal = newNormal;
    texture = newTexture;
    screen = newScreen;
    wObjId = newWObjId;
    nObjId = newNObjId;
    tObjId = newTObjId;
    color = newColor;

    // Resize faces (with offset for the added vertices)
    ULLInt newNumFs = numFs + mesh.numFs;
    ULLInt newBlockNumFs = (newNumFs + blockSize - 1) / blockSize;

    Vec3x3ulli *newFaces;
    UInt *newFObjId;

    Vec3ulli *newFaceWs;
    Vec3ulli *newFaceNs;
    Vec3ulli *newFaceTs;
    
    hipMalloc(&newFaces, newNumFs * sizeof(Vec3x3ulli));
    hipMalloc(&newFObjId, newNumFs * sizeof(UInt));

    hipMalloc(&newFaceWs, newNumFs * sizeof(Vec3ulli));
    hipMalloc(&newFaceNs, newNumFs * sizeof(Vec3ulli));
    hipMalloc(&newFaceTs, newNumFs * sizeof(Vec3ulli));

    hipMemcpy(newFaces, faces, numFs * sizeof(Vec3x3ulli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFObjId, fObjId, numFs * sizeof(UInt), hipMemcpyDeviceToDevice);

    hipMemcpy(newFaceWs, faceWs, numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceNs, faceNs, numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceTs, faceTs, numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);

    hipMemcpy(newFaces + numFs, mesh.faces, mesh.numFs * sizeof(Vec3x3ulli), hipMemcpyDeviceToDevice);
    incrementFacesIdxKernel<<<newBlockNumFs, blockSize>>>(newFaces, numWs, numNs, numTs, numFs, newNumFs);
    hipMemcpy(newFObjId + numFs, mesh.fObjId, mesh.numFs * sizeof(UInt), hipMemcpyDeviceToDevice);

    hipMemcpy(newFaceWs + numFs, mesh.faceWs, mesh.numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceNs + numFs, mesh.faceNs, mesh.numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFaceTs + numFs, mesh.faceTs, mesh.numFs * sizeof(Vec3ulli), hipMemcpyDeviceToDevice);
    incrementFaceIdxKernel<<<newBlockNumFs, blockSize>>>(newFaceWs, numWs, numFs, newNumFs);
    incrementFaceIdxKernel<<<newBlockNumFs, blockSize>>>(newFaceNs, numNs, numFs, newNumFs);
    incrementFaceIdxKernel<<<newBlockNumFs, blockSize>>>(newFaceTs, numTs, numFs, newNumFs);

    // Still a WIP so we are going to be extra careful
    Vec3x3x1ulli *newFacesVisible;
    hipMalloc(&newFacesVisible, newNumFs * sizeof(Vec3x3x1ulli));
    hipMemcpy(newFacesVisible, fsVisible, numFs * sizeof(Vec3x3x1ulli), hipMemcpyDeviceToDevice);
    hipMemcpy(newFacesVisible + numFs, mesh.fsVisible, mesh.numFs * sizeof(Vec3x3x1ulli), hipMemcpyDeviceToDevice);

    freeFaces();

    faces = newFaces;
    fObjId = newFObjId;

    faceWs = newFaceWs;
    faceNs = newFaceNs;
    faceTs = newFaceTs;
    
    fsVisible = newFacesVisible;

    // Update number of vertices and faces
    numWs = newNumWs;
    numNs = newNumNs;
    numTs = newNumTs;
    numFs = newNumFs;
    blockNumWs = (numWs + blockSize - 1) / blockSize;
    blockNumNs = (numNs + blockSize - 1) / blockSize;
    blockNumTs = (numTs + blockSize - 1) / blockSize;
    blockNumFs = (numFs + blockSize - 1) / blockSize;
}

// Transformations (with obj Id)

void Mesh3D::translate(UInt objID, Vec3f t) {
    translateWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, false, numWs, objID, t);
}
void Mesh3D::rotate(UInt objID, Vec3f origin, Vec3f rot) {
    rotateWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, false, numWs, objID, origin, rot);
    rotateNormalKernel<<<blockNumNs, blockSize>>>(normal, nObjId, false, numNs, objID, origin, rot);
}
void Mesh3D::scale(UInt objID, Vec3f origin, Vec3f scl) {
    scaleWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, false, numWs, objID, origin, scl);
    scaleNormalKernel<<<blockNumNs, blockSize>>>(normal, nObjId, false, numNs, objID, origin, scl);
}

// Transformations (all obj Ids)

void Mesh3D::translate(Vec3f t) {
    translateWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, true, numWs, 0, t);
}
void Mesh3D::rotate(Vec3f origin, Vec3f rot) {
    rotateWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, true, numWs, 0, origin, rot);
    rotateNormalKernel<<<blockNumNs, blockSize>>>(normal, nObjId, true, numNs, 0, origin, rot);
}
void Mesh3D::scale(Vec3f origin, Vec3f scl) {
    scaleWorldKernel<<<blockNumWs, blockSize>>>(world, wObjId, true, numWs, 0, origin, scl);
    scaleNormalKernel<<<blockNumNs, blockSize>>>(normal, nObjId, true, numNs, 0, origin, scl);
}

// Kernel for preparing vertices
__global__ void incrementFacesIdxKernel(Vec3x3ulli *faces, ULLInt offsetW, ULLInt offsetN, ULLInt offsetT, ULLInt numFs, ULLInt newNumFs) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < newNumFs && idx >= numFs) {
        faces[idx].v += offsetW;
        faces[idx].n += offsetN;
        faces[idx].t += offsetT;
    }
}

__global__ void incrementFaceIdxKernel(Vec3ulli *faces, ULLInt offset, ULLInt numFs, ULLInt newNumFs) { // BETA
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < newNumFs && idx >= numFs) faces[idx] += offset;
}

__global__ void setObjIdKernel(UInt *objId, ULLInt numWs, UInt id) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWs) objId[idx] = id;
}

// Kernel for transforming vertices
__global__ void translateWorldKernel(Vec3f *world, UInt *wObjId, bool allId, ULLInt numWs, UInt objID, Vec3f t) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numWs || (!allId && wObjId[idx] != objID)) return;

    world[idx].translate(t);
}
__global__ void rotateWorldKernel(Vec3f *world, UInt *wObjId, bool allId, ULLInt numWs, UInt objID, Vec3f origin, Vec3f rot) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numWs || (!allId && wObjId[idx] != objID)) return;

    world[idx].rotate(origin, rot);
}
__global__ void scaleWorldKernel(Vec3f *world, UInt *wObjId, bool allId, ULLInt numWs, UInt objID, Vec3f origin, Vec3f scl) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numWs || (!allId && wObjId[idx] != objID)) return;

    world[idx].scale(origin, scl);
}

// Rotate and scale normals
__global__ void rotateNormalKernel(Vec3f *normal, UInt *nObjId, bool allID, ULLInt numNs, UInt objID, Vec3f origin, Vec3f rot) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numNs || (!allID && nObjId[idx] != objID)) return;

    normal[idx].rotate(origin, rot);
}
__global__ void scaleNormalKernel(Vec3f *normal, UInt *nObjId, bool allID, ULLInt numNs, UInt objID, Vec3f origin, Vec3f scl) {
    ULLInt idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numNs || (!allID && nObjId[idx] != objID)) return;

    normal[idx].scale(origin, scl);
}
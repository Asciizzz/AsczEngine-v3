#include "hip/hip_runtime.h"
#include <Render3D.cuh>

void Render3D::setResolution(float w, float h) {
    res = {w, h};
    res_half = {w / 2, h / 2};
    camera.setResolution(w, h);
    buffer.resize(w, h, pixel_size);
}

void Render3D::allocateProjection() {
    hipMalloc(&projection, mesh.numVs * sizeof(Vec4f));
}
void Render3D::freeProjection() {
    if (projection) hipFree(projection);
}
void Render3D::resizeProjection() {
    freeProjection();
    allocateProjection();
}

// Pipeline

void Render3D::vertexProjection() {
    vertexProjectionKernel<<<mesh.blockNumVs, mesh.blockSize>>>(
        projection, mesh.world, camera, mesh.numVs
    );
    hipDeviceSynchronize();
}

// Kernels

__global__ void vertexProjectionKernel(Vec4f *projection, Vec3f *world, Camera3D camera, ULLInt numVs) {
    ULLInt i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numVs) return;

    Vec4f v4 = world[i].toVec4f();
    Vec4f t4 = camera.mvp * v4;
    Vec3f t3 = t4.toVec3f();

    // Screen space 
    t3.x = (t3.x + 1) * camera.res.x / 2;
    t3.y = (1 - t3.y) * camera.res.y / 2;
    t3.z = camera.near + (camera.far - camera.near) * t3.z;

    Vec4f p = t3.toVec4f();
    p.w = camera.isInsideFrustum(world[i]) ? 1 : 0;

    projection[i] = p;
}
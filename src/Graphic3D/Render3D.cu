#include <Render3D.cuh>

void Render3D::setResolution(float w, float h) {
    res = {w, h};
    res_half = {w / 2, h / 2};
    camera.setResolution(w, h);
    buffer.resize(w, h, pixel_size);
}
#include <Graphic3D.cuh>

#include <SFML/Graphics.hpp>
// Face3D

void Face3D::malloc(ULLInt size) {
    this->size = size;
    hipMalloc(&sx, sizeof(float) * size);
    hipMalloc(&sy, sizeof(float) * size);
    hipMalloc(&sz, sizeof(float) * size);
    hipMalloc(&sw, sizeof(float) * size);
    hipMalloc(&wx, sizeof(float) * size);
    hipMalloc(&wy, sizeof(float) * size);
    hipMalloc(&wz, sizeof(float) * size);
    hipMalloc(&tu, sizeof(float) * size);
    hipMalloc(&tv, sizeof(float) * size);
    hipMalloc(&nx, sizeof(float) * size);
    hipMalloc(&ny, sizeof(float) * size);
    hipMalloc(&nz, sizeof(float) * size);
    hipMalloc(&active, sizeof(bool) * size / 3);
    hipMalloc(&mat, sizeof(LLInt) * size / 3);
    hipMalloc(&area, sizeof(float) * size / 3);
}
void Face3D::free() {
    size = 0;

    if (sx) hipFree(sx); if (sy) hipFree(sy); if (sz) hipFree(sz); if (sw) hipFree(sw);
    if (wx) hipFree(wx); if (wy) hipFree(wy); if (wz) hipFree(wz);
    if (tu) hipFree(tu); if (tv) hipFree(tv);
    if (nx) hipFree(nx); if (ny) hipFree(ny); if (nz) hipFree(nz);

    if (active) hipFree(active);
    if (mat) hipFree(mat);
    if (area) hipFree(area);
}

// Graphic stuff below

void Graphic3D::setResolution(float w, float h, float ps) {
    pixelSize = ps;
    res = {w, h};
    res_half = {w / 2, h / 2};
    camera.aspect = w / h;
    buffer.resize(w, h, pixelSize);
}

// Free everything
void Graphic3D::free() {
    mesh.free();
    buffer.free();
    freeRuntimeFaces();
    destroyRuntimeStreams();

    freeShadowMap();
}

// Graphic faces (runtime)
void Graphic3D::mallocRuntimeFaces() {
    rtFaces.malloc(mesh.f.size * 4);

    hipMalloc(&d_rtCount1, sizeof(ULLInt));
    hipMalloc(&d_rtCount2, sizeof(ULLInt));
    hipMalloc(&rtIndex1, sizeof(ULLInt) * rtFaces.size / 3);
    hipMalloc(&rtIndex2, sizeof(ULLInt) * rtFaces.size / 3);
}
void Graphic3D::freeRuntimeFaces() {
    rtFaces.free();

    if (d_rtCount1) hipFree(d_rtCount1);
    if (d_rtCount2) hipFree(d_rtCount2);
    if (rtIndex1) hipFree(rtIndex1);
    if (rtIndex2) hipFree(rtIndex2);
}
void Graphic3D::resizeRuntimeFaces() {
    freeRuntimeFaces();
    mallocRuntimeFaces();
}

void Graphic3D::createRuntimeStreams() {
    for (int i = 0; i < 4; i++) {
        hipStreamCreate(&rtStreams[i]);
    }
}
void Graphic3D::destroyRuntimeStreams() {
    for (int i = 0; i < 4; i++) {
        hipStreamDestroy(rtStreams[i]);
    }
}

// =========================================================================
// ============================= BETAs SECTION =============================
// =========================================================================

// Beta: Shadow mapping
void Graphic3D::createShadowMap(int w, int h, int tw, int th) {
    shdwWidth = w;
    shdwHeight = h;
    shdwTileSizeX = tw;
    shdwTileSizeY = th;
    shdwTileNumX = w / tw;
    shdwTileNumY = h / th;
    shdwTileNum = shdwTileNumX * shdwTileNumY;

    hipMalloc(&shadowDepth, sizeof(float) * w * h);
}

void Graphic3D::freeShadowMap() {
    if (shadowDepth) hipFree(shadowDepth);
}
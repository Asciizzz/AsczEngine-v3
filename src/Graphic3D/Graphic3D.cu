#include <Graphic3D.cuh>

#include <SFML/Graphics.hpp>
// Face3D

void Face3D::malloc(ULLInt size) {
    hipMalloc(&wx, sizeof(float) * size);
    hipMalloc(&wy, sizeof(float) * size);
    hipMalloc(&wz, sizeof(float) * size);
    hipMalloc(&nx, sizeof(float) * size);
    hipMalloc(&ny, sizeof(float) * size);
    hipMalloc(&nz, sizeof(float) * size);
    hipMalloc(&tu, sizeof(float) * size);
    hipMalloc(&tv, sizeof(float) * size);
    hipMalloc(&cr, sizeof(float) * size);
    hipMalloc(&cg, sizeof(float) * size);
    hipMalloc(&cb, sizeof(float) * size);
    hipMalloc(&ca, sizeof(float) * size);
    hipMalloc(&sx, sizeof(float) * size);
    hipMalloc(&sy, sizeof(float) * size);
    hipMalloc(&sz, sizeof(float) * size);
    hipMalloc(&sw, sizeof(float) * size);
}
void Face3D::free() {
    if (wx) hipFree(wx); if (wy) hipFree(wy); if (wz) hipFree(wz);
    if (nx) hipFree(nx); if (ny) hipFree(ny); if (nz) hipFree(nz);
    if (tu) hipFree(tu); if (tv) hipFree(tv);
    if (cr) hipFree(cr); if (cg) hipFree(cg); if (cb) hipFree(cb); if (ca) hipFree(ca);
    if (sx) hipFree(sx); if (sy) hipFree(sy); if (sz) hipFree(sz); if (sw) hipFree(sw);
}

// Graphic stuff below

void Graphic3D::setResolution(float w, float h, float ps) {
    pixelSize = ps;
    res = {w, h};
    res_half = {w / 2, h / 2};
    camera.aspect = w / h;
    buffer.resize(w, h, pixelSize);
}

void Graphic3D::setTileSize(int tw, int th) {
    tileSizeX = tw;
    tileSizeY = th;

    // Buffer W/H must be divisible by tile W/H, otherwise throw an error
    // It's a bit forceful, but it's better to have a consistent tile size
    // Otherwise the entire tile-based rasterization will be broken
    // Trust me, I've been there
    if (buffer.width % tileSizeX != 0 || buffer.height % tileSizeY != 0) {
        std::cerr << "Buffer W/H must be divisible by tile W/H" << std::endl;
        exit(1);
    }

    tileNumX = buffer.width / tileSizeX;
    tileNumY = buffer.height / tileSizeY;
    tileNum = tileNumX * tileNumY;
}

// Free everything
void Graphic3D::free() {
    mesh.free();
    buffer.free();
    freeRuntimeFaces();

    freeTexture();
    freeShadowMap();
}

// Graphic faces (runtime)
void Graphic3D::mallocRuntimeFaces() {
    hipMalloc(&d_rtCount, sizeof(ULLInt));
    rtFaces.malloc(mesh.faces.size * 12);
}
void Graphic3D::freeRuntimeFaces() {
    if (d_rtCount) hipFree(d_rtCount);
    rtFaces.free();
}
void Graphic3D::resizeRuntimeFaces() {
    freeRuntimeFaces();
    mallocRuntimeFaces();
}

// =========================================================================
// ============================= BETAs SECTION =============================
// =========================================================================

// BETA: Texture mapping
void Graphic3D::createTexture(const std::string &path) {
    sf::Image textureImage;
    if (!textureImage.loadFromFile(path)) {
        throw std::runtime_error("Failed to load texture image.");
    }

    textureWidth = textureImage.getSize().x;
    textureHeight = textureImage.getSize().y;

    std::vector<Vec3f> texture(textureWidth * textureHeight);

    for (int y = 0; y < textureHeight; y++) {
        for (int x = 0; x < textureWidth; x++) {
            sf::Color color = textureImage.getPixel(x, y);
            int idx = x + y * textureWidth;
            texture[idx] = {float(color.r), float(color.g), float(color.b)};
        }
    }

    if (textureSet) freeTexture();
    else textureSet = true;

    hipMalloc(&d_texture, sizeof(Vec3f) * texture.size());
    hipMemcpy(d_texture, texture.data(), sizeof(Vec3f) * texture.size(), hipMemcpyHostToDevice);
}

void Graphic3D::freeTexture() {
    if (d_texture) hipFree(d_texture);
}

// Beta: Shadow mapping
void Graphic3D::createShadowMap(int w, int h, int tw, int th) {
    shdwWidth = w;
    shdwHeight = h;
    shdwTileSizeX = tw;
    shdwTileSizeY = th;
    shdwTileNumX = w / tw;
    shdwTileNumY = h / th;
    shdwTileNum = shdwTileNumX * shdwTileNumY;

    hipMalloc(&shadowDepth, sizeof(float) * w * h);
}

void Graphic3D::freeShadowMap() {
    if (shadowDepth) hipFree(shadowDepth);
}
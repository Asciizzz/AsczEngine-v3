#include <Graphic3D.cuh>

#include <SFML/Graphics.hpp>
// Face3D

void Face3D::malloc(ULLInt size) {
    this->size = size;
    hipMalloc(&wx, sizeof(float) * size);
    hipMalloc(&wy, sizeof(float) * size);
    hipMalloc(&wz, sizeof(float) * size);
    hipMalloc(&nx, sizeof(float) * size);
    hipMalloc(&ny, sizeof(float) * size);
    hipMalloc(&nz, sizeof(float) * size);
    hipMalloc(&tu, sizeof(float) * size);
    hipMalloc(&tv, sizeof(float) * size);
    hipMalloc(&cr, sizeof(float) * size);
    hipMalloc(&cg, sizeof(float) * size);
    hipMalloc(&cb, sizeof(float) * size);
    hipMalloc(&ca, sizeof(float) * size);
    hipMalloc(&sx, sizeof(float) * size);
    hipMalloc(&sy, sizeof(float) * size);
    hipMalloc(&sz, sizeof(float) * size);
    hipMalloc(&sw, sizeof(float) * size);
    hipMalloc(&area, sizeof(float) * size / 3);
    hipMalloc(&active, sizeof(bool) * size / 3);
}
void Face3D::free() {
    size = 0;
    if (wx) hipFree(wx); if (wy) hipFree(wy); if (wz) hipFree(wz);
    if (nx) hipFree(nx); if (ny) hipFree(ny); if (nz) hipFree(nz);
    if (tu) hipFree(tu); if (tv) hipFree(tv);
    if (cr) hipFree(cr); if (cg) hipFree(cg); if (cb) hipFree(cb); if (ca) hipFree(ca);
    if (sx) hipFree(sx); if (sy) hipFree(sy); if (sz) hipFree(sz); if (sw) hipFree(sw);
    if (area) hipFree(area);
    if (active) hipFree(active);
}

// Graphic stuff below

void Graphic3D::setResolution(float w, float h, float ps) {
    pixelSize = ps;
    res = {w, h};
    res_half = {w / 2, h / 2};
    camera.aspect = w / h;
    buffer.resize(w, h, pixelSize);
}

// Free everything
void Graphic3D::free() {
    mesh.free();
    buffer.free();
    freeRuntimeFaces();
    destroyRuntimeStreams();

    freeTexture();
    freeShadowMap();
}

// Graphic faces (runtime)
void Graphic3D::mallocRuntimeFaces() {
    rtFaces.malloc(mesh.faces.size * 4);

    hipMalloc(&d_rtCount1, sizeof(ULLInt));
    hipMalloc(&d_rtCount2, sizeof(ULLInt));
    hipMalloc(&d_rtCount3, sizeof(ULLInt));
    hipMalloc(&d_rtCount4, sizeof(ULLInt));
    hipMalloc(&rtIndex2, sizeof(ULLInt) * rtFaces.size / 3);
    hipMalloc(&rtIndex3, sizeof(ULLInt) * rtFaces.size / 3);
    hipMalloc(&rtIndex4, sizeof(ULLInt) * rtFaces.size / 3);
    hipMalloc(&rtIndex1, sizeof(ULLInt) * rtFaces.size / 3);
}
void Graphic3D::freeRuntimeFaces() {
    rtFaces.free();

    if (d_rtCount1) hipFree(d_rtCount1);
    if (d_rtCount2) hipFree(d_rtCount2);
    if (d_rtCount3) hipFree(d_rtCount3);
    if (d_rtCount4) hipFree(d_rtCount4);
    if (rtIndex1) hipFree(rtIndex1);
    if (rtIndex2) hipFree(rtIndex2);
    if (rtIndex3) hipFree(rtIndex3);
    if (rtIndex4) hipFree(rtIndex4);
}
void Graphic3D::resizeRuntimeFaces() {
    freeRuntimeFaces();
    mallocRuntimeFaces();
}

void Graphic3D::createRuntimeStreams() {
    for (int i = 0; i < 4; i++) {
        hipStreamCreate(&rtStreams[i]);
    }
}
void Graphic3D::destroyRuntimeStreams() {
    for (int i = 0; i < 4; i++) {
        hipStreamDestroy(rtStreams[i]);
    }
}

// =========================================================================
// ============================= BETAs SECTION =============================
// =========================================================================

// BETA: Texture mapping
void Graphic3D::createTexture(const std::string &path) {
    sf::Image textureImage;
    if (!textureImage.loadFromFile(path)) {
        throw std::runtime_error("Failed to load texture image.");
    }

    textureWidth = textureImage.getSize().x;
    textureHeight = textureImage.getSize().y;

    std::vector<Vec3f> texture(textureWidth * textureHeight);

    for (int y = 0; y < textureHeight; y++) {
        for (int x = 0; x < textureWidth; x++) {
            sf::Color color = textureImage.getPixel(x, y);
            int idx = x + y * textureWidth;
            texture[idx] = {float(color.r), float(color.g), float(color.b)};
        }
    }

    if (textureSet) freeTexture();
    else textureSet = true;

    hipMalloc(&d_texture, sizeof(Vec3f) * texture.size());
    hipMemcpy(d_texture, texture.data(), sizeof(Vec3f) * texture.size(), hipMemcpyHostToDevice);
}

void Graphic3D::freeTexture() {
    if (d_texture) hipFree(d_texture);
}

// Beta: Shadow mapping
void Graphic3D::createShadowMap(int w, int h, int tw, int th) {
    shdwWidth = w;
    shdwHeight = h;
    shdwTileSizeX = tw;
    shdwTileSizeY = th;
    shdwTileNumX = w / tw;
    shdwTileNumY = h / th;
    shdwTileNum = shdwTileNumX * shdwTileNumY;

    hipMalloc(&shadowDepth, sizeof(float) * w * h);
}

void Graphic3D::freeShadowMap() {
    if (shadowDepth) hipFree(shadowDepth);
}
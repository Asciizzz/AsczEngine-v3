#include "hip/hip_runtime.h"
#include <Graphic3D.cuh>

#include <SFML/Graphics.hpp>
// Face3D

void Face3D::malloc(ULLInt size) {
    hipMalloc(&wx, sizeof(float) * size);
    hipMalloc(&wy, sizeof(float) * size);
    hipMalloc(&wz, sizeof(float) * size);
    hipMalloc(&nx, sizeof(float) * size);
    hipMalloc(&ny, sizeof(float) * size);
    hipMalloc(&nz, sizeof(float) * size);
    hipMalloc(&tu, sizeof(float) * size);
    hipMalloc(&tv, sizeof(float) * size);
    hipMalloc(&cr, sizeof(float) * size);
    hipMalloc(&cg, sizeof(float) * size);
    hipMalloc(&cb, sizeof(float) * size);
    hipMalloc(&ca, sizeof(float) * size);
    hipMalloc(&sx, sizeof(float) * size);
    hipMalloc(&sy, sizeof(float) * size);
    hipMalloc(&sz, sizeof(float) * size);
    hipMalloc(&sw, sizeof(float) * size);
}
void Face3D::free() {
    if (wx) hipFree(wx); if (wy) hipFree(wy); if (wz) hipFree(wz);
    if (nx) hipFree(nx); if (ny) hipFree(ny); if (nz) hipFree(nz);
    if (tu) hipFree(tu); if (tv) hipFree(tv);
    if (cr) hipFree(cr); if (cg) hipFree(cg); if (cb) hipFree(cb); if (ca) hipFree(ca);
    if (sx) hipFree(sx); if (sy) hipFree(sy); if (sz) hipFree(sz); if (sw) hipFree(sw);
}

// Graphic stuff below

void Graphic3D::setResolution(float w, float h, float ps) {
    pixelSize = ps;
    res = {w, h};
    res_half = {w / 2, h / 2};
    camera.aspect = w / h;
    buffer.resize(w, h, pixelSize);
}

void Graphic3D::setTileSize(int tw, int th) {
    tileWidth = tw;
    tileHeight = th;

    // Buffer W/H must be divisible by tile W/H, otherwise throw an error
    // It's a bit forceful, but it's better to have a consistent tile size
    // Otherwise the entire tile-based rasterization will be broken
    // Trust me, I've been there
    if (buffer.width % tileWidth != 0 || buffer.height % tileHeight != 0) {
        std::cerr << "Buffer W/H must be divisible by tile W/H" << std::endl;
        exit(1);
    }

    tileNumX = buffer.width / tileWidth;
    tileNumY = buffer.height / tileHeight;
    tileNum = tileNumX * tileNumY;
}

// Free everything
void Graphic3D::free() {
    mesh.free();
    buffer.free();
    freeRuntimeFaces();
    freeFaceStreams();

    freeTexture();
}

// Graphic faces (runtime)
void Graphic3D::mallocRuntimeFaces() {
    hipMalloc(&d_faceCounter, sizeof(ULLInt));
    // In the worst case scenario, each face when culled can be split into 4 faces
    rtFaces.malloc(mesh.faces.size * 4);
}
void Graphic3D::freeRuntimeFaces() {
    if (d_faceCounter) hipFree(d_faceCounter);
    rtFaces.free();
}
void Graphic3D::resizeRuntimeFaces() {
    freeRuntimeFaces();
    mallocRuntimeFaces();
}

// Face stream for chunking very large number of faces
void Graphic3D::mallocFaceStreams() {
    chunkNum = (mesh.faces.size + chunkSize - 1) / chunkSize;

    // Stream for asynchronous execution (very helpful)
    faceStreams = (hipStream_t*)malloc(chunkNum * sizeof(hipStream_t));
    for (int i = 0; i < chunkNum; i++) {
        hipStreamCreate(&faceStreams[i]);
    }
}
void Graphic3D::freeFaceStreams() {
    // for (int i = 0; i < chunkSize; i++) {
    //     if (faceStreams) hipStreamDestroy(faceStreams[i]);
    // }
    // if (faceStreams) delete[] faceStreams;
}
void Graphic3D::resizeFaceStreams() {
    int newChunkNum = (mesh.faces.size + chunkSize - 1) / chunkSize;
    if (newChunkNum == chunkNum) return;

    freeFaceStreams();
    mallocFaceStreams();
}

// Atomic functions
__device__ bool atomicMinFloat(float* addr, float value) {
    int* addr_as_int = (int*)addr;
    int old = *addr_as_int, assumed;

    do {
        assumed = old;
        old = atomicCAS(addr_as_int, assumed, __float_as_int(fminf(value, __int_as_float(assumed))));
    } while (assumed != old);

    return __int_as_float(old) > value;
}

// BETA: Texture mapping
void Graphic3D::createTexture(const std::string &path) {
    sf::Image textureImage;
    if (!textureImage.loadFromFile(path)) {
        throw std::runtime_error("Failed to load texture image.");
    }

    textureWidth = textureImage.getSize().x;
    textureHeight = textureImage.getSize().y;

    std::vector<Vec3f> texture(textureWidth * textureHeight);

    for (int y = 0; y < textureHeight; y++) {
        for (int x = 0; x < textureWidth; x++) {
            sf::Color color = textureImage.getPixel(x, y);
            int idx = x + y * textureWidth;
            texture[idx] = {float(color.r), float(color.g), float(color.b)};
        }
    }

    if (textureSet) freeTexture();
    else textureSet = true;

    hipMalloc(&d_texture, sizeof(Vec3f) * texture.size());
    hipMemcpy(d_texture, texture.data(), sizeof(Vec3f) * texture.size(), hipMemcpyHostToDevice);
}

void Graphic3D::freeTexture() {
    if (d_texture) hipFree(d_texture);
}
#include "hip/hip_runtime.h"
#include <Camera3D.cuh>

void Camera3D::restrictRot() {
    if (rot.x <= -M_PI_2) rot.x = -M_PI_2 + 0.001;
    else if (rot.x >= M_PI_2) rot.x = M_PI_2 - 0.001;

    if (rot.y > M_2_PI) rot.y -= M_2_PI;
    else if (rot.y < 0) rot.y += M_2_PI;
}

void Camera3D::updateView() {
    forward.x = sin(rot.y) * cos(rot.x);
    forward.y = sin(rot.x);
    forward.z = cos(rot.y) * cos(rot.x);
    forward.norm();

    right = forward & Vec3f(0, 1, 0);
    right.norm();

    up = right & forward;
    up.norm();

    // Translation matrix
    float tMat[4][4] = {
        {1, 0, 0, -pos.x},
        {0, 1, 0, -pos.y},
        {0, 0, 1, -pos.z},
        {0, 0, 0, 1}
    };

    // Rotation matrix
    float rMat[4][4] = {
        {right.x, right.y, right.z, 0},
        {up.x, up.y, up.z, 0},
        {-forward.x, -forward.y, -forward.z, 0},
        {0, 0, 0, 1}
    };

    view = Mat4f(rMat) * Mat4f(tMat);
}

void Camera3D::updateProjection() {
    float f = 1 / tan(fov / 2);
    float ar = aspect;

    float p22 = (far + near) / (near - far);
    float p23 = (2 * far * near) / (near - far);

    float pMat[4][4] = {
        {f / ar, 0, 0, 0},
        {0, f, 0, 0},
        {0, 0, p22, p23},
        {0, 0, -1, 0}
    };
    projection = Mat4f(pMat);
}

void Camera3D::updateMVP() {
    mvp = projection * view;
}

void Camera3D::update() {
    restrictRot();

    // Model-View-Projection matrix
    updateView();
    updateProjection();
    updateMVP();

    // Camera plane
    updatePlane();
}

void Camera3D::updatePlane() {
    nearPlane = Plane3D(forward, pos + forward * near);
    farPlane = Plane3D(forward * -1, pos + forward * far);

    /*
    The right normal is basically just the forward vector rotated by fov/2 + 90 degrees.
    */

    Vec3f rightNormal = Vec3f::rotate(forward, Vec3f(), Vec3f(0, fov / 2 + M_PI_2, 0));
    rightNormal.y = 0;
    rightNormal.norm();
    rightNormal *= -1; // Facing inwards the frustum
    rightPlane = Plane3D(rightNormal, pos);

    Vec3f leftNormal = Vec3f::rotate(forward, Vec3f(), Vec3f(0, -fov / 2 - M_PI_2, 0));
    leftNormal.y = 0;
    leftNormal.norm();
    leftNormal *= -1; // Facing inwards the frustum
    leftPlane = Plane3D(leftNormal, pos);

    // Get the vertical fov
    float vFov = 2 * atan(tan(fov / 2) / aspect);

    float upAngle = vFov + M_PI_2;
    float downAngle = -vFov - M_PI_2;

    float cosU = cos(upAngle);
    float sinU = sin(upAngle);
    float cosD = cos(downAngle);
    float sinD = sin(downAngle);

    Vec3f upNormal = up * cosU + (right & up) * sinU + right * (right * up) * (1 - cosU);
    upNormal.norm();
    upPlane = Plane3D(upNormal, pos);

    Vec3f downNormal = up * cosD + (right & up) * sinD + right * (right * up) * (1 - cosD);
    downNormal.norm();
    downPlane = Plane3D(downNormal, pos);
}

// Debug
std::string Camera3D::data() {
    std::string str = "Camera3D\n";
    str += "| Pos: " + std::to_string(pos.x) + ", " + std::to_string(pos.y) + ", " + std::to_string(pos.z) + "\n";
    str += "| Rot: " + std::to_string(rot.x) + ", " + std::to_string(rot.y) + ", " + std::to_string(rot.z) + "\n";
    str += "| Fd: " + std::to_string(forward.x) + ", " + std::to_string(forward.y) + ", " + std::to_string(forward.z) + "\n";
    str += "| Rg: " + std::to_string(right.x) + ", " + std::to_string(right.y) + ", " + std::to_string(right.z) + "\n";
    str += "| Up: " + std::to_string(up.x) + ", " + std::to_string(up.y) + ", " + std::to_string(up.z) + "\n";
    str += "| Fov: " + std::to_string(fov * 180 / M_PI) + "\n";
    str += "| Asp: " + std::to_string(aspect) + "\n";
    str += "| Nrr: " + std::to_string(near) + "\n";
    str += "| Far: " + std::to_string(far) + "\n";
    return str;
}
#include "hip/hip_runtime.h"
#include <Camera3D.cuh>

void Camera3D::restrictRot() {
    if (rot.x <= -M_PI_2) rot.x = -M_PI_2 + 0.001;
    else if (rot.x >= M_PI_2) rot.x = M_PI_2 - 0.001;

    if (rot.y > M_2_PI) rot.y -= M_2_PI;
    else if (rot.y < 0) rot.y += M_2_PI;
}

void Camera3D::updateView() {
    forward.x = sin(rot.y) * cos(rot.x);
    forward.y = sin(rot.x);
    forward.z = cos(rot.y) * cos(rot.x);
    forward.norm();

    right = forward & Vec3f(0, 1, 0);
    right.norm();

    up = right & forward;
    up.norm();

    // Translation matrix
    float tMat[4][4] = {
        {1, 0, 0, -pos.x},
        {0, 1, 0, -pos.y},
        {0, 0, 1, -pos.z},
        {0, 0, 0, 1}
    };

    // Rotation matrix
    float rMat[4][4] = {
        {right.x, right.y, right.z, 0},
        {up.x, up.y, up.z, 0},
        {-forward.x, -forward.y, -forward.z, 0},
        {0, 0, 0, 1}
    };

    view = Mat4f(rMat) * Mat4f(tMat);
}

void Camera3D::updateProjection() {
    float f = 1 / tan(fov / 2);
    float ar = aspect;

    float p22 = (far + near) / (near - far);
    float p23 = (2 * far * near) / (near - far);

    float pMat[4][4] = {
        {f / ar, 0, 0, 0},
        {0, f, 0, 0},
        {0, 0, p22, p23},
        {0, 0, -1, 0}
    };
    projection = Mat4f(pMat);
}

void Camera3D::updateMVP() {
    mvp = projection * view;
}

void Camera3D::update() {
    restrictRot();

    // Model-View-Projection matrix
    updateView();
    updateProjection();
    updateMVP();

    // Camera plane
    updatePlane();
}

void Camera3D::updatePlane() {
    nearPlane = Plane3D(forward, pos + forward * near);
    farPlane = Plane3D(forward * -1, pos + forward * far);
}

// Debug
std::string Camera3D::data() {
    std::string str = "Camera3D\n";
    str += "| Pos: " + std::to_string(pos.x) + ", " + std::to_string(pos.y) + ", " + std::to_string(pos.z) + "\n";
    str += "| Rot: " + std::to_string(rot.x) + ", " + std::to_string(rot.y) + ", " + std::to_string(rot.z) + "\n";
    str += "| Fd: " + std::to_string(forward.x) + ", " + std::to_string(forward.y) + ", " + std::to_string(forward.z) + "\n";
    str += "| Rg: " + std::to_string(right.x) + ", " + std::to_string(right.y) + ", " + std::to_string(right.z) + "\n";
    str += "| Up: " + std::to_string(up.x) + ", " + std::to_string(up.y) + ", " + std::to_string(up.z) + "\n";
    str += "| Fov: " + std::to_string(fov * 180 / M_PI) + "\n";
    str += "| Asp: " + std::to_string(aspect) + "\n";
    str += "| Nrr: " + std::to_string(near) + "\n";
    str += "| Far: " + std::to_string(far) + "\n";
    return str;
}
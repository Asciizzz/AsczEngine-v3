#include "hip/hip_runtime.h"
#include <FragmentShader.cuh>

// Phong Shading

void FragmentShader::phongShading() {
    Graphic3D &graphic = Graphic3D::instance();
    Buffer3D &buffer = graphic.buffer;

    phongShadingKernel<<<buffer.blockCount, buffer.blockSize>>>(
        buffer.active, buffer.color, buffer.world, buffer.normal, buffer.texture,
        buffer.width, buffer.height
    );
    hipDeviceSynchronize();
}

__global__ void phongShadingKernel(
    bool *buffActive, Vec4f *buffColor, Vec3f *buffWorld, Vec3f *buffNormal, Vec2f *buffTexture,
    int buffWidth, int buffHeight
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= buffWidth * buffHeight || !buffActive[i]) return;
    
    // We will apply simple directional lighting
    Vec3f lightDir = Vec3f(1, 1, 1);
    Vec3f n = buffNormal[i];

    // Calculate the cosine of the angle between the normal and the light direction
    float dot = n * lightDir;
    
    float cosA = dot / (n.mag() * lightDir.mag());
    if (cosA < 0) cosA = 0;

    float diff = 0.1 + 1.1 * cosA;

    // Apply the light
    buffColor[i] = buffColor[i] * diff;
    buffColor[i].limit(0, 255);
}
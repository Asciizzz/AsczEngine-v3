#include "hip/hip_runtime.h"
#include <VertexShader.cuh>

// Render functions

Vec4f VertexShader::toScreenSpace(Camera3D &camera, Vec3f world, int buffWidth, int buffHeight) {
    Vec4f v4 = world.toVec4f();
    Vec4f t4 = camera.mvp * v4;
    Vec3f t3 = t4.toVec3f(); // Convert to NDC [-1, 1]
    Vec4f p = t3.toVec4f();
    p.w = camera.isInsideFrustum(world);

    return p;
}

// Render pipeline

void VertexShader::cameraProjection() {
    Graphic3D &graphic = Graphic3D::instance();
    Mesh3D &mesh = graphic.mesh;
    Camera3D &camera = graphic.camera;
    Buffer3D &buffer = graphic.buffer;
    Vec4f *projection = graphic.projection;

    cameraProjectionKernel<<<mesh.blockNumWs, mesh.blockSize>>>(
        projection, mesh.world, camera, buffer.width, buffer.height, mesh.numWs
    );
    hipDeviceSynchronize();
}

void VertexShader::createDepthMap() {
    Graphic3D &graphic = Graphic3D::instance();
    Mesh3D &mesh = graphic.mesh;
    Buffer3D &buffer = graphic.buffer;
    Vec4f *projection = graphic.projection;

    buffer.clearBuffer();
    buffer.nightSky(); // Cool effect

    for (int i = 0; i < 2; i++)
        createDepthMapKernel<<<mesh.blockNumFs, mesh.blockSize>>>(
            projection, mesh.world, mesh.faces, mesh.numFs,
            buffer.active, buffer.depth, buffer.faceID, buffer.bary, buffer.width, buffer.height
        );
    hipDeviceSynchronize();
}

void VertexShader::rasterization() {
    Graphic3D &graphic = Graphic3D::instance();
    Mesh3D &mesh = graphic.mesh;
    Buffer3D &buffer = graphic.buffer;

    rasterizationKernel<<<buffer.blockCount, buffer.blockSize>>>(
        mesh.world, buffer.world, mesh.wMeshId, buffer.wMeshId,
        mesh.normal, buffer.normal, mesh.nMeshId, buffer.nMeshId,
        mesh.texture, buffer.texture, mesh.tMeshId, buffer.tMeshId,
        mesh.color, buffer.color,
        mesh.faces, buffer.faceID, buffer.bary, buffer.bary,
        buffer.active, buffer.width, buffer.height
    );
    hipDeviceSynchronize();
}

// Kernels
__global__ void cameraProjectionKernel(
    Vec4f *projection, Vec3f *world, Camera3D camera, int buffWidth, int buffHeight, ULLInt numWs
) {
    ULLInt i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numWs) return;

    Vec4f p = VertexShader::toScreenSpace(camera, world[i], buffWidth, buffHeight);
    projection[i] = p;
}

__global__ void createDepthMapKernel(
    Vec4f *projection, Vec3f *world, Vec3x3uli *faces, ULLInt numFs,
    bool *buffActive, float *buffDepth, ULLInt *buffFaceId, Vec3f *buffBary,
    int buffWidth, int buffHeight
) {
    ULLInt i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numFs) return;

    Vec3uli f = faces[i].v;
    Vec4f p0 = projection[f.x];
    Vec4f p1 = projection[f.y];
    Vec4f p2 = projection[f.z];

    // Entirely outside the frustum
    if (p0.w <= 0 && p1.w <= 0 && p2.w <= 0) return;

    p0.x = (p0.x + 1) * buffWidth / 2;
    p0.y = (1 - p0.y) * buffHeight / 2;
    p1.x = (p1.x + 1) * buffWidth / 2;
    p1.y = (1 - p1.y) * buffHeight / 2;
    p2.x = (p2.x + 1) * buffWidth / 2;
    p2.y = (1 - p2.y) * buffHeight / 2;

    // Bounding box
    int minX = min(min(p0.x, p1.x), p2.x);
    int maxX = max(max(p0.x, p1.x), p2.x);
    int minY = min(min(p0.y, p1.y), p2.y);
    int maxY = max(max(p0.y, p1.y), p2.y);

    // Clip the bounding box
    minX = max(minX, 0);
    maxX = min(maxX, buffWidth - 1);
    minY = max(minY, 0);
    maxY = min(maxY, buffHeight - 1);

    for (int x = minX; x <= maxX; x++)
    for (int y = minY; y <= maxY; y++) {
        int bIdx = x + y * buffWidth;

        Vec3f bary = Vec3f::bary(
            Vec2f(x, y), Vec2f(p0.x, p0.y), Vec2f(p1.x, p1.y), Vec2f(p2.x, p2.y)
        );

        if (bary.x < 0 || bary.y < 0 || bary.z < 0) continue;

        float zDepth = bary.x * p0.z + bary.y * p1.z + bary.z * p2.z;

        if (atomicMinFloat(&buffDepth[bIdx], zDepth)) {
            buffActive[bIdx] = true;
            buffDepth[bIdx] = zDepth;
            buffFaceId[bIdx] = i;
            buffBary[bIdx] = bary;
        }
    }
}

__global__ void rasterizationKernel(
    Vec3f *world, Vec3f *buffWorld, UInt *wMeshId, UInt *buffWMeshId,
    Vec3f *normal, Vec3f *buffNormal, UInt *nMeshId, UInt *buffNMeshId,
    Vec2f *texture, Vec2f *buffTexture, UInt *tMeshId, UInt *buffTMeshId,
    Vec4f *color, Vec4f *buffColor,
    Vec3x3uli *faces, ULLInt *buffFaceId, Vec3f *bary, Vec3f *buffBary,
    bool *buffActive, int buffWidth, int buffHeight
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= buffWidth * buffHeight || !buffActive[i]) return;

    ULLInt fIdx = buffFaceId[i];

    // Set vertex, texture, and normal indices
    Vec3uli vIdx = faces[fIdx].v;
    Vec3uli tIdx = faces[fIdx].t;
    Vec3uli nIdx = faces[fIdx].n;

    // Get barycentric coordinates
    float alp = buffBary[i].x;
    float bet = buffBary[i].y;
    float gam = buffBary[i].z;

    // Set color
    Vec4f c0 = color[vIdx.x];
    Vec4f c1 = color[vIdx.y];
    Vec4f c2 = color[vIdx.z];
    buffColor[i] = c0 * alp + c1 * bet + c2 * gam;

    // Set world position
    Vec3f w0 = world[vIdx.x];
    Vec3f w1 = world[vIdx.y];
    Vec3f w2 = world[vIdx.z];
    buffWorld[i] = w0 * alp + w1 * bet + w2 * gam;

    // Set normal
    Vec3f n0 = normal[nIdx.x];
    Vec3f n1 = normal[nIdx.y];
    Vec3f n2 = normal[nIdx.z];
    n0.norm(); n1.norm(); n2.norm();
    buffNormal[i] = n0 * alp + n1 * bet + n2 * gam;
    buffNormal[i].norm();

    // Set texture
    Vec2f t0 = texture[tIdx.x];
    Vec2f t1 = texture[tIdx.y];
    Vec2f t2 = texture[tIdx.z];
    buffTexture[i] = t0 * alp + t1 * bet + t2 * gam;

    // Set mesh ID
    buffWMeshId[i] = wMeshId[vIdx.x];
    buffNMeshId[i] = nMeshId[nIdx.x];
    buffTMeshId[i] = tMeshId[tIdx.x];
}
#include "hip/hip_runtime.h"
#include <Vector.cuh>
#include <Matrix.cuh>

// VEC2f
__host__ __device__ Vec2f::Vec2f() : x(0), y(0) {}
__host__ __device__ Vec2f::Vec2f(float x, float y) : x(x), y(y) {}

// VEC3uli (unsigned long int)
__host__ __device__ Vec3uli::Vec3uli() : x(0), y(0), z(0) {}
__host__ __device__ Vec3uli::Vec3uli(int x, int y, int z) : x(x), y(y), z(z) {}
__host__ __device__ void Vec3uli::operator+=(unsigned long int d) {
    x += d; y += d; z += d;
}

// VEC3f
__host__ __device__ Vec3f::Vec3f() : x(0), y(0), z(0) {}
__host__ __device__ Vec3f::Vec3f(float x, float y, float z) : x(x), y(y), z(z) {}
__host__ __device__ Vec4f Vec3f::toVec4f() {
    return Vec4f(x, y, z, 1);
}

__host__ __device__ Vec3f Vec3f::operator+(const Vec3f& v) {
    return Vec3f(x + v.x, y + v.y, z + v.z);
}
__host__ __device__ Vec3f Vec3f::operator-(const Vec3f& v) {
    return Vec3f(x - v.x, y - v.y, z - v.z);
}
__host__ __device__ Vec3f Vec3f::operator*(const float scalar) {
    return Vec3f(x * scalar, y * scalar, z * scalar);
}
__host__ __device__ void Vec3f::operator+=(const Vec3f& v) {
    x += v.x; y += v.y; z += v.z;
}
__host__ __device__ void Vec3f::operator-=(const Vec3f& v) {
    x -= v.x; y -= v.y; z -= v.z;
}
__host__ __device__ void Vec3f::operator*=(const float scalar) {
    x *= scalar; y *= scalar; z *= scalar;
}

__host__ __device__ float Vec3f::operator*(const Vec3f& v) {
    return x * v.x + y * v.y + z * v.z;
}
__host__ __device__ Vec3f Vec3f::operator&(const Vec3f& v) {
    return Vec3f(y * v.z - z * v.y, z * v.x - x * v.z, x * v.y - y * v.x);
}
__host__ __device__ float Vec3f::mag() {
    return sqrt(x * x + y * y + z * z);
}
__host__ __device__ void Vec3f::norm() {
    float m = mag();
    x /= m; y /= m; z /= m;
}

// Transformations
__host__ __device__ Vec3f Vec3f::translate(Vec3f& vec, const Vec3f& t) {
    return vec + t;
}
__host__ __device__ Vec3f Vec3f::rotate(Vec3f& vec, const Vec3f& origin, const Vec3f& rot) {
    // Translate to origin
    Vec3f diff = vec - origin;

    float cosX = cos(rot.x), sinX = sin(rot.x);
    float cosY = cos(rot.y), sinY = sin(rot.y);
    float cosZ = cos(rot.z), sinZ = sin(rot.z);

    // Rotation matrices
    float rX[4][4] = {
        {1, 0, 0, 0},
        {0, cosX, -sinX, 0},
        {0, sinX, cosX, 0},
        {0, 0, 0, 1}
    };
    float rY[4][4] = {
        {cosY, 0, sinY, 0},
        {0, 1, 0, 0},
        {-sinY, 0, cosY, 0},
        {0, 0, 0, 1}
    };
    float rZ[4][4] = {
        {cosZ, -sinZ, 0, 0},
        {sinZ, cosZ, 0, 0},
        {0, 0, 1, 0},
        {0, 0, 0, 1}
    };
    Mat4f rMat = Mat4f(rX) * Mat4f(rY) * Mat4f(rZ);

    Vec4f rVec4 = rMat * diff.toVec4f();
    Vec3f rVec3 = rVec4.toVec3f();
    rVec3 += origin;

    return rVec3;
}
__host__ __device__ Vec3f Vec3f::scale(Vec3f& vec, const Vec3f& origin, const Vec3f& scl) {
    Vec3f diff = vec - origin;
    return Vec3f(
        origin.x + diff.x * scl.x,
        origin.y + diff.y * scl.y,
        origin.z + diff.z * scl.z
    );
}
__host__ __device__ Vec3f Vec3f::scale(Vec3f& vec, const Vec3f& origin, const float scl) {
    return scale(vec, origin, Vec3f(scl, scl, scl));
}

// Transformations but on self
__host__ __device__ void Vec3f::translate(const Vec3f& t) {
    *this += t;
}
__host__ __device__ void Vec3f::rotate(const Vec3f& origin, const Vec3f& rot) {
    *this = rotate(*this, origin, rot);
}
__host__ __device__ void Vec3f::scale(const Vec3f& origin, const Vec3f& scl) {
    *this = scale(*this, origin, scl);
}
__host__ __device__ void Vec3f::scale(const Vec3f& origin, const float scl) {
    *this = scale(*this, origin, scl);
}

// VEC4
__host__ __device__ Vec4f::Vec4f() : x(0), y(0), z(0), w(0) {}
__host__ __device__ Vec4f::Vec4f(float x, float y, float z, float w) : x(x), y(y), z(z), w(w) {}
__host__ __device__ Vec3f Vec4f::toVec3f() {
    return Vec3f(x / w, y / w, z / w);
}